#define PETSC_SKIP_COMPLEX
#define PETSC_SKIP_SPINLOCK

#include <petscconf.h>
#include <../src/mat/impls/aij/mpi/mpiaij.h>   /*I "petscmat.h" I*/
#include <../src/mat/impls/aij/mpi/mpicusparse/mpicusparsematimpl.h>


#undef __FUNCT__
#define __FUNCT__ "MatMPIAIJSetPreallocation_MPIAIJCUSPARSE"
PetscErrorCode  MatMPIAIJSetPreallocation_MPIAIJCUSPARSE(Mat B,PetscInt d_nz,const PetscInt d_nnz[],PetscInt o_nz,const PetscInt o_nnz[])
{
  Mat_MPIAIJ         *b               = (Mat_MPIAIJ*)B->data;
  Mat_MPIAIJCUSPARSE * cusparseStruct = (Mat_MPIAIJCUSPARSE*)b->spptr;
  PetscErrorCode     ierr;
  PetscInt           i;

  PetscFunctionBegin;
  if (d_nz == PETSC_DEFAULT || d_nz == PETSC_DECIDE) d_nz = 5;
  if (o_nz == PETSC_DEFAULT || o_nz == PETSC_DECIDE) o_nz = 2;
  if (d_nz < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"d_nz cannot be less than 0: value %D",d_nz);
  if (o_nz < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"o_nz cannot be less than 0: value %D",o_nz);

  ierr = PetscLayoutSetUp(B->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(B->cmap);CHKERRQ(ierr);
  if (d_nnz) {
    for (i=0; i<B->rmap->n; i++) {
      if (d_nnz[i] < 0) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"d_nnz cannot be less than 0: local row %D value %D",i,d_nnz[i]);
    }
  }
  if (o_nnz) {
    for (i=0; i<B->rmap->n; i++) {
      if (o_nnz[i] < 0) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"o_nnz cannot be less than 0: local row %D value %D",i,o_nnz[i]);
    }
  }
  if (!B->preallocated) {
    /* Explicitly create 2 MATSEQAIJCUSPARSE matrices. */
    ierr = MatCreate(PETSC_COMM_SELF,&b->A);CHKERRQ(ierr);
    ierr = MatSetSizes(b->A,B->rmap->n,B->cmap->n,B->rmap->n,B->cmap->n);CHKERRQ(ierr);
    ierr = MatSetType(b->A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)B,(PetscObject)b->A);CHKERRQ(ierr);
    ierr = MatCreate(PETSC_COMM_SELF,&b->B);CHKERRQ(ierr);
    ierr = MatSetSizes(b->B,B->rmap->n,B->cmap->N,B->rmap->n,B->cmap->N);CHKERRQ(ierr);
    ierr = MatSetType(b->B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)B,(PetscObject)b->B);CHKERRQ(ierr);
  }
  ierr = MatSeqAIJSetPreallocation(b->A,d_nz,d_nnz);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation(b->B,o_nz,o_nnz);CHKERRQ(ierr);
  ierr = MatCUSPARSESetFormat(b->A,MAT_CUSPARSE_MULT,cusparseStruct->diagGPUMatFormat);CHKERRQ(ierr);
  ierr = MatCUSPARSESetFormat(b->B,MAT_CUSPARSE_MULT,cusparseStruct->offdiagGPUMatFormat);CHKERRQ(ierr);
  ierr = MatCUSPARSESetHandle(b->A,cusparseStruct->handle);CHKERRQ(ierr);
  ierr = MatCUSPARSESetHandle(b->B,cusparseStruct->handle);CHKERRQ(ierr);
  ierr = MatCUSPARSESetStream(b->A,cusparseStruct->stream);CHKERRQ(ierr);
  ierr = MatCUSPARSESetStream(b->B,cusparseStruct->stream);CHKERRQ(ierr);

  B->preallocated = PETSC_TRUE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCreateVecs_MPIAIJCUSPARSE"
PetscErrorCode  MatCreateVecs_MPIAIJCUSPARSE(Mat mat,Vec *right,Vec *left)
{
  PetscErrorCode ierr;
  PetscInt rbs,cbs;

  PetscFunctionBegin;
  ierr = MatGetBlockSizes(mat,&rbs,&cbs);CHKERRQ(ierr);
  if (right) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),right);CHKERRQ(ierr);
    ierr = VecSetSizes(*right,mat->cmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*right,cbs);CHKERRQ(ierr);
    ierr = VecSetType(*right,VECCUSP);CHKERRQ(ierr);
    ierr = VecSetLayout(*right,mat->cmap);CHKERRQ(ierr);
  }
  if (left) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),left);CHKERRQ(ierr);
    ierr = VecSetSizes(*left,mat->rmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*left,rbs);CHKERRQ(ierr);
    ierr = VecSetType(*left,VECCUSP);CHKERRQ(ierr);
    ierr = VecSetLayout(*left,mat->rmap);CHKERRQ(ierr);


  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMult_MPIAIJCUSPARSE"
PetscErrorCode MatMult_MPIAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  /* This multiplication sequence is different sequence
     than the CPU version. In particular, the diagonal block
     multiplication kernel is launched in one stream. Then,
     in a separate stream, the data transfers from DeviceToHost
     (with MPI messaging in between), then HostToDevice are
     launched. Once the data transfer stream is synchronized,
     to ensure messaging is complete, the MatMultAdd kernel
     is launched in the original (MatMult) stream to protect
     against race conditions.

     This sequence should only be called for GPU computation. */
  Mat_MPIAIJ     *a = (Mat_MPIAIJ*)A->data;
  PetscErrorCode ierr;
  PetscInt       nt;

  PetscFunctionBegin;
  ierr = VecGetLocalSize(xx,&nt);CHKERRQ(ierr);
  if (nt != A->cmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Incompatible partition of A (%D) and xx (%D)",A->cmap->n,nt);
  ierr = VecScatterInitializeForGPU(a->Mvctx,xx,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = (*a->A->ops->mult)(a->A,xx,yy);CHKERRQ(ierr);
  ierr = VecScatterBegin(a->Mvctx,xx,a->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = VecScatterEnd(a->Mvctx,xx,a->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = (*a->B->ops->multadd)(a->B,a->lvec,yy,yy);CHKERRQ(ierr);
  ierr = VecScatterFinalizeForGPU(a->Mvctx);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMult_MPIAIJCUSPARSE"
PetscErrorCode MatMultTranspose_MPIAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  /* This multiplication sequence is different sequence
     than the CPU version. In particular, the diagonal block
     multiplication kernel is launched in one stream. Then,
     in a separate stream, the data transfers from DeviceToHost
     (with MPI messaging in between), then HostToDevice are
     launched. Once the data transfer stream is synchronized,
     to ensure messaging is complete, the MatMultAdd kernel
     is launched in the original (MatMult) stream to protect
     against race conditions.

     This sequence should only be called for GPU computation. */
  Mat_MPIAIJ     *a = (Mat_MPIAIJ*)A->data;
  PetscErrorCode ierr;
  PetscInt       nt;

  PetscFunctionBegin;
  ierr = VecGetLocalSize(xx,&nt);CHKERRQ(ierr);
  if (nt != A->cmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Incompatible partition of A (%D) and xx (%D)",A->cmap->n,nt);
  ierr = VecScatterInitializeForGPU(a->Mvctx,xx,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = (*a->A->ops->multtranspose)(a->A,xx,yy);CHKERRQ(ierr);
  ierr = VecScatterBegin(a->Mvctx,xx,a->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = VecScatterEnd(a->Mvctx,xx,a->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = (*a->B->ops->multtransposeadd)(a->B,a->lvec,yy,yy);CHKERRQ(ierr);
  ierr = VecScatterFinalizeForGPU(a->Mvctx);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSESetFormat_MPIAIJCUSPARSE"
PetscErrorCode MatCUSPARSESetFormat_MPIAIJCUSPARSE(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  Mat_MPIAIJ         *a               = (Mat_MPIAIJ*)A->data;
  Mat_MPIAIJCUSPARSE * cusparseStruct = (Mat_MPIAIJCUSPARSE*)a->spptr;

  PetscFunctionBegin;
  switch (op) {
  case MAT_CUSPARSE_MULT_DIAG:
    cusparseStruct->diagGPUMatFormat = format;
    break;
  case MAT_CUSPARSE_MULT_OFFDIAG:
    cusparseStruct->offdiagGPUMatFormat = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparseStruct->diagGPUMatFormat    = format;
    cusparseStruct->offdiagGPUMatFormat = format;
    break;
  default:
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPARSEFormatOperation. Only MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_DIAG, and MAT_CUSPARSE_MULT_ALL are currently supported.",op);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSetFromOptions_MPIAIJCUSPARSE"
PetscErrorCode MatSetFromOptions_MPIAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat A)
{
  MatCUSPARSEStorageFormat format;
  PetscErrorCode           ierr;
  PetscBool                flg;
  Mat_MPIAIJ               *a = (Mat_MPIAIJ*)A->data;
  Mat_MPIAIJCUSPARSE       *cusparseStruct = (Mat_MPIAIJCUSPARSE*)a->spptr;

  PetscFunctionBegin;
  ierr = PetscOptionsHead(PetscOptionsObject,"MPIAIJCUSPARSE options");CHKERRQ(ierr);
  ierr = PetscObjectOptionsBegin((PetscObject)A);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = PetscOptionsEnum("-mat_cusparse_mult_diag_storage_format","sets storage format of the diagonal blocks of (mpi)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparseStruct->diagGPUMatFormat,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT_DIAG,format);CHKERRQ(ierr);
    }
    ierr = PetscOptionsEnum("-mat_cusparse_mult_offdiag_storage_format","sets storage format of the off-diagonal blocks (mpi)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparseStruct->offdiagGPUMatFormat,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT_OFFDIAG,format);CHKERRQ(ierr);
    }
    ierr = PetscOptionsEnum("-mat_cusparse_storage_format","sets storage format of the diagonal and off-diagonal blocks (mpi)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparseStruct->diagGPUMatFormat,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_ALL,format);CHKERRQ(ierr);
    }
  }
  ierr = PetscOptionsEnd();CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatDestroy_MPIAIJCUSPARSE"
PetscErrorCode MatDestroy_MPIAIJCUSPARSE(Mat A)
{
  PetscErrorCode     ierr;
  Mat_MPIAIJ         *a              = (Mat_MPIAIJ*)A->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE*)a->spptr;
  hipError_t        err;
  hipsparseStatus_t   stat;

  PetscFunctionBegin;
  try {
    ierr = MatCUSPARSEClearHandle(a->A);CHKERRQ(ierr);
    ierr = MatCUSPARSEClearHandle(a->B);CHKERRQ(ierr);
    stat = hipsparseDestroy(cusparseStruct->handle);CHKERRCUSP(stat);
    err = hipStreamDestroy(cusparseStruct->stream);CHKERRCUSP(err);
    delete cusparseStruct;
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Mat_MPIAIJCUSPARSE error: %s", ex);
  }
  cusparseStruct = 0;

  ierr = MatDestroy_MPIAIJ(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCreate_MPIAIJCUSPARSE"
PETSC_EXTERN PetscErrorCode MatCreate_MPIAIJCUSPARSE(Mat A)
{
  PetscErrorCode     ierr;
  Mat_MPIAIJ         *a;
  Mat_MPIAIJCUSPARSE * cusparseStruct;
  hipError_t        err;
  hipsparseStatus_t   stat;

  PetscFunctionBegin;
  ierr = MatCreate_MPIAIJ(A);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatMPIAIJSetPreallocation_C",MatMPIAIJSetPreallocation_MPIAIJCUSPARSE);CHKERRQ(ierr);
  a        = (Mat_MPIAIJ*)A->data;
  a->spptr = new Mat_MPIAIJCUSPARSE;

  cusparseStruct                      = (Mat_MPIAIJCUSPARSE*)a->spptr;
  cusparseStruct->diagGPUMatFormat    = MAT_CUSPARSE_CSR;
  cusparseStruct->offdiagGPUMatFormat = MAT_CUSPARSE_CSR;
  stat = hipsparseCreate(&(cusparseStruct->handle));CHKERRCUSP(stat);
  err = hipStreamCreate(&(cusparseStruct->stream));CHKERRCUSP(err);

  A->ops->getvecs        = MatCreateVecs_MPIAIJCUSPARSE;
  A->ops->mult           = MatMult_MPIAIJCUSPARSE;
  A->ops->multtranspose  = MatMultTranspose_MPIAIJCUSPARSE;
  A->ops->setfromoptions = MatSetFromOptions_MPIAIJCUSPARSE;
  A->ops->destroy        = MatDestroy_MPIAIJCUSPARSE;

  ierr = PetscObjectChangeTypeName((PetscObject)A,MATMPIAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatCUSPARSESetFormat_C",  MatCUSPARSESetFormat_MPIAIJCUSPARSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*@
   MatCreateAIJCUSPARSE - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format).  This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSPARSE library for calculations. For good matrix
   assembly performance the user should preallocate the matrix storage by setting
   the parameter nz (or the array nnz).  By setting these parameters accurately,
   performance during matrix assembly can be increased by more than a factor of 50.

   Collective on MPI_Comm

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows
         (possibly different for each row) or NULL

   Output Parameter:
.  A - the matrix

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradigm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=NULL for PETSc to control dynamic memory
   allocation.  For large problems you MUST preallocate memory or you
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to
   improve numerical efficiency of matrix-vector products and solves. We
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ(), MATMPIAIJCUSPARSE, MATAIJCUSPARSE
@*/
#undef __FUNCT__
#define __FUNCT__ "MatCreateAIJCUSPARSE"
PetscErrorCode  MatCreateAIJCUSPARSE(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt M,PetscInt N,PetscInt d_nz,const PetscInt d_nnz[],PetscInt o_nz,const PetscInt o_nnz[],Mat *A)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;

  PetscFunctionBegin;
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,M,N);CHKERRQ(ierr);
  ierr = MPI_Comm_size(comm,&size);CHKERRQ(ierr);
  if (size > 1) {
    ierr = MatSetType(*A,MATMPIAIJCUSPARSE);CHKERRQ(ierr);
    ierr = MatMPIAIJSetPreallocation(*A,d_nz,d_nnz,o_nz,o_nnz);CHKERRQ(ierr);
  } else {
    ierr = MatSetType(*A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
    ierr = MatSeqAIJSetPreallocation(*A,d_nz,d_nnz);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*M
   MATAIJCUSPARSE - MATMPIAIJCUSPARSE = "aijcusparse" = "mpiaijcusparse" - A matrix type to be used for sparse matrices.

   A matrix type type whose data resides on Nvidia GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. The ELL and HYB formats require CUDA 4.2 or later.
   All matrix calculations are performed on Nvidia GPUs using the CUSPARSE library.

   This matrix type is identical to MATSEQAIJCUSPARSE when constructed with a single process communicator,
   and MATMPIAIJCUSPARSE otherwise.  As a result, for single process communicators,
   MatSeqAIJSetPreallocation is supported, and similarly MatMPIAIJSetPreallocation is supported
   for communicators controlling multiple processes.  It is recommended that you call both of
   the above preallocation routines for simplicity.

   Options Database Keys:
+  -mat_type mpiaijcusparse - sets the matrix type to "mpiaijcusparse" during a call to MatSetFromOptions()
.  -mat_cusparse_storage_format csr - sets the storage format of diagonal and off-diagonal matrices during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
.  -mat_cusparse_mult_diag_storage_format csr - sets the storage format of diagonal matrix during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
-  -mat_cusparse_mult_offdiag_storage_format csr - sets the storage format of off-diagonal matrix during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).

  Level: beginner

 .seealso: MatCreateAIJCUSPARSE(), MATSEQAIJCUSPARSE, MatCreateSeqAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M
M*/
