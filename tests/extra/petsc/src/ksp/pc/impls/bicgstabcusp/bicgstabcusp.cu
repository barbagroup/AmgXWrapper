#include "hip/hip_runtime.h"

/*  -------------------------------------------------------------------- */

/*
   Include files needed for the CUSP BiCGSTAB preconditioner:
     pcimpl.h - private include file intended for use by all preconditioners
*/
#define PETSC_SKIP_SPINLOCK

#include <petsc/private/pcimpl.h>   /*I "petscpc.h" I*/
#include <../src/mat/impls/aij/seq/aij.h>
#include <cusp/monitor.h>
#include <cusp/krylov/bicgstab.h>
#include <../src/vec/vec/impls/dvecimpl.h>
#include <../src/mat/impls/aij/seq/seqcusp/cuspmatimpl.h>


/*
   Private context (data structure) for the CUSP BiCGStab preconditioner.
 */
typedef struct {
  PetscInt   maxits;
  PetscReal  rtol;
  PetscBool  monitorverbose;
  CUSPMATRIX * mat;
} PC_BiCGStabCUSP;

#undef __FUNCT__
#define __FUNCT__ "PCBiCGStabCUSPSetTolerance_BiCGStabCUSP"
static PetscErrorCode PCBiCGStabCUSPSetTolerance_BiCGStabCUSP(PC pc,PetscReal rtol)
{
  PC_BiCGStabCUSP *bicg = (PC_BiCGStabCUSP*)pc->data;

  PetscFunctionBegin;
  bicg->rtol = rtol;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PCBiCGStabCUSPSetUseVerboseMonitor_BiCGStabCUSP"
static PetscErrorCode PCBiCGStabCUSPSetUseVerboseMonitor_BiCGStabCUSP(PC pc, PetscBool useverbose)
{
  PC_BiCGStabCUSP *bicg = (PC_BiCGStabCUSP*)pc->data;

  PetscFunctionBegin;
  bicg->monitorverbose = useverbose;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PCBiCGStabCUSPSetUseVerboseMonitor"
PetscErrorCode PCBiCGStabCUSPSetUseVerboseMonitor(PC pc, PetscBool useverbose)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(pc, PC_CLASSID,1);
  ierr = PetscTryMethod(pc, "PCBiCGStabCUSPSetUseVerboseMonitors_C",(PC,PetscBool),(pc,useverbose));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PCBiCGStabCUSPSetIterations_BiCGStabCUSP"
static PetscErrorCode PCBiCGStabCUSPSetIterations_BiCGStabCUSP(PC pc, PetscInt its)
{
  PC_BiCGStabCUSP *bicg = (PC_BiCGStabCUSP*)pc->data;

  PetscFunctionBegin;
  bicg->maxits = its;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PCBiCGStabCUSPSetIterations"
PetscErrorCode PCBiCGStabCUSPSetITerations(PC pc, PetscInt its)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(pc,PC_CLASSID,1);
  ierr = PetscTryMethod(pc, "PCBiCGStabCUSPSetIterations_C",(PC,PetscInt),(pc,its));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PCBiCGStabCUSPSetTolerance"
PetscErrorCode PCBiCGStabCUSPSetTolerance(PC pc, PetscReal rtol)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(pc,PC_CLASSID,1);
  ierr = PetscTryMethod(pc, "PCBiCGStabCUSPSetTolerance_C",(PC,PetscReal),(pc,rtol));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* -------------------------------------------------------------------------- */
/*
   PCSetUp_BiCGStabCUSP - Prepares for the use of the CUSP BiCGStab preconditioner
                    by setting data structures and options.

   Input Parameter:
.  pc - the preconditioner context

   Application Interface Routine: PCSetUp()

   Notes:
   The interface routine PCSetUp() is not usually called directly by
   the user, but instead is called by PCApply() if necessary.
*/
#undef __FUNCT__
#define __FUNCT__ "PCSetUp_BiCGStabCUSP"
static PetscErrorCode PCSetUp_BiCGStabCUSP(PC pc)
{
  PC_BiCGStabCUSP *bicg = (PC_BiCGStabCUSP*)pc->data;
  PetscBool       flg   = PETSC_FALSE;
  Mat_SeqAIJCUSP  *gpustruct;
  PetscErrorCode  ierr;

  PetscFunctionBegin;
  ierr = PetscObjectTypeCompare((PetscObject)pc->pmat,MATSEQAIJCUSP,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ(PetscObjectComm((PetscObject)pc),PETSC_ERR_SUP,"Currently only handles CUSP matrices");
  try {
    ierr      = MatCUSPCopyToGPU(pc->pmat);CHKERRQ(ierr);
    gpustruct = (Mat_SeqAIJCUSP*)(pc->pmat->spptr);
    bicg->mat = (CUSPMATRIX*)gpustruct->mat;
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s",ex);
  }
  PetscFunctionReturn(0);
}

/* -------------------------------------------------------------------------- */
/*
   PCApply_BiCGStabCUSP - Applies the BiCGStabCUSP preconditioner to a vector.

   Input Parameters:
.  pc - the preconditioner context
.  x - input vector

   Output Parameter:
.  y - output vector

   Application Interface Routine: PCApply()
 */
#undef __FUNCT__
#define __FUNCT__ "PCApply_BiCGStabCUSP"
static PetscErrorCode PCApply_BiCGStabCUSP(PC pc,Vec x,Vec y)
{
  PC_BiCGStabCUSP *bicg = (PC_BiCGStabCUSP*)pc->data;
  PetscErrorCode  ierr;
  PetscBool       flg1,flg2;
  CUSPARRAY       *xarray=NULL,*yarray=NULL;

  PetscFunctionBegin;
  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQCUSP,&flg1);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)y,VECSEQCUSP,&flg2);CHKERRQ(ierr);
  if (!(flg1 && flg2)) SETERRQ(PetscObjectComm((PetscObject)pc),PETSC_ERR_SUP, "Currently only handles CUSP vectors");
  if (!bicg->mat) {
    ierr = PCSetUp_BiCGStabCUSP(pc);CHKERRQ(ierr);
  }
  ierr = VecSet(y,0.0);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(x,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(y,&yarray);CHKERRQ(ierr);
  try {
#if defined(CUSP_VERSION) && CUSP_VERSION >= 500
    cusp::monitor<PetscReal> monitor(*xarray,bicg->maxits,bicg->rtol);
    cusp::krylov::bicgstab(*bicg->mat,*yarray,*xarray,monitor);
#else
    cusp::default_monitor<PetscReal> monitor(*xarray,bicg->maxits,bicg->rtol);
    if (bicg->monitorverbose) {
      cusp::verbose_monitor<PetscReal> verbosemonitor(*xarray,bicg->maxits,bicg->rtol);
      cusp::krylov::bicgstab(*bicg->mat,*yarray,*xarray,verbosemonitor);
    } else {
      cusp::krylov::bicgstab(*bicg->mat,*yarray,*xarray,monitor);
    }
#endif
  } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(x,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(y,&yarray);CHKERRQ(ierr);
  ierr = PetscObjectStateIncrease((PetscObject)y);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
/* -------------------------------------------------------------------------- */
/*
   PCDestroy_BiCGStabCUSP - Destroys the private context for the BiCGStabCUSP preconditioner
   that was created with PCCreate_BiCGStabCUSP().

   Input Parameter:
.  pc - the preconditioner context

   Application Interface Routine: PCDestroy()
*/
#undef __FUNCT__
#define __FUNCT__ "PCDestroy_BiCGStabCUSP"
static PetscErrorCode PCDestroy_BiCGStabCUSP(PC pc)
{
  PetscErrorCode  ierr;

  PetscFunctionBegin;
  /*
      Free the private data structure that was hanging off the PC
  */
  ierr = PetscFree(pc->data);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PCSetFromOptions_BiCGStabCUSP"
static PetscErrorCode PCSetFromOptions_BiCGStabCUSP(PetscOptionItems *PetscOptionsObject,PC pc)
{
  PC_BiCGStabCUSP *bicg = (PC_BiCGStabCUSP*)pc->data;
  PetscErrorCode  ierr;

  PetscFunctionBegin;
  ierr = PetscOptionsHead(PetscOptionsObject,"BiCGStabCUSP options");CHKERRQ(ierr);
  ierr = PetscOptionsReal("-pc_bicgstabcusp_rtol","relative tolerance for BiCGStabCUSP preconditioner","PCBiCGStabCUSPSetTolerance",bicg->rtol,&bicg->rtol,0);CHKERRQ(ierr);
  ierr = PetscOptionsInt("-pc_bicgstabcusp_max_it","maximum iterations for BiCGStabCUSP preconditioner","PCBiCGStabCUSPSetIterations",bicg->maxits,&bicg->maxits,0);CHKERRQ(ierr);
  ierr = PetscOptionsBool("-pc_bicgstabcusp_monitor_verbose","Print information about GPU BiCGStabCUSP iterations","PCBiCGStabCUSPSetUseVerboseMonitor",bicg->monitorverbose,&bicg->monitorverbose,0);CHKERRQ(ierr);
  ierr = PetscOptionsTail();CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* -------------------------------------------------------------------------- */

#undef __FUNCT__
#define __FUNCT__ "PCCreate_BiCGStabCUSP"
PETSC_EXTERN PetscErrorCode PCCreate_BiCGStabCUSP(PC pc)
{
  PC_BiCGStabCUSP *bicg;
  PetscErrorCode  ierr;

  PetscFunctionBegin;
  /*
     Creates the private data structure for this preconditioner and
     attach it to the PC object.
   */
  ierr = PetscNewLog(pc,&bicg);CHKERRQ(ierr);
  /*
     Set default values.  We don't actually want to set max iterations as far as I know, but the Cusp monitor requires them so we use a large number.
   */
  bicg->maxits         = 1000;
  bicg->rtol           = 1.e-1;
  bicg->monitorverbose = PETSC_FALSE;
  pc->data             = (void*)bicg;
  /*
      Set the pointers for the functions that are provided above.
      Now when the user-level routines (such as PCApply(), PCDestroy(), etc.)
      are called, they will automatically call these functions.  Note we
      choose not to provide a couple of these functions since they are
      not needed.
  */
  pc->ops->apply               = PCApply_BiCGStabCUSP;
  pc->ops->applytranspose      = 0;
  pc->ops->setup               = PCSetUp_BiCGStabCUSP;
  pc->ops->destroy             = PCDestroy_BiCGStabCUSP;
  pc->ops->setfromoptions      = PCSetFromOptions_BiCGStabCUSP;
  pc->ops->view                = 0;
  pc->ops->applyrichardson     = 0;
  pc->ops->applysymmetricleft  = 0;
  pc->ops->applysymmetricright = 0;

  ierr = PetscObjectComposeFunction((PetscObject)pc,"PCBiCGStabCUSPSetTolerance_C",PCBiCGStabCUSPSetTolerance_BiCGStabCUSP);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)pc, "PCBiCGStabCUSPSetIterations_C",PCBiCGStabCUSPSetIterations_BiCGStabCUSP);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)pc, "PCBiCGStabCUSPSetUseVerboseMonitor_C", PCBiCGStabCUSPSetUseVerboseMonitor_BiCGStabCUSP);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

