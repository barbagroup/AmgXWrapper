#include "hip/hip_runtime.h"
/*
   Implements the sequential cusp vectors.
*/

#define PETSC_SKIP_COMPLEX
#define PETSC_SKIP_SPINLOCK

#include <petscconf.h>
#include <petsc/private/vecimpl.h>          /*I "petscvec.h" I*/
#include <../src/vec/vec/impls/dvecimpl.h>
#include <../src/vec/vec/impls/seq/seqcusp/cuspvecimpl.h>

#include <hip/hip_runtime.h>

#undef __FUNCT__
#define __FUNCT__ "VecCUSPAllocateCheckHost"
/*
    Allocates space for the vector array on the Host if it does not exist.
    Does NOT change the PetscCUSPFlag for the vector
    Does NOT zero the CUSP array
 */
PetscErrorCode VecCUSPAllocateCheckHost(Vec v)
{
  PetscErrorCode ierr;
  PetscScalar    *array;
  Vec_Seq        *s = (Vec_Seq*)v->data;
  PetscInt       n = v->map->n;

  PetscFunctionBegin;
  if (!s) {
    ierr = PetscNewLog((PetscObject)v,&s);CHKERRQ(ierr);
    v->data = s;
  }
  if (!s->array) {
    ierr               = PetscMalloc1(n,&array);CHKERRQ(ierr);
    ierr               = PetscLogObjectMemory((PetscObject)v,n*sizeof(PetscScalar));CHKERRQ(ierr);
    s->array           = array;
    s->array_allocated = array;
    if (v->valid_GPU_array == PETSC_CUSP_UNALLOCATED) {
      v->valid_GPU_array = PETSC_CUSP_CPU;
    }
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPAllocateCheck"
/*
    Allocates space for the vector array on the GPU if it does not exist.
    Does NOT change the PetscCUSPFlag for the vector
    Does NOT zero the CUSP array

 */
PetscErrorCode VecCUSPAllocateCheck(Vec v)
{
  hipError_t    err;
  hipStream_t   stream;
  Vec_CUSP       *veccusp;

  PetscFunctionBegin;
  if (!v->spptr) {
    try {
      v->spptr = new Vec_CUSP;
      veccusp = (Vec_CUSP*)v->spptr;
      veccusp->GPUarray = new CUSPARRAY;
      veccusp->GPUarray->resize((PetscBLASInt)v->map->n);
      err = hipStreamCreate(&stream);CHKERRCUSP(err);
      veccusp->stream = stream;
      veccusp->hostDataRegisteredAsPageLocked = PETSC_FALSE;
      v->ops->destroy = VecDestroy_SeqCUSP;
      if (v->valid_GPU_array == PETSC_CUSP_UNALLOCATED) {
        if (v->data && ((Vec_Seq*)v->data)->array) {
          v->valid_GPU_array = PETSC_CUSP_CPU;
        } else {
          v->valid_GPU_array = PETSC_CUSP_GPU;
        }
      }
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCUSPCopyToGPU"
/* Copies a vector from the CPU to the GPU unless we already have an up-to-date copy on the GPU */
PetscErrorCode VecCUSPCopyToGPU(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUSP       *veccusp;
  CUSPARRAY      *varray;

  PetscFunctionBegin;
  ierr = VecCUSPAllocateCheck(v);CHKERRQ(ierr);
  if (v->valid_GPU_array == PETSC_CUSP_CPU) {
    ierr = PetscLogEventBegin(VEC_CUSPCopyToGPU,v,0,0,0);CHKERRQ(ierr);
    try {
      veccusp=(Vec_CUSP*)v->spptr;
      varray=veccusp->GPUarray;
      err = hipMemcpy(varray->data().get(),((Vec_Seq*)v->data)->array,v->map->n*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUSP(err);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = PetscLogEventEnd(VEC_CUSPCopyToGPU,v,0,0,0);CHKERRQ(ierr);
    v->valid_GPU_array = PETSC_CUSP_BOTH;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPCopyToGPUSome"
static PetscErrorCode VecCUSPCopyToGPUSome(Vec v, PetscCUSPIndices ci)
{
  CUSPARRAY      *varray;
  PetscErrorCode ierr;
  hipError_t    err;
  PetscScalar    *cpuPtr, *gpuPtr;
  Vec_Seq        *s;
  VecScatterCUSPIndices_PtoP ptop_scatter = (VecScatterCUSPIndices_PtoP)ci->scatter;

  PetscFunctionBegin;
  ierr = VecCUSPAllocateCheck(v);CHKERRQ(ierr);
  if (v->valid_GPU_array == PETSC_CUSP_CPU) {
    s = (Vec_Seq*)v->data;

    ierr   = PetscLogEventBegin(VEC_CUSPCopyToGPUSome,v,0,0,0);CHKERRQ(ierr);
    varray = ((Vec_CUSP*)v->spptr)->GPUarray;
    gpuPtr = varray->data().get() + ptop_scatter->recvLowestIndex;
    cpuPtr = s->array + ptop_scatter->recvLowestIndex;

    /* Note : this code copies the smallest contiguous chunk of data
       containing ALL of the indices */
    err = hipMemcpy(gpuPtr,cpuPtr,ptop_scatter->nr*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUSP(err);

    // Set the buffer states
    v->valid_GPU_array = PETSC_CUSP_BOTH;
    ierr = PetscLogEventEnd(VEC_CUSPCopyToGPUSome,v,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCUSPCopyFromGPU"
/*
     VecCUSPCopyFromGPU - Copies a vector from the GPU to the CPU unless we already have an up-to-date copy on the CPU
*/
PetscErrorCode VecCUSPCopyFromGPU(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUSP       *veccusp;
  CUSPARRAY      *varray;

  PetscFunctionBegin;
  ierr = VecCUSPAllocateCheckHost(v);CHKERRQ(ierr);
  if (v->valid_GPU_array == PETSC_CUSP_GPU) {
    ierr = PetscLogEventBegin(VEC_CUSPCopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    try {
      veccusp=(Vec_CUSP*)v->spptr;
      varray=veccusp->GPUarray;
      err = hipMemcpy(((Vec_Seq*)v->data)->array,varray->data().get(),v->map->n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUSP(err);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = PetscLogEventEnd(VEC_CUSPCopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    v->valid_GPU_array = PETSC_CUSP_BOTH;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPCopyFromGPUSome"
/* Note that this function only copies *some* of the values up from the GPU to CPU,
   which means that we need recombine the data at some point before using any of the standard functions.
   We could add another few flag-types to keep track of this, or treat things like VecGetArray VecRestoreArray
   where you have to always call in pairs
*/
PetscErrorCode VecCUSPCopyFromGPUSome(Vec v, PetscCUSPIndices ci)
{
  CUSPARRAY      *varray;
  PetscErrorCode ierr;
  hipError_t    err;
  PetscScalar    *cpuPtr, *gpuPtr;
  Vec_Seq        *s;
  VecScatterCUSPIndices_PtoP ptop_scatter = (VecScatterCUSPIndices_PtoP)ci->scatter;

  PetscFunctionBegin;
  ierr = VecCUSPAllocateCheckHost(v);CHKERRQ(ierr);
  if (v->valid_GPU_array == PETSC_CUSP_GPU) {
    ierr   = PetscLogEventBegin(VEC_CUSPCopyFromGPUSome,v,0,0,0);CHKERRQ(ierr);

    varray=((Vec_CUSP*)v->spptr)->GPUarray;
    s = (Vec_Seq*)v->data;
    gpuPtr = varray->data().get() + ptop_scatter->sendLowestIndex;
    cpuPtr = s->array + ptop_scatter->sendLowestIndex;

    /* Note : this code copies the smallest contiguous chunk of data
       containing ALL of the indices */
    err = hipMemcpy(cpuPtr,gpuPtr,ptop_scatter->ns*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUSP(err);

    ierr = VecCUSPRestoreArrayRead(v,&varray);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(VEC_CUSPCopyFromGPUSome,v,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCopy_SeqCUSP_Private"
static PetscErrorCode VecCopy_SeqCUSP_Private(Vec xin,Vec yin)
{
  PetscScalar       *ya;
  const PetscScalar *xa;
  PetscErrorCode    ierr;

  PetscFunctionBegin;
  ierr = VecCUSPAllocateCheckHost(xin);
  ierr = VecCUSPAllocateCheckHost(yin);
  if (xin != yin) {
    ierr = VecGetArrayRead(xin,&xa);CHKERRQ(ierr);
    ierr = VecGetArray(yin,&ya);CHKERRQ(ierr);
    ierr = PetscMemcpy(ya,xa,xin->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
    ierr = VecRestoreArrayRead(xin,&xa);CHKERRQ(ierr);
    ierr = VecRestoreArray(yin,&ya);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecSetRandom_SeqCUSP_Private"
static PetscErrorCode VecSetRandom_SeqCUSP_Private(Vec xin,PetscRandom r)
{
  PetscErrorCode ierr;
  PetscInt       n = xin->map->n,i;
  PetscScalar    *xx;

  PetscFunctionBegin;
  ierr = VecGetArray(xin,&xx);CHKERRQ(ierr);
  for (i=0; i<n; i++) {ierr = PetscRandomGetValue(r,&xx[i]);CHKERRQ(ierr);}
  ierr = VecRestoreArray(xin,&xx);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecDestroy_SeqCUSP_Private"
static PetscErrorCode VecDestroy_SeqCUSP_Private(Vec v)
{
  Vec_Seq        *vs = (Vec_Seq*)v->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscObjectSAWsViewOff(v);CHKERRQ(ierr);
#if defined(PETSC_USE_LOG)
  PetscLogObjectState((PetscObject)v,"Length=%D",v->map->n);
#endif
  if (vs) {
    if (vs->array_allocated) ierr = PetscFree(vs->array_allocated);CHKERRQ(ierr);
    ierr = PetscFree(vs);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecResetArray_SeqCUSP_Private"
static PetscErrorCode VecResetArray_SeqCUSP_Private(Vec vin)
{
  Vec_Seq *v = (Vec_Seq*)vin->data;

  PetscFunctionBegin;
  v->array         = v->unplacedarray;
  v->unplacedarray = 0;
  PetscFunctionReturn(0);
}

/* these following 3 public versions are necessary because we use CUSP in the regular PETSc code and these need to be called from plain C code. */
#undef __FUNCT__
#define __FUNCT__ "VecCUSPAllocateCheck_Public"
PetscErrorCode VecCUSPAllocateCheck_Public(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPAllocateCheck(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPCopyToGPU_Public"
PetscErrorCode VecCUSPCopyToGPU_Public(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPCopyToGPU(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "VecCUSPCopyToGPUSome_Public"
/*
    VecCUSPCopyToGPUSome_Public - Copies certain entries down to the GPU from the CPU of a vector

   Input Parameters:
+    v - the vector
-    indices - the requested indices, this should be created with CUSPIndicesCreate()

*/
PetscErrorCode VecCUSPCopyToGPUSome_Public(Vec v, PetscCUSPIndices ci)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPCopyToGPUSome(v,ci);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPCopyFromGPUSome_Public"
/*
  VecCUSPCopyFromGPUSome_Public - Copies certain entries up to the CPU from the GPU of a vector

  Input Parameters:
 +    v - the vector
 -    indices - the requested indices, this should be created with CUSPIndicesCreate()
*/
PetscErrorCode VecCUSPCopyFromGPUSome_Public(Vec v, PetscCUSPIndices ci)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPCopyFromGPUSome(v,ci);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*MC
   VECSEQCUSP - VECSEQCUSP = "seqcusp" - The basic sequential vector, modified to use CUSP

   Options Database Keys:
. -vec_type seqcusp - sets the vector type to VECSEQCUSP during a call to VecSetFromOptions()

  Level: beginner

.seealso: VecCreate(), VecSetType(), VecSetFromOptions(), VecCreateSeqWithArray(), VECMPI, VecType, VecCreateMPI(), VecCreateSeq()
M*/

/* for VecAYPX_SeqCUSP*/
namespace cusp
{
namespace blas
{
namespace detail
{
  template <typename T>
    struct AYPX : public thrust::binary_function<T,T,T>
    {
      T alpha;

      AYPX(T _alpha) : alpha(_alpha) {}

      __host__ __device__
      T operator()(T x, T y)
      {
        return alpha * y + x;
      }
    };
}

 template <typename ForwardIterator1,
           typename ForwardIterator2,
           typename ScalarType>
void aypx(ForwardIterator1 first1,ForwardIterator1 last1,ForwardIterator2 first2,ScalarType alpha)
           {
             thrust::transform(first1,last1,first2,first2,detail::AYPX<ScalarType>(alpha));
           }
 template <typename Array1, typename Array2, typename ScalarType>
   void aypx(const Array1& x, Array2& y, ScalarType alpha)
 {
#if defined(CUSP_VERSION) && CUSP_VERSION >= 500
   cusp::assert_same_dimensions(x,y);
#else
   detail::assert_same_dimensions(x,y);
#endif
   aypx(x.begin(),x.end(),y.begin(),alpha);
 }
}
}

#undef __FUNCT__
#define __FUNCT__ "VecAYPX_SeqCUSP"
PetscErrorCode VecAYPX_SeqCUSP(Vec yin, PetscScalar alpha, Vec xin)
{
  CUSPARRAY      *xarray,*yarray;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
  try {
    if (alpha != 0.0) {
      cusp::blas::aypx(*xarray,*yarray,alpha);
      ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
    } else {
      cusp::blas::copy(*xarray,*yarray);
    }
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecAXPY_SeqCUSP"
PetscErrorCode VecAXPY_SeqCUSP(Vec yin,PetscScalar alpha,Vec xin)
{
  CUSPARRAY      *xarray,*yarray;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (alpha != 0.0) {
    ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    try {
      cusp::blas::axpy(*xarray,*yarray,alpha);
      ierr = WaitForGPU();CHKERRCUSP(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

struct VecCUSPPointwiseDivide
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) / thrust::get<2>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "VecPointwiseDivide_SeqCUSP"
PetscErrorCode VecPointwiseDivide_SeqCUSP(Vec win, Vec xin, Vec yin)
{
  CUSPARRAY      *warray=NULL,*xarray=NULL,*yarray=NULL;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(win,&warray);CHKERRQ(ierr);
  try {
    thrust::for_each(
      thrust::make_zip_iterator(
        thrust::make_tuple(
          warray->begin(),
          xarray->begin(),
          yarray->begin())),
      thrust::make_zip_iterator(
        thrust::make_tuple(
          warray->end(),
          xarray->end(),
          yarray->end())),
      VecCUSPPointwiseDivide());
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(win,&warray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


struct VecCUSPWAXPY
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) + thrust::get<2>(t)*thrust::get<3>(t);
  }
};

struct VecCUSPSum
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) + thrust::get<2>(t);
  }
};

struct VecCUSPDiff
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) - thrust::get<2>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "VecWAXPY_SeqCUSP"
PetscErrorCode VecWAXPY_SeqCUSP(Vec win,PetscScalar alpha,Vec xin, Vec yin)
{
  CUSPARRAY      *xarray=NULL,*yarray=NULL,*warray=NULL;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (alpha == 0.0) {
    ierr = VecCopy_SeqCUSP(yin,win);CHKERRQ(ierr);
  } else {
    ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(yin,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(win,&warray);CHKERRQ(ierr);
    if (alpha == 1.0) {
      try {
        thrust::for_each(
          thrust::make_zip_iterator(
            thrust::make_tuple(
              warray->begin(),
              yarray->begin(),
              xarray->begin())),
          thrust::make_zip_iterator(
            thrust::make_tuple(
              warray->end(),
              yarray->end(),
              xarray->end())),
          VecCUSPSum());
      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
      }
      ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
    } else if (alpha == -1.0) {
      try {
        thrust::for_each(
          thrust::make_zip_iterator(
            thrust::make_tuple(
              warray->begin(),
              yarray->begin(),
              xarray->begin())),
          thrust::make_zip_iterator(
            thrust::make_tuple(
              warray->end(),
              yarray->end(),
              xarray->end())),
          VecCUSPDiff());
      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
      }
      ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
    } else {
      try {
        thrust::for_each(
          thrust::make_zip_iterator(
            thrust::make_tuple(
              warray->begin(),
              yarray->begin(),
              thrust::make_constant_iterator(alpha),
              xarray->begin())),
          thrust::make_zip_iterator(
            thrust::make_tuple(
              warray->end(),
              yarray->end(),
              thrust::make_constant_iterator(alpha),
              xarray->end())),
          VecCUSPWAXPY());
      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
      }
      ierr = PetscLogFlops(2*win->map->n);CHKERRQ(ierr);
    }
    ierr = WaitForGPU();CHKERRCUSP(ierr);
    ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(win,&warray);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/* These functions are for the CUSP implementation of MAXPY with the loop unrolled on the CPU */
struct VecCUSPMAXPY4
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2 + 13*x3 +a4*x4 */
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t)+thrust::get<5>(t)*thrust::get<6>(t)+thrust::get<7>(t)*thrust::get<8>(t);
  }
};


struct VecCUSPMAXPY3
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2 + a3*x3 */
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t)+thrust::get<5>(t)*thrust::get<6>(t);
  }
};

struct VecCUSPMAXPY2
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2*/
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t);
  }
};
#undef __FUNCT__
#define __FUNCT__ "VecMAXPY_SeqCUSP"
PetscErrorCode VecMAXPY_SeqCUSP(Vec xin, PetscInt nv,const PetscScalar *alpha,Vec *y)
{
  PetscErrorCode ierr;
  CUSPARRAY      *xarray,*yy0,*yy1,*yy2,*yy3;
  PetscInt       n = xin->map->n,j,j_rem;
  PetscScalar    alpha0,alpha1,alpha2,alpha3;

  PetscFunctionBegin;
  ierr = PetscLogFlops(nv*2.0*n);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
  switch (j_rem=nv&0x3) {
  case 3:
    alpha0 = alpha[0];
    alpha1 = alpha[1];
    alpha2 = alpha[2];
    alpha += 3;
    ierr   = VecCUSPGetArrayRead(y[0],&yy0);CHKERRQ(ierr);
    ierr   = VecCUSPGetArrayRead(y[1],&yy1);CHKERRQ(ierr);
    ierr   = VecCUSPGetArrayRead(y[2],&yy2);CHKERRQ(ierr);
    try {
      thrust::for_each(
        thrust::make_zip_iterator(
          thrust::make_tuple(
            xarray->begin(),
            thrust::make_constant_iterator(alpha0),
            yy0->begin(),
            thrust::make_constant_iterator(alpha1),
            yy1->begin(),
            thrust::make_constant_iterator(alpha2),
            yy2->begin())),
        thrust::make_zip_iterator(
          thrust::make_tuple(
            xarray->end(),
            thrust::make_constant_iterator(alpha0),
            yy0->end(),
            thrust::make_constant_iterator(alpha1),
            yy1->end(),
            thrust::make_constant_iterator(alpha2),
            yy2->end())),
        VecCUSPMAXPY3());
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = VecCUSPRestoreArrayRead(y[0],&yy0);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(y[1],&yy1);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(y[2],&yy2);CHKERRQ(ierr);
    y   += 3;
    break;
  case 2:
    alpha0 = alpha[0];
    alpha1 = alpha[1];
    alpha +=2;
    ierr   = VecCUSPGetArrayRead(y[0],&yy0);CHKERRQ(ierr);
    ierr   = VecCUSPGetArrayRead(y[1],&yy1);CHKERRQ(ierr);
    try {
      thrust::for_each(
        thrust::make_zip_iterator(
          thrust::make_tuple(
            xarray->begin(),
            thrust::make_constant_iterator(alpha0),
            yy0->begin(),
            thrust::make_constant_iterator(alpha1),
            yy1->begin())),
        thrust::make_zip_iterator(
          thrust::make_tuple(
            xarray->end(),
            thrust::make_constant_iterator(alpha0),
            yy0->end(),
            thrust::make_constant_iterator(alpha1),
            yy1->end())),
        VecCUSPMAXPY2());
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    y +=2;
    break;
  case 1:
    alpha0 = *alpha++;
    ierr   = VecAXPY_SeqCUSP(xin,alpha0,y[0]);
    y     +=1;
    break;
  }
  for (j=j_rem; j<nv; j+=4) {
    alpha0 = alpha[0];
    alpha1 = alpha[1];
    alpha2 = alpha[2];
    alpha3 = alpha[3];
    alpha += 4;
    ierr   = VecCUSPGetArrayRead(y[0],&yy0);CHKERRQ(ierr);
    ierr   = VecCUSPGetArrayRead(y[1],&yy1);CHKERRQ(ierr);
    ierr   = VecCUSPGetArrayRead(y[2],&yy2);CHKERRQ(ierr);
    ierr   = VecCUSPGetArrayRead(y[3],&yy3);CHKERRQ(ierr);
    try {
      thrust::for_each(
        thrust::make_zip_iterator(
          thrust::make_tuple(
            xarray->begin(),
            thrust::make_constant_iterator(alpha0),
            yy0->begin(),
            thrust::make_constant_iterator(alpha1),
            yy1->begin(),
            thrust::make_constant_iterator(alpha2),
            yy2->begin(),
            thrust::make_constant_iterator(alpha3),
            yy3->begin())),
        thrust::make_zip_iterator(
          thrust::make_tuple(
            xarray->end(),
            thrust::make_constant_iterator(alpha0),
            yy0->end(),
            thrust::make_constant_iterator(alpha1),
            yy1->end(),
            thrust::make_constant_iterator(alpha2),
            yy2->end(),
            thrust::make_constant_iterator(alpha3),
            yy3->end())),
        VecCUSPMAXPY4());
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = VecCUSPRestoreArrayRead(y[0],&yy0);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(y[1],&yy1);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(y[2],&yy2);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(y[3],&yy3);CHKERRQ(ierr);
    y   += 4;
  }
  ierr = VecCUSPRestoreArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDot_SeqCUSP"
PetscErrorCode VecDot_SeqCUSP(Vec xin,Vec yin,PetscScalar *z)
{
  CUSPARRAY      *xarray,*yarray;
  PetscErrorCode ierr;
  //  PetscScalar    *xptr,*yptr,*zgpu;
  //PetscReal tmp;

  PetscFunctionBegin;
  //VecNorm_SeqCUSP(xin, NORM_2, &tmp);
  //VecNorm_SeqCUSP(yin, NORM_2, &tmp);
  ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  try {
#if defined(PETSC_USE_COMPLEX)
    *z = cusp::blas::dotc(*yarray,*xarray);
#else
    *z = cusp::blas::dot(*yarray,*xarray);
#endif
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  if (xin->map->n >0) {
    ierr = PetscLogFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

//
// CUDA kernels for MDot to follow
//

// set work group size to be a power of 2 (128 is usually a good compromise between portability and speed)
#define MDOT_WORKGROUP_SIZE 128
#define MDOT_WORKGROUP_NUM  128

// M = 2:
__global__ void VecMDot_SeqCUSP_kernel2(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[2*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = min((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
  }
  tmp_buffer[threadIdx.x]                       = group_sum0;
  tmp_buffer[threadIdx.x + MDOT_WORKGROUP_SIZE] = group_sum1;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                      ] += tmp_buffer[threadIdx.x+stride                      ];
      tmp_buffer[threadIdx.x + MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x]             = tmp_buffer[0];
    group_results[blockIdx.x + gridDim.x] = tmp_buffer[MDOT_WORKGROUP_SIZE];
  }
}

// M = 3:
__global__ void VecMDot_SeqCUSP_kernel3(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[3*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = min((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
  }
}

// M = 4:
__global__ void VecMDot_SeqCUSP_kernel4(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,const PetscScalar *y3,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[4*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = min((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  PetscScalar group_sum3 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
    group_sum3 += entry_x * y3[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;
  tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] = group_sum3;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 3 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 3 * gridDim.x] = tmp_buffer[3 * MDOT_WORKGROUP_SIZE];
  }
}

// M = 8:
__global__ void VecMDot_SeqCUSP_kernel8(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,const PetscScalar *y3,
                                          const PetscScalar *y4,const PetscScalar *y5,const PetscScalar *y6,const PetscScalar *y7,
                                          PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[8*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = min((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  PetscScalar group_sum3 = 0;
  PetscScalar group_sum4 = 0;
  PetscScalar group_sum5 = 0;
  PetscScalar group_sum6 = 0;
  PetscScalar group_sum7 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
    group_sum3 += entry_x * y3[i];
    group_sum4 += entry_x * y4[i];
    group_sum5 += entry_x * y5[i];
    group_sum6 += entry_x * y6[i];
    group_sum7 += entry_x * y7[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;
  tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] = group_sum3;
  tmp_buffer[threadIdx.x + 4 * MDOT_WORKGROUP_SIZE] = group_sum4;
  tmp_buffer[threadIdx.x + 5 * MDOT_WORKGROUP_SIZE] = group_sum5;
  tmp_buffer[threadIdx.x + 6 * MDOT_WORKGROUP_SIZE] = group_sum6;
  tmp_buffer[threadIdx.x + 7 * MDOT_WORKGROUP_SIZE] = group_sum7;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 3 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 4 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 4 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 5 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 5 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 6 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 6 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 7 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 7 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 3 * gridDim.x] = tmp_buffer[3 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 4 * gridDim.x] = tmp_buffer[4 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 5 * gridDim.x] = tmp_buffer[5 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 6 * gridDim.x] = tmp_buffer[6 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 7 * gridDim.x] = tmp_buffer[7 * MDOT_WORKGROUP_SIZE];
  }
}


#undef __FUNCT__
#define __FUNCT__ "VecMDot_SeqCUSP"
PetscErrorCode VecMDot_SeqCUSP(Vec xin,PetscInt nv,const Vec yin[],PetscScalar *z)
{
  PetscErrorCode ierr;
  PetscInt       i,j,n = xin->map->n,current_y_index = 0;
  CUSPARRAY      *xarray,*y0array,*y1array,*y2array,*y3array,*y4array,*y5array,*y6array,*y7array;
  PetscScalar    *group_results_gpu,*xptr,*y0ptr,*y1ptr,*y2ptr,*y3ptr,*y4ptr,*y5ptr,*y6ptr,*y7ptr;
  PetscScalar    group_results_cpu[MDOT_WORKGROUP_NUM * 8]; // we process at most eight vectors in one kernel
  hipError_t    cuda_ierr;

  PetscFunctionBegin;
  if (nv <= 0) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"Number of vectors provided to VecMDot_SeqCUSP not positive.");
  /* Handle the case of local size zero first */
  if (!xin->map->n) {
    for (i=0; i<nv; ++i) z[i] = 0;
    PetscFunctionReturn(0);
  }

  // allocate scratchpad memory for the results of individual work groups:
  cuda_ierr = hipMalloc((void**)&group_results_gpu, sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 8);
  if (cuda_ierr != hipSuccess) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Could not allocate CUDA work memory. Error code: %d", (int)cuda_ierr);

  ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  xptr = thrust::raw_pointer_cast(xarray->data());

  while (current_y_index < nv)
  {
    switch (nv - current_y_index) {

    case 7:
    case 6:
    case 5:
    case 4:
      ierr = VecCUSPGetArrayRead(yin[current_y_index  ],&y0array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+1],&y1array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+2],&y2array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+3],&y3array);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
      z[current_y_index]   = cusp::blas::dot(*y0array,*xarray);
      z[current_y_index+1] = cusp::blas::dot(*y1array,*xarray);
      z[current_y_index+2] = cusp::blas::dot(*y2array,*xarray);
      z[current_y_index+3] = cusp::blas::dot(*y3array,*xarray);
#else
      // extract raw device pointers:
      y0ptr = thrust::raw_pointer_cast(y0array->data());
      y1ptr = thrust::raw_pointer_cast(y1array->data());
      y2ptr = thrust::raw_pointer_cast(y2array->data());
      y3ptr = thrust::raw_pointer_cast(y3array->data());

      // run kernel:
      VecMDot_SeqCUSP_kernel4<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,y3ptr,n,group_results_gpu);

      // copy results back to
      cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 4,hipMemcpyDeviceToHost);
      if (cuda_ierr != hipSuccess) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Could not copy CUDA buffer to host. Error code: %d", (int)cuda_ierr);

      // sum group results into z:
      for (j=0; j<4; ++j) {
        z[current_y_index + j] = 0;
        for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
      }
#endif
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index  ],&y0array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+1],&y1array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+2],&y2array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+3],&y3array);CHKERRQ(ierr);
      current_y_index += 4;
      break;

    case 3:
      ierr = VecCUSPGetArrayRead(yin[current_y_index  ],&y0array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+1],&y1array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+2],&y2array);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
      z[current_y_index]   = cusp::blas::dot(*y0array,*xarray);
      z[current_y_index+1] = cusp::blas::dot(*y1array,*xarray);
      z[current_y_index+2] = cusp::blas::dot(*y2array,*xarray);
#else
      // extract raw device pointers:
      y0ptr = thrust::raw_pointer_cast(y0array->data());
      y1ptr = thrust::raw_pointer_cast(y1array->data());
      y2ptr = thrust::raw_pointer_cast(y2array->data());

      // run kernel:
      VecMDot_SeqCUSP_kernel3<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,n,group_results_gpu);

      // copy results back to
      cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 3,hipMemcpyDeviceToHost);
      if (cuda_ierr != hipSuccess) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Could not copy CUDA buffer to host. Error code: %d", (int)cuda_ierr);

      // sum group results into z:
      for (j=0; j<3; ++j) {
        z[current_y_index + j] = 0;
        for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
      }
#endif

      ierr = VecCUSPRestoreArrayRead(yin[current_y_index  ],&y0array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+1],&y1array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+2],&y2array);CHKERRQ(ierr);
      current_y_index += 3;
      break;

    case 2:
      ierr = VecCUSPGetArrayRead(yin[current_y_index],&y0array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+1],&y1array);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
      z[current_y_index]   = cusp::blas::dot(*y0array,*xarray);
      z[current_y_index+1] = cusp::blas::dot(*y1array,*xarray);
#else
      // extract raw device pointers:
      y0ptr = thrust::raw_pointer_cast(y0array->data());
      y1ptr = thrust::raw_pointer_cast(y1array->data());

      // run kernel:
      VecMDot_SeqCUSP_kernel2<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,n,group_results_gpu);

      // copy results back to 
      cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 2,hipMemcpyDeviceToHost);
      if (cuda_ierr != hipSuccess) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Could not copy CUDA buffer to host. Error code: %d", (int)cuda_ierr);

      // sum group results into z:
      for (j=0; j<2; ++j) {
        z[current_y_index + j] = 0;
        for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
      }
#endif
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index],&y0array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+1],&y1array);CHKERRQ(ierr);
      current_y_index += 2;
      break;

    case 1:
      ierr = VecCUSPGetArrayRead(yin[current_y_index],&y0array);CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
      z[current_y_index] = cusp::blas::dotc(*y0array, *xarray);
#else
      z[current_y_index] = cusp::blas::dot(*xarray, *y0array);
#endif
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index],&y0array);CHKERRQ(ierr);
      current_y_index += 1;
      break;

    default: // 8 or more vectors left
      ierr = VecCUSPGetArrayRead(yin[current_y_index  ],&y0array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+1],&y1array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+2],&y2array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+3],&y3array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+4],&y4array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+5],&y5array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+6],&y6array);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayRead(yin[current_y_index+7],&y7array);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
      z[current_y_index]   = cusp::blas::dot(*y0array,*xarray);
      z[current_y_index+1] = cusp::blas::dot(*y1array,*xarray);
      z[current_y_index+2] = cusp::blas::dot(*y2array,*xarray);
      z[current_y_index+3] = cusp::blas::dot(*y3array,*xarray);
      z[current_y_index+4] = cusp::blas::dot(*y4array,*xarray);
      z[current_y_index+5] = cusp::blas::dot(*y5array,*xarray);
      z[current_y_index+6] = cusp::blas::dot(*y6array,*xarray);
      z[current_y_index+7] = cusp::blas::dot(*y7array,*xarray);
#else
      // extract raw device pointers:
      y0ptr = thrust::raw_pointer_cast(y0array->data());
      y1ptr = thrust::raw_pointer_cast(y1array->data());
      y2ptr = thrust::raw_pointer_cast(y2array->data());
      y3ptr = thrust::raw_pointer_cast(y3array->data());
      y4ptr = thrust::raw_pointer_cast(y4array->data());
      y5ptr = thrust::raw_pointer_cast(y5array->data());
      y6ptr = thrust::raw_pointer_cast(y6array->data());
      y7ptr = thrust::raw_pointer_cast(y7array->data());

      // run kernel:
      VecMDot_SeqCUSP_kernel8<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,y3ptr,y4ptr,y5ptr,y6ptr,y7ptr,n,group_results_gpu);

      // copy results back to
      cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 8,hipMemcpyDeviceToHost);
      if (cuda_ierr != hipSuccess) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Could not copy CUDA buffer to host. Error code: %d", (int)cuda_ierr);

      // sum group results into z:
      for (j=0; j<8; ++j) {
        z[current_y_index + j] = 0;
        for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
      }
#endif
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index  ],&y0array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+1],&y1array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+2],&y2array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+3],&y3array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+4],&y4array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+5],&y5array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+6],&y6array);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayRead(yin[current_y_index+7],&y7array);CHKERRQ(ierr);
      current_y_index += 8;
      break;
    }
  }
  ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);

  cuda_ierr = hipFree(group_results_gpu);
  if (cuda_ierr != hipSuccess) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Could not copy CUDA buffer to host: %d", (int)cuda_ierr);
  ierr = PetscLogFlops(PetscMax(nv*(2.0*n-1),0.0));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef MDOT_WORKGROUP_SIZE
#undef MDOT_WORKGROUP_NUM



#undef __FUNCT__
#define __FUNCT__ "VecSet_SeqCUSP"
PetscErrorCode VecSet_SeqCUSP(Vec xin,PetscScalar alpha)
{
  CUSPARRAY      *xarray=NULL;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* if there's a faster way to do the case alpha=0.0 on the GPU we should do that*/
  ierr = VecCUSPGetArrayWrite(xin,&xarray);CHKERRQ(ierr);
  try {
    cusp::blas::fill(*xarray,alpha);
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = VecCUSPRestoreArrayWrite(xin,&xarray);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecScale_SeqCUSP"
PetscErrorCode VecScale_SeqCUSP(Vec xin, PetscScalar alpha)
{
  CUSPARRAY      *xarray;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (alpha == 0.0) {
    ierr = VecSet_SeqCUSP(xin,alpha);CHKERRQ(ierr);
  } else if (alpha != 1.0) {
    ierr = VecCUSPGetArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
    try {
      cusp::blas::scal(*xarray,alpha);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = VecCUSPRestoreArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(xin->map->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecTDot_SeqCUSP"
PetscErrorCode VecTDot_SeqCUSP(Vec xin,Vec yin,PetscScalar *z)
{
  CUSPARRAY      *xarray,*yarray;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  //#if defined(PETSC_USE_COMPLEX)
  /*Not working for complex*/
  //#else
  ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  try {
    *z = cusp::blas::dot(*xarray,*yarray);
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  //#endif
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  if (xin->map->n > 0) {
    ierr = PetscLogFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  ierr = VecCUSPRestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
#undef __FUNCT__
#define __FUNCT__ "VecCopy_SeqCUSP"
PetscErrorCode VecCopy_SeqCUSP(Vec xin,Vec yin)
{
  CUSPARRAY      *xarray,*yarray;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (xin != yin) {
    if (xin->valid_GPU_array == PETSC_CUSP_GPU) {
      ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
      ierr = VecCUSPGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
      try {
        cusp::blas::copy(*xarray,*yarray);
      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
      }
      ierr = WaitForGPU();CHKERRCUSP(ierr);
      ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
      ierr = VecCUSPRestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);

    } else if (xin->valid_GPU_array == PETSC_CUSP_CPU) {
      /* copy in CPU if we are on the CPU*/
      ierr = VecCopy_SeqCUSP_Private(xin,yin);CHKERRQ(ierr);
    } else if (xin->valid_GPU_array == PETSC_CUSP_BOTH) {
      /* if xin is valid in both places, see where yin is and copy there (because it's probably where we'll want to next use it) */
      if (yin->valid_GPU_array == PETSC_CUSP_CPU) {
        /* copy in CPU */
        ierr = VecCopy_SeqCUSP_Private(xin,yin);CHKERRQ(ierr);

      } else if (yin->valid_GPU_array == PETSC_CUSP_GPU) {
        /* copy in GPU */
        ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUSPGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
        try {
          cusp::blas::copy(*xarray,*yarray);
          ierr = WaitForGPU();CHKERRCUSP(ierr);
        } catch(char *ex) {
          SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
        }
        ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUSPRestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);
      } else if (yin->valid_GPU_array == PETSC_CUSP_BOTH) {
        /* xin and yin are both valid in both places (or yin was unallocated before the earlier call to allocatecheck
           default to copy in GPU (this is an arbitrary choice) */
        ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUSPGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
        try {
          cusp::blas::copy(*xarray,*yarray);
          ierr = WaitForGPU();CHKERRCUSP(ierr);
        } catch(char *ex) {
          SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
        }
        ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUSPRestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);
      } else {
        ierr = VecCopy_SeqCUSP_Private(xin,yin);CHKERRQ(ierr);
      }
    }
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecSwap_SeqCUSP"
PetscErrorCode VecSwap_SeqCUSP(Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscBLASInt   one = 1,bn;
  CUSPARRAY      *xarray,*yarray;

  PetscFunctionBegin;
  ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
  if (xin != yin) {
    ierr = VecCUSPGetArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
#if defined(PETSC_USE_REAL_SINGLE)
    hipblasCswap(bn,(hipFloatComplex*)VecCUSPCastToRawPtr(*xarray),one,(hipFloatComplex*)VecCUSPCastToRawPtr(*yarray),one);
#else
    hipblasZswap(bn,(hipDoubleComplex*)VecCUSPCastToRawPtr(*xarray),one,(hipDoubleComplex*)VecCUSPCastToRawPtr(*yarray),one);
#endif
#else
#if defined(PETSC_USE_REAL_SINGLE)
    hipblasSswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#else
    hipblasDswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#endif
#endif
    ierr = cublasGetError();CHKERRCUSP(ierr);
    ierr = WaitForGPU();CHKERRCUSP(ierr);
    ierr = VecCUSPRestoreArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

struct VecCUSPAX
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t)*thrust::get<2>(t);
  }
};
#undef __FUNCT__
#define __FUNCT__ "VecAXPBY_SeqCUSP"
PetscErrorCode VecAXPBY_SeqCUSP(Vec yin,PetscScalar alpha,PetscScalar beta,Vec xin)
{
  PetscErrorCode ierr;
  PetscScalar    a = alpha,b = beta;
  CUSPARRAY      *xarray,*yarray;

  PetscFunctionBegin;
  if (a == 0.0) {
    ierr = VecScale_SeqCUSP(yin,beta);CHKERRQ(ierr);
  } else if (b == 1.0) {
    ierr = VecAXPY_SeqCUSP(yin,alpha,xin);CHKERRQ(ierr);
  } else if (a == 1.0) {
    ierr = VecAYPX_SeqCUSP(yin,beta,xin);CHKERRQ(ierr);
  } else if (b == 0.0) {
    ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    try {
      thrust::for_each(
        thrust::make_zip_iterator(
          thrust::make_tuple(
            yarray->begin(),
            thrust::make_constant_iterator(a),
            xarray->begin())),
        thrust::make_zip_iterator(
          thrust::make_tuple(
            yarray->end(),
            thrust::make_constant_iterator(a),
            xarray->end())),
        VecCUSPAX());
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = PetscLogFlops(xin->map->n);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUSP(ierr);
    ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
  } else {
    ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    try {
      cusp::blas::axpby(*xarray,*yarray,*yarray,a,b);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUSP(ierr);
    ierr = PetscLogFlops(3.0*xin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/* structs below are for special cases of VecAXPBYPCZ_SeqCUSP */
struct VecCUSPXPBYPCZ
{
  /* z = x + b*y + c*z */
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t)*thrust::get<0>(t)+thrust::get<2>(t)+thrust::get<4>(t)*thrust::get<3>(t);
  }
};
struct VecCUSPAXPBYPZ
{
  /* z = ax + b*y + z */
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) += thrust::get<2>(t)*thrust::get<1>(t)+thrust::get<4>(t)*thrust::get<3>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "VecAXPBYPCZ_SeqCUSP"
PetscErrorCode VecAXPBYPCZ_SeqCUSP(Vec zin,PetscScalar alpha,PetscScalar beta,PetscScalar gamma,Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscInt       n = zin->map->n;
  CUSPARRAY      *xarray,*yarray,*zarray;

  PetscFunctionBegin;
  ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayReadWrite(zin,&zarray);CHKERRQ(ierr);
  if (alpha == 1.0) {
    try {
      thrust::for_each(
        thrust::make_zip_iterator(
          thrust::make_tuple(
            zarray->begin(),
            thrust::make_constant_iterator(gamma),
            xarray->begin(),
            yarray->begin(),
            thrust::make_constant_iterator(beta))),
        thrust::make_zip_iterator(
          thrust::make_tuple(
            zarray->end(),
            thrust::make_constant_iterator(gamma),
            xarray->end(),
            yarray->end(),
            thrust::make_constant_iterator(beta))),
        VecCUSPXPBYPCZ());
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  } else if (gamma == 1.0) {
    try {
      thrust::for_each(
        thrust::make_zip_iterator(
          thrust::make_tuple(
            zarray->begin(),
            xarray->begin(),
            thrust::make_constant_iterator(alpha),
            yarray->begin(),
            thrust::make_constant_iterator(beta))),
        thrust::make_zip_iterator(
          thrust::make_tuple(
            zarray->end(),
            xarray->end(),
            thrust::make_constant_iterator(alpha),
            yarray->end(),
            thrust::make_constant_iterator(beta))),
        VecCUSPAXPBYPZ());
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  } else {
    try {
      cusp::blas::axpbypcz(*xarray,*yarray,*zarray,*zarray,alpha,beta,gamma);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = VecCUSPRestoreArrayReadWrite(zin,&zarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
    ierr = PetscLogFlops(5.0*n);CHKERRQ(ierr);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecPointwiseMult_SeqCUSP"
PetscErrorCode VecPointwiseMult_SeqCUSP(Vec win,Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscInt       n = win->map->n;
  CUSPARRAY      *xarray,*yarray,*warray;

  PetscFunctionBegin;
  ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayReadWrite(win,&warray);CHKERRQ(ierr);
  try {
    cusp::blas::xmy(*xarray,*yarray,*warray);
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayReadWrite(win,&warray);CHKERRQ(ierr);
  ierr = PetscLogFlops(n);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  PetscFunctionReturn(0);
}


/* should do infinity norm in cusp */

#undef __FUNCT__
#define __FUNCT__ "VecNorm_SeqCUSP"
PetscErrorCode VecNorm_SeqCUSP(Vec xin,NormType type,PetscReal *z)
{
  const PetscScalar *xx;
  PetscErrorCode    ierr;
  PetscInt          n = xin->map->n;
  PetscBLASInt      one = 1, bn;
  CUSPARRAY         *xarray;

  PetscFunctionBegin;
  ierr = PetscBLASIntCast(n,&bn);CHKERRQ(ierr);
  if (type == NORM_2 || type == NORM_FROBENIUS) {
    ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    try {
      *z = cusp::blas::nrm2(*xarray);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = WaitForGPU();CHKERRCUSP(ierr);
    ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = PetscLogFlops(PetscMax(2.0*n-1,0.0));CHKERRQ(ierr);
  } else if (type == NORM_INFINITY) {
    PetscInt  i;
    PetscReal max = 0.0,tmp;

    ierr = VecGetArrayRead(xin,&xx);CHKERRQ(ierr);
    for (i=0; i<n; i++) {
      if ((tmp = PetscAbsScalar(*xx)) > max) max = tmp;
      /* check special case of tmp == NaN */
      if (tmp != tmp) {max = tmp; break;}
      xx++;
    }
    ierr = VecRestoreArrayRead(xin,&xx);CHKERRQ(ierr);
    *z   = max;
  } else if (type == NORM_1) {
    ierr = VecCUSPGetArrayRead(xin,&xarray);CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
#if defined(PETSC_USE_REAL_SINGLE)
    *z = hipblasScasum(bn,(hipFloatComplex*)VecCUSPCastToRawPtr(*xarray),one);
#else
    *z = hipblasDzasum(bn,(hipDoubleComplex*)VecCUSPCastToRawPtr(*xarray),one);
#endif
#else
#if defined(PETSC_USE_REAL_SINGLE)
    *z = hipblasSasum(bn,VecCUSPCastToRawPtr(*xarray),one);
#else
    *z = hipblasDasum(bn,VecCUSPCastToRawPtr(*xarray),one);
#endif
#endif
    ierr = cublasGetError();CHKERRCUSP(ierr);
    ierr = VecCUSPRestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUSP(ierr);
    ierr = PetscLogFlops(PetscMax(n-1.0,0.0));CHKERRQ(ierr);
  } else if (type == NORM_1_AND_2) {
    ierr = VecNorm_SeqCUSP(xin,NORM_1,z);CHKERRQ(ierr);
    ierr = VecNorm_SeqCUSP(xin,NORM_2,z+1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}


/*the following few functions should be modified to actually work with the GPU so they don't force unneccesary allocation of CPU memory */

#undef __FUNCT__
#define __FUNCT__ "VecSetRandom_SeqCUSP"
PetscErrorCode VecSetRandom_SeqCUSP(Vec xin,PetscRandom r)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecSetRandom_SeqCUSP_Private(xin,r);CHKERRQ(ierr);
  xin->valid_GPU_array = PETSC_CUSP_CPU;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecResetArray_SeqCUSP"
PetscErrorCode VecResetArray_SeqCUSP(Vec vin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPCopyFromGPU(vin);CHKERRQ(ierr);
  ierr = VecResetArray_SeqCUSP_Private(vin);CHKERRQ(ierr);
  vin->valid_GPU_array = PETSC_CUSP_CPU;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecPlaceArray_SeqCUSP"
PetscErrorCode VecPlaceArray_SeqCUSP(Vec vin,const PetscScalar *a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPCopyFromGPU(vin);CHKERRQ(ierr);
  ierr = VecPlaceArray_Seq(vin,a);CHKERRQ(ierr);
  vin->valid_GPU_array = PETSC_CUSP_CPU;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecReplaceArray_SeqCUSP"
PetscErrorCode VecReplaceArray_SeqCUSP(Vec vin,const PetscScalar *a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUSPCopyFromGPU(vin);CHKERRQ(ierr);
  ierr = VecReplaceArray_Seq(vin,a);CHKERRQ(ierr);
  vin->valid_GPU_array = PETSC_CUSP_CPU;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCreateSeqCUSP"
/*@
   VecCreateSeqCUSP - Creates a standard, sequential array-style vector.

   Collective on MPI_Comm

   Input Parameter:
+  comm - the communicator, should be PETSC_COMM_SELF
-  n - the vector length

   Output Parameter:
.  V - the vector

   Notes:
   Use VecDuplicate() or VecDuplicateVecs() to form additional vectors of the
   same type as an existing vector.

   Level: intermediate

   Concepts: vectors^creating sequential

.seealso: VecCreateMPI(), VecCreate(), VecDuplicate(), VecDuplicateVecs(), VecCreateGhost()
@*/
PetscErrorCode  VecCreateSeqCUSP(MPI_Comm comm,PetscInt n,Vec *v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCreate(comm,v);CHKERRQ(ierr);
  ierr = VecSetSizes(*v,n,n);CHKERRQ(ierr);
  ierr = VecSetType(*v,VECSEQCUSP);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*The following template functions are for VecDotNorm2_SeqCUSP.  Note that there is no complex support as currently written*/
template <typename T>
struct cuspdotnormcalculate : thrust::unary_function<T,T>
{
  __host__ __device__
  T operator()(T x)
  {
#if defined(PETSC_USE_COMPLEX)
    //return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x), thrust::get<1>(x)*thrust::get<1>(x));
#else
    return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x), thrust::get<1>(x)*thrust::get<1>(x));
#endif
  }
};

template <typename T>
struct cuspdotnormreduce : thrust::binary_function<T,T,T>
{
  __host__ __device__
  T operator()(T x,T y)
  {
    return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y), thrust::get<1>(x)+thrust::get<1>(y));
  }
};

#undef __FUNCT__
#define __FUNCT__ "VecDotNorm2_SeqCUSP"
PetscErrorCode VecDotNorm2_SeqCUSP(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm)
{
  PetscErrorCode                         ierr;
  PetscScalar                            zero = 0.0;
  PetscReal                              n=s->map->n;
  thrust::tuple<PetscScalar,PetscScalar> result;
  CUSPARRAY                              *sarray,*tarray;

  PetscFunctionBegin;
  /*ierr = VecCUSPCopyToGPU(s);CHKERRQ(ierr);
   ierr = VecCUSPCopyToGPU(t);CHKERRQ(ierr);*/
  ierr = VecCUSPGetArrayRead(s,&sarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(t,&tarray);CHKERRQ(ierr);
  try {
#if defined(PETSC_USE_COMPLEX)
    ierr = VecDot_SeqCUSP(s,t,dp);CHKERRQ(ierr);
    ierr = VecDot_SeqCUSP(t,t,nm);CHKERRQ(ierr);
    //printf("VecDotNorm2_SeqCUSP=%1.5g,%1.5g\n",PetscRealPart(*dp),PetscImaginaryPart(*dp));
    //printf("VecDotNorm2_SeqCUSP=%1.5g,%1.5g\n",PetscRealPart(*nm),PetscImaginaryPart(*nm));
#else
    result = thrust::transform_reduce(
              thrust::make_zip_iterator(
                thrust::make_tuple(
                  sarray->begin(),
                  tarray->begin())),
              thrust::make_zip_iterator(
                thrust::make_tuple(
                  sarray->end(),
                  tarray->end())),
              cuspdotnormcalculate<thrust::tuple<PetscScalar,PetscScalar> >(),
              thrust::make_tuple(zero,zero),                                   /*init */
              cuspdotnormreduce<thrust::tuple<PetscScalar, PetscScalar> >());  /* binary function */
    *dp = thrust::get<0>(result);
    *nm = thrust::get<1>(result);
#endif
  } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(s,&sarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayRead(t,&tarray);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecDuplicate_SeqCUSP"
PetscErrorCode VecDuplicate_SeqCUSP(Vec win,Vec *V)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCreateSeqCUSP(PetscObjectComm((PetscObject)win),win->map->n,V);CHKERRQ(ierr);
  ierr = PetscLayoutReference(win->map,&(*V)->map);CHKERRQ(ierr);
  ierr = PetscObjectListDuplicate(((PetscObject)win)->olist,&((PetscObject)(*V))->olist);CHKERRQ(ierr);
  ierr = PetscFunctionListDuplicate(((PetscObject)win)->qlist,&((PetscObject)(*V))->qlist);CHKERRQ(ierr);
  (*V)->stash.ignorenegidx = win->stash.ignorenegidx;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecDestroy_SeqCUSP"
PetscErrorCode VecDestroy_SeqCUSP(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  PetscFunctionBegin;
  try {
    if (v->spptr) {
      delete ((Vec_CUSP*)v->spptr)->GPUarray;
      err = hipStreamDestroy(((Vec_CUSP*)v->spptr)->stream);CHKERRCUSP(err);
      delete (Vec_CUSP*)v->spptr;
    }
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = VecDestroy_SeqCUSP_Private(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#if defined(PETSC_USE_COMPLEX)
struct conjugate 
{
  __host__ __device__
  PetscScalar operator()(PetscScalar x)
  {
    return cusp::conj(x);
  }
};
#endif


#undef __FUNCT__
#define __FUNCT__ "VecConjugate_SeqCUSP"
PetscErrorCode VecConjugate_SeqCUSP(Vec xin)
{
  PetscErrorCode ierr;
  CUSPARRAY      *xarray;

  PetscFunctionBegin;
  ierr = VecCUSPGetArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
  thrust::transform(xarray->begin(), xarray->end(), xarray->begin(), conjugate());
#endif
  ierr = VecCUSPRestoreArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecGetLocalVector_SeqCUSP"
PetscErrorCode VecGetLocalVector_SeqCUSP(Vec v,Vec w)
{
  VecType        t;
  PetscErrorCode ierr;
  hipError_t    err;
  PetscBool      flg;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  PetscValidHeaderSpecific(w,VEC_CLASSID,2);
  ierr = VecGetType(w,&t);CHKERRQ(ierr);
  ierr = PetscStrcmp(t,VECSEQCUSP,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Vector of type %s passed to argument #2. Should be %s.\n",t,VECSEQCUSP);

  if (w->data) {
    if (((Vec_Seq*)w->data)->array_allocated) PetscFree(((Vec_Seq*)w->data)->array_allocated);
    ((Vec_Seq*)w->data)->array = 0;
    ((Vec_Seq*)w->data)->array_allocated = 0;
    ((Vec_Seq*)w->data)->unplacedarray = 0;
  }
  if (w->spptr) {
    if (((Vec_CUSP*)w->spptr)->GPUarray) delete ((Vec_CUSP*)w->spptr)->GPUarray;
    err = hipStreamDestroy(((Vec_CUSP*)w->spptr)->stream);CHKERRCUSP(err);
    delete (Vec_CUSP*)w->spptr;
    w->spptr = 0;
  }

  if (v->petscnative) {
    ierr = PetscFree(w->data);CHKERRQ(ierr);
    w->data = v->data;
    w->valid_GPU_array = v->valid_GPU_array;
    w->spptr = v->spptr;
    ierr = PetscObjectStateIncrease((PetscObject)w);CHKERRQ(ierr);
  } else {
    ierr = VecGetArray(v,&((Vec_Seq*)w->data)->array);CHKERRQ(ierr);
    w->valid_GPU_array = PETSC_CUSP_CPU;
    ierr = VecCUSPAllocateCheck(w);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecRestoreLocalVector_SeqCUSP"
PetscErrorCode VecRestoreLocalVector_SeqCUSP(Vec v,Vec w)
{
  VecType        t;
  PetscErrorCode ierr;
  hipError_t    err;
  PetscBool      flg;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  PetscValidHeaderSpecific(w,VEC_CLASSID,2);
  ierr = VecGetType(w,&t);CHKERRQ(ierr);
  ierr = PetscStrcmp(t,VECSEQCUSP,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Vector of type %s passed to argument #2. Should be %s.\n",t,VECSEQCUSP);

  if (v->petscnative) {
    v->data = w->data;
    v->valid_GPU_array = w->valid_GPU_array;
    v->spptr = w->spptr;
    ierr = VecCUSPCopyFromGPU(v);CHKERRQ(ierr);
    ierr = PetscObjectStateIncrease((PetscObject)v);CHKERRQ(ierr);
    w->data = 0;
    w->valid_GPU_array = PETSC_CUSP_UNALLOCATED;
    w->spptr = 0;
  } else {
    ierr = VecRestoreArray(v,&((Vec_Seq*)w->data)->array);CHKERRQ(ierr);
    if ((Vec_CUSP*)w->spptr) {
      delete ((Vec_CUSP*)w->spptr)->GPUarray;
      err = hipStreamDestroy(((Vec_CUSP*)w->spptr)->stream);CHKERRCUSP(err);
      delete (Vec_CUSP*)w->spptr;
    }
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCreate_SeqCUSP"
PETSC_EXTERN PetscErrorCode VecCreate_SeqCUSP(Vec V)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;

  PetscFunctionBegin;
  ierr = MPI_Comm_size(PetscObjectComm((PetscObject)V),&size);CHKERRQ(ierr);
  if (size > 1) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Cannot create VECSEQCUSP on more than one process");
  ierr = VecCreate_Seq_Private(V,0);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)V,VECSEQCUSP);CHKERRQ(ierr);

  V->ops->dot                    = VecDot_SeqCUSP;
  V->ops->norm                   = VecNorm_SeqCUSP;
  V->ops->tdot                   = VecTDot_SeqCUSP;
  V->ops->scale                  = VecScale_SeqCUSP;
  V->ops->copy                   = VecCopy_SeqCUSP;
  V->ops->set                    = VecSet_SeqCUSP;
  V->ops->swap                   = VecSwap_SeqCUSP;
  V->ops->axpy                   = VecAXPY_SeqCUSP;
  V->ops->axpby                  = VecAXPBY_SeqCUSP;
  V->ops->axpbypcz               = VecAXPBYPCZ_SeqCUSP;
  V->ops->pointwisemult          = VecPointwiseMult_SeqCUSP;
  V->ops->pointwisedivide        = VecPointwiseDivide_SeqCUSP;
  V->ops->setrandom              = VecSetRandom_SeqCUSP;
  V->ops->dot_local              = VecDot_SeqCUSP;
  V->ops->tdot_local             = VecTDot_SeqCUSP;
  V->ops->norm_local             = VecNorm_SeqCUSP;
  V->ops->mdot_local             = VecMDot_SeqCUSP;
  V->ops->maxpy                  = VecMAXPY_SeqCUSP;
  V->ops->mdot                   = VecMDot_SeqCUSP;
  V->ops->aypx                   = VecAYPX_SeqCUSP;
  V->ops->waxpy                  = VecWAXPY_SeqCUSP;
  V->ops->dotnorm2               = VecDotNorm2_SeqCUSP;
  V->ops->placearray             = VecPlaceArray_SeqCUSP;
  V->ops->replacearray           = VecReplaceArray_SeqCUSP;
  V->ops->resetarray             = VecResetArray_SeqCUSP;
  V->ops->destroy                = VecDestroy_SeqCUSP;
  V->ops->duplicate              = VecDuplicate_SeqCUSP;
  V->ops->conjugate              = VecConjugate_SeqCUSP;
  V->ops->getlocalvector         = VecGetLocalVector_SeqCUSP;
  V->ops->restorelocalvector     = VecRestoreLocalVector_SeqCUSP;
  V->ops->getlocalvectorread     = VecGetLocalVector_SeqCUSP;
  V->ops->restorelocalvectorread = VecRestoreLocalVector_SeqCUSP;

  ierr = VecCUSPAllocateCheck(V);CHKERRQ(ierr);
  V->valid_GPU_array      = PETSC_CUSP_GPU;
  ierr = VecSet(V,0.0);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPGetArrayReadWrite"
PETSC_EXTERN PetscErrorCode VecCUSPGetArrayReadWrite(Vec v, CUSPARRAY **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  *a   = 0;
  ierr = VecCUSPCopyToGPU(v);CHKERRQ(ierr);
  *a   = ((Vec_CUSP*)v->spptr)->GPUarray;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPRestoreArrayReadWrite"
PETSC_EXTERN PetscErrorCode VecCUSPRestoreArrayReadWrite(Vec v, CUSPARRAY **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  v->valid_GPU_array = PETSC_CUSP_GPU;

  ierr = PetscObjectStateIncrease((PetscObject)v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPGetArrayRead"
PETSC_EXTERN PetscErrorCode VecCUSPGetArrayRead(Vec v, CUSPARRAY **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  *a   = 0;
  ierr = VecCUSPCopyToGPU(v);CHKERRQ(ierr);
  *a   = ((Vec_CUSP*)v->spptr)->GPUarray;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPRestoreArrayRead"
PETSC_EXTERN PetscErrorCode VecCUSPRestoreArrayRead(Vec v, CUSPARRAY **a)
{
  PetscFunctionBegin;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPGetArrayWrite"
PETSC_EXTERN PetscErrorCode VecCUSPGetArrayWrite(Vec v, CUSPARRAY **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  *a   = 0;
  ierr = VecCUSPAllocateCheck(v);CHKERRQ(ierr);
  *a   = ((Vec_CUSP*)v->spptr)->GPUarray;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUSPRestoreArrayWrite"
PETSC_EXTERN PetscErrorCode VecCUSPRestoreArrayWrite(Vec v, CUSPARRAY **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  v->valid_GPU_array = PETSC_CUSP_GPU;

  ierr = PetscObjectStateIncrease((PetscObject)v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCUSPGetCUDAArray"
/*MC
   VecCUSPGetCUDAArray - Provides write access to the CUDA buffer inside a vector.

   Input Parameter:
-  v - the vector

   Output Parameter:
.  a - the CUDA pointer

   Level: intermediate

.seealso: VecCUSPGetArrayRead(), VecCUSPGetArrayWrite()
M*/
PETSC_EXTERN PetscErrorCode VecCUSPGetCUDAArray(Vec v, PetscScalar **a)
{
  PetscErrorCode ierr;
  CUSPARRAY      *cusparray;

  PetscFunctionBegin;
  PetscValidPointer(a,1);
  ierr = VecCUSPAllocateCheck(v);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(v, &cusparray);CHKERRQ(ierr);
  *a   = thrust::raw_pointer_cast(cusparray->data());
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "VecCUSPRestoreCUDAArray"
PETSC_EXTERN PetscErrorCode VecCUSPRestoreCUDAArray(Vec v, PetscScalar **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* Note: cannot call VecCUSPRestoreArrayWrite() here because the CUSP vector is not available. */
  v->valid_GPU_array = PETSC_CUSP_GPU;
  ierr = PetscObjectStateIncrease((PetscObject)v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
