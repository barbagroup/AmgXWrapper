#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 * \file consolidate.cu
 * \brief Definition of member functions related to matrix consolidation.
 * \author Matt Martineau (mmartineau@nvidia.com)
 * \date 2020-07-31
 */

#include <AmgXSolver.hpp>

#include <numeric>

/*
    Changes local row offsets to describe the consolidated row space on
    the root rank.
*/
__global__ void fixConsolidatedRowOffsets(int nLocalRows, int offset, int* rowOffsets)
{
    for(int i = threadIdx.x + blockIdx.x*blockDim.x; i < nLocalRows; i += blockDim.x*gridDim.x)
    {
        rowOffsets[i] += offset;
    }
}

// A set of handles to the device data storing a consolidated CSR matrix
struct ConsolidationHandles
{
    hipIpcMemHandle_t rhsConsHandle;
    hipIpcMemHandle_t solConsHandle;
    hipIpcMemHandle_t rowOffsetsConsHandle;
    hipIpcMemHandle_t colIndicesConsHandle;
    hipIpcMemHandle_t valuesConsHandle;
};

/* \implements AmgXSolver::initializeConsolidation */
PetscErrorCode AmgXSolver::initializeConsolidation(
    const PetscInt nLocalRows,
    const PetscInt nLocalNz,
    const PetscScalar* values)
{
    PetscFunctionBeginUser;

    // Check if multiple ranks are associated with a device
    if (devWorldSize == 1)
    {
        consolidationStatus = ConsolidationStatus::None;
        PetscFunctionReturn(0);
    }

    nRowsInDevWorld.resize(devWorldSize);
    nnzInDevWorld.resize(devWorldSize);
    rowDispls.resize(devWorldSize+1, 0);
    nzDispls.resize(devWorldSize+1, 0);

    // Fetch to all the number of local rows on each rank
    MPI_Request req[2];
    int ierr = MPI_Iallgather(&nLocalRows, 1, MPI_INT, nRowsInDevWorld.data(), 1, MPI_INT, devWorld, &req[0]); CHK;

    // Fetch to all the number of non zeros on each rank
    ierr = MPI_Iallgather(&nLocalNz, 1, MPI_INT, nnzInDevWorld.data(), 1, MPI_INT, devWorld, &req[1]); CHK;
    MPI_Waitall(2, req, MPI_STATUSES_IGNORE);

    // Calculate consolidate number of rows, non-zeros, and calculate row, non-zero displacements
    nConsNz = std::accumulate(nnzInDevWorld.begin(), nnzInDevWorld.end(), 0);
    nConsRows = std::accumulate(nRowsInDevWorld.begin(), nRowsInDevWorld.end(), 0);
    std::partial_sum(nRowsInDevWorld.begin(), nRowsInDevWorld.end(), rowDispls.begin()+1);
    std::partial_sum(nnzInDevWorld.begin(), nnzInDevWorld.end(), nzDispls.begin()+1);

    // Consolidate the CSR matrix data from multiple ranks sharing a single GPU to a
    // root rank, allowing multiple ranks per GPU. This allows overdecomposing the problem
    // when there are more CPU cores than GPUs, without the inefficiences of performing
    // the linear solve on multiple separate domains.
    // If the data is a device pointer then use IPC handles to perform the intra-GPU
    // copies from the allocations of different processes operating the same GPU.
    // Opening the handles as an initialization step means it is not necessary to
    // repeatedly call hipIpcOpenMemHandle, which can be expensive.
    hipPointerAttribute_t att;
    hipError_t err = hipPointerGetAttributes(&att, values);
    if (err != hipErrorInvalidValue && att.type == hipMemoryTypeDevice)
    {
        ConsolidationHandles handles;
        // The data is already on the GPU so consolidate there
        if (gpuProc == 0)
        {
            // We are consolidating data that already exists on the GPU
            CHECK(hipMalloc((void**)&rhsCons, sizeof(PetscScalar) * nConsRows));
            CHECK(hipMalloc((void**)&pCons, sizeof(PetscScalar) * nConsRows));
            CHECK(hipMalloc((void**)&rowOffsetsCons, sizeof(PetscInt) * (nConsRows+1)));
            CHECK(hipMalloc((void**)&colIndicesGlobalCons, sizeof(PetscInt) * nConsNz));
            CHECK(hipMalloc((void**)&valuesCons, sizeof(PetscScalar) * nConsNz));

            CHECK(hipIpcGetMemHandle(&handles.rhsConsHandle, rhsCons));
            CHECK(hipIpcGetMemHandle(&handles.solConsHandle, pCons));
            CHECK(hipIpcGetMemHandle(&handles.rowOffsetsConsHandle, rowOffsetsCons));
            CHECK(hipIpcGetMemHandle(&handles.colIndicesConsHandle, colIndicesGlobalCons));
            CHECK(hipIpcGetMemHandle(&handles.valuesConsHandle, valuesCons));
        }

        MPI_Bcast(&handles, sizeof(ConsolidationHandles), MPI_BYTE, 0, devWorld);

        if(gpuProc == MPI_UNDEFINED)
        {
            CHECK(hipIpcOpenMemHandle((void**)&rhsCons, handles.rhsConsHandle, hipIpcMemLazyEnablePeerAccess));
            CHECK(hipIpcOpenMemHandle((void**)&pCons, handles.solConsHandle, hipIpcMemLazyEnablePeerAccess));
            CHECK(hipIpcOpenMemHandle((void**)&rowOffsetsCons, handles.rowOffsetsConsHandle, hipIpcMemLazyEnablePeerAccess));
            CHECK(hipIpcOpenMemHandle((void**)&colIndicesGlobalCons, handles.colIndicesConsHandle, hipIpcMemLazyEnablePeerAccess));
            CHECK(hipIpcOpenMemHandle((void**)&valuesCons, handles.valuesConsHandle, hipIpcMemLazyEnablePeerAccess));
        }

        consolidationStatus = ConsolidationStatus::Device;
    }
    else
    {
        if (gpuProc == 0)
        {
            // The data is already on the CPU so consolidate there
            rowOffsetsCons = new PetscInt[nConsRows+1];
            colIndicesGlobalCons = new PetscInt[nConsNz];
            valuesCons = new PetscScalar[nConsNz];
            rhsCons = new PetscScalar[nConsRows];
            pCons = new PetscScalar[nConsRows];
        }

        consolidationStatus = ConsolidationStatus::Host;
    }

    PetscFunctionReturn(0);
}

/* \implements AmgXSolver::consolidateMatrix */
PetscErrorCode AmgXSolver::consolidateMatrix(
    const PetscInt nLocalRows,
    const PetscInt nLocalNz,
    const PetscInt* rowOffsets,
    const PetscInt* colIndicesGlobal,
    const PetscScalar* values)
{
    PetscFunctionBeginUser;

    // Consolidation has been previously used, must deallocate the structures
    if (consolidationStatus != ConsolidationStatus::Uninitialized)
    {
        // XXX Would the maintainers be happy to include a warning message here,
        // that makes it clear updateA should be preferentially adopted by developers
        // if the sparsity pattern does not change? This would avoid many costs,
        // including re-consolidation costs.

        finalizeConsolidation();
    }

    // Allocate space for the structures required to consolidate
    initializeConsolidation(nLocalRows, nLocalNz, values);

    switch(consolidationStatus)
    {

    case ConsolidationStatus::None:
    {
        // Consolidation is not required
        PetscFunctionReturn(0);
    }
    case ConsolidationStatus::Uninitialized:
    {
        SETERRQ(MPI_COMM_WORLD, PETSC_ERR_SUP_SYS,
                "Attempting to consolidate before consolidation is initialized.\n");
        break;
    }
    case ConsolidationStatus::Device:
    {
        // Copy the data to the consolidation buffer
        CHECK(hipMemcpy(&rowOffsetsCons[rowDispls[myDevWorldRank]], rowOffsets, sizeof(PetscInt) * nLocalRows, hipMemcpyDefault));
        CHECK(hipMemcpy(&colIndicesGlobalCons[nzDispls[myDevWorldRank]], colIndicesGlobal, sizeof(PetscInt) * nLocalNz, hipMemcpyDefault));
        CHECK(hipMemcpy(&valuesCons[nzDispls[myDevWorldRank]], values, sizeof(PetscScalar) * nLocalNz, hipMemcpyDefault));

        // hipMemcpy does not block the host in the cases above, device to device copies,
        // so sychronize with device to ensure operation is complete. Barrier on all devWorld
        // ranks to ensure full arrays are populated before the root process uses the data.
        CHECK(hipDeviceSynchronize());
        int ierr = MPI_Barrier(devWorld); CHK;

        if (gpuProc == 0)
        {
            // Adjust merged row offsets so that they are correct for the consolidated matrix
            for (int i = 1; i < devWorldSize; ++i)
            {
                int nthreads = 128;
                int nblocks = nRowsInDevWorld[i] / nthreads + 1;
                fixConsolidatedRowOffsets<<<nblocks, nthreads>>>(nRowsInDevWorld[i], nzDispls[i], &rowOffsetsCons[rowDispls[i]]);
            }

            // Manually add the last entry of the rowOffsets list, which is the
            // number of non-zeros in the CSR matrix
            CHECK(hipMemcpy(&rowOffsetsCons[nConsRows], &nConsNz, sizeof(int), hipMemcpyDefault));
        }
        else
        {
            // Close IPC handles and deallocate for consolidation
            CHECK(hipIpcCloseMemHandle(rowOffsetsCons));
            CHECK(hipIpcCloseMemHandle(colIndicesGlobalCons));
        }

        CHECK(hipDeviceSynchronize());
        break;
    }
    case ConsolidationStatus::Host:
    {
        // Gather the matrix data to the root rank for consolidation
        MPI_Request req[3];
        int ierr = MPI_Igatherv(rowOffsets, nLocalRows, MPI_INT, rowOffsetsCons, nRowsInDevWorld.data(), rowDispls.data(), MPI_INT, 0, devWorld, &req[0]); CHK;
        ierr = MPI_Igatherv(colIndicesGlobal, nLocalNz, MPI_INT, colIndicesGlobalCons, nnzInDevWorld.data(), nzDispls.data(), MPI_INT, 0, devWorld, &req[1]); CHK;
        ierr = MPI_Igatherv(values, nLocalNz, MPI_DOUBLE, valuesCons, nnzInDevWorld.data(), nzDispls.data(), MPI_DOUBLE, 0, devWorld, &req[2]); CHK;
        MPI_Waitall(3, req, MPI_STATUSES_IGNORE);

        if (gpuProc == 0)
        {
            // Adjust merged row offsets so that they are correct for the consolidated matrix
            for (int j = 1; j < devWorldSize; ++j)
            {
                for(int i = 0; i < nRowsInDevWorld[j]; ++i)
                {
                    rowOffsetsCons[rowDispls[j] + i] += nzDispls[j];
                }
            }

            // Manually add the last entry of the rowOffsets list, which is the
            // number of non-zeros in the CSR matrix
            rowOffsetsCons[nConsRows] = nConsNz;
        }

        break;
    }
    default:
    {
        SETERRQ(MPI_COMM_WORLD, PETSC_ERR_SUP_SYS,
                "Incorrect consolidation status set.\n");
        break;
    }

    }

    PetscFunctionReturn(0);
}

/* \implements AmgXSolver::reconsolidateValues */
PetscErrorCode AmgXSolver::reconsolidateValues(
    const PetscInt nLocalNz,
    const PetscScalar* values)
{
    PetscFunctionBeginUser;

    switch (consolidationStatus)
    {

    case ConsolidationStatus::None:
    {
        // Consolidation is not required
        PetscFunctionReturn(0);
    }
    case ConsolidationStatus::Uninitialized:
    {
        SETERRQ(MPI_COMM_WORLD, PETSC_ERR_SUP_SYS,
                "Attempting to re-consolidate before consolidation is initialized.\n");
        break;
    }
    case ConsolidationStatus::Device:
    {
        CHECK(hipDeviceSynchronize());
        int ierr = MPI_Barrier(devWorld); CHK;

        // The data is already on the GPU so consolidate there
        CHECK(hipMemcpy(&valuesCons[nzDispls[myDevWorldRank]], values, sizeof(PetscScalar) * nLocalNz, hipMemcpyDefault));

        CHECK(hipDeviceSynchronize());
        ierr = MPI_Barrier(devWorld); CHK;

        break;
    }
    case ConsolidationStatus::Host:
    {
        // Gather the matrix values to the root rank for consolidation
        int ierr = MPI_Gatherv(values, nLocalNz, MPI_DOUBLE, valuesCons, nnzInDevWorld.data(), nzDispls.data(), MPI_DOUBLE, 0, devWorld); CHK;
        break;
    }
    default:
    {
        SETERRQ(MPI_COMM_WORLD, PETSC_ERR_SUP_SYS,
                "Incorrect consolidation status set.\n");
        break;
    }

    }

    PetscFunctionReturn(0);
}

/* \implements AmgXSolver::freeConsStructure */
PetscErrorCode AmgXSolver::freeConsStructure()
{
    PetscFunctionBeginUser;

    // Only the root rank maintains a consolidated structure
    if(gpuProc == MPI_UNDEFINED)
    {
        PetscFunctionReturn(0);
    }

    switch(consolidationStatus)
    {

    case ConsolidationStatus::None:
    {
        // Consolidation is not required
        PetscFunctionReturn(0);
    }
    case ConsolidationStatus::Uninitialized:
    {
        SETERRQ(MPI_COMM_WORLD, PETSC_ERR_SUP_SYS,
                 "Attempting to free consolidation structures before consolidation is initialized.\n");
        break;
    }
    case ConsolidationStatus::Device:
    {
        // Free the device allocated consolidated CSR matrix structure
        CHECK(hipFree(rowOffsetsCons));
        CHECK(hipFree(colIndicesGlobalCons));
        break;
    }
    case ConsolidationStatus::Host:
    {
        // Free the host allocated consolidated CSR matrix structure
        delete[] rowOffsetsCons;
        delete[] colIndicesGlobalCons;
        break;
    }
    default:
    {
        SETERRQ(MPI_COMM_WORLD, PETSC_ERR_SUP_SYS,
                "Incorrect consolidation status set.\n");
        break;
    }

    }

    PetscFunctionReturn(0);
}

/* \implements AmgXSolver::finalizeConsolidation */
PetscErrorCode AmgXSolver::finalizeConsolidation()
{
    PetscFunctionBeginUser;

    switch(consolidationStatus)
    {

    case ConsolidationStatus::None:
    case ConsolidationStatus::Uninitialized:
    {
        // Consolidation is not required or uninitialized
        PetscFunctionReturn(0);
    }
    case ConsolidationStatus::Device:
    {
        if (gpuProc == 0)
        {
            // Deallocate the CSR matrix values, solution and RHS
            CHECK(hipFree(valuesCons));
            CHECK(hipFree(pCons));
            CHECK(hipFree(rhsCons));
        }
        else
        {
            // Close the remaining IPC memory handles
            CHECK(hipIpcCloseMemHandle(valuesCons));
            CHECK(hipIpcCloseMemHandle(pCons));
            CHECK(hipIpcCloseMemHandle(rhsCons));
        }
        break;
    }
    case ConsolidationStatus::Host:
    {
        if(gpuProc == 0)
        {
            delete[] valuesCons;
            delete[] pCons;
            delete[] rhsCons;
        }
        break;
    }
    default:
    {
        SETERRQ(MPI_COMM_WORLD, PETSC_ERR_SUP_SYS,
                "Incorrect consolidation status set.\n");
        break;
    }

    }

    // Free the local GPU partitioning structures
    if(consolidationStatus == ConsolidationStatus::Device || consolidationStatus == ConsolidationStatus::Host)
    {
        nRowsInDevWorld.clear();
        nnzInDevWorld.clear();
        rowDispls.clear();
        nzDispls.clear();
    }

    consolidationStatus = ConsolidationStatus::Uninitialized;

    PetscFunctionReturn(0);
}