#include "hip/hip_runtime.h"
static char help[] = "Solves -Laplacian u - exp(u) = 0,  0 < x < 1 using GPU\n\n";
/*
   Same as ex47.c except it also uses the GPU to evaluate the function
*/

#include <petscdm.h>
#include <petscdmda.h>
#include <petscsnes.h>
#include <petsccusp.h>

#include <thrust/for_each.h>
#include <thrust/tuple.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>

extern PetscErrorCode ComputeFunction(SNES,Vec,Vec,void*), ComputeJacobian(SNES,Vec,Mat,Mat,void*);
PetscBool useCUSP = PETSC_FALSE;

int main(int argc,char **argv)
{
  SNES           snes;
  Vec            x,f;
  Mat            J;
  DM             da;
  PetscErrorCode ierr;
  char           *tmp,typeName[256];
  PetscBool      flg;

  PetscInitialize(&argc,&argv,(char*)0,help);
  ierr = PetscOptionsGetString(NULL,NULL,"-dm_vec_type",typeName,256,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = PetscStrstr(typeName,"cusp",&tmp);CHKERRQ(ierr);
    if (tmp) useCUSP = PETSC_TRUE;
  }

  ierr = DMDACreate1d(PETSC_COMM_WORLD,DM_BOUNDARY_NONE,-8,1,1,NULL,&da);CHKERRQ(ierr);
  ierr = DMCreateGlobalVector(da,&x); VecDuplicate(x,&f);CHKERRQ(ierr);
  ierr = DMSetMatType(da,MATAIJ);CHKERRQ(ierr);
  ierr = DMCreateMatrix(da,&J);CHKERRQ(ierr);

  ierr = SNESCreate(PETSC_COMM_WORLD,&snes);CHKERRQ(ierr);
  ierr = SNESSetFunction(snes,f,ComputeFunction,da);CHKERRQ(ierr);
  ierr = SNESSetJacobian(snes,J,J,ComputeJacobian,da);CHKERRQ(ierr);
  ierr = SNESSetFromOptions(snes);CHKERRQ(ierr);
  ierr = SNESSolve(snes,NULL,x);CHKERRQ(ierr);

  ierr = MatDestroy(&J);CHKERRQ(ierr);
  ierr = VecDestroy(&x);CHKERRQ(ierr);
  ierr = VecDestroy(&f);CHKERRQ(ierr);
  ierr = SNESDestroy(&snes);CHKERRQ(ierr);
  ierr = DMDestroy(&da);CHKERRQ(ierr);

  PetscFinalize();
  return 0;
}

struct ApplyStencil
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /* f = (2*x_i - x_(i+1) - x_(i-1))/h - h*exp(x_i) */
    thrust::get<0>(t) = 1;
    if ((thrust::get<4>(t) > 0) && (thrust::get<4>(t) < thrust::get<5>(t)-1)) {
      thrust::get<0>(t) = (2.0*thrust::get<1>(t) - thrust::get<2>(t) - thrust::get<3>(t)) / (thrust::get<6>(t)) - (thrust::get<6>(t))*exp(thrust::get<1>(t));
    } else if (thrust::get<4>(t) == 0) {
      thrust::get<0>(t) = thrust::get<1>(t) / (thrust::get<6>(t));
    } else if (thrust::get<4>(t) == thrust::get<5>(t)-1) {
      thrust::get<0>(t) = thrust::get<1>(t) / (thrust::get<6>(t));
    }
  }
};

PetscErrorCode ComputeFunction(SNES snes,Vec x,Vec f,void *ctx)
{
  PetscInt       i,Mx,xs,xm,xstartshift,xendshift,fstart,lsize;
  PetscScalar    *xx,*ff,hx;
  DM             da = (DM) ctx;
  Vec            xlocal;
  PetscErrorCode ierr;
  PetscMPIInt    rank,size;
  MPI_Comm       comm;
  cusp::array1d<PetscScalar,cusp::device_memory> *xarray,*farray;

  ierr = DMDAGetInfo(da,PETSC_IGNORE,&Mx,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
  hx   = 1.0/(PetscReal)(Mx-1);
  ierr = DMGetLocalVector(da,&xlocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalBegin(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalEnd(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);

  if (useCUSP) {
    ierr = VecCUSPGetArrayRead(xlocal,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(f,&farray);CHKERRQ(ierr);
    ierr = PetscObjectGetComm((PetscObject)da,&comm);CHKERRQ(ierr);
    ierr = MPI_Comm_size(comm,&size);CHKERRQ(ierr);
    ierr = MPI_Comm_rank(comm,&rank);CHKERRQ(ierr);
    if (rank) xstartshift = 1;
    else xstartshift = 0;
    if (rank != size-1) xendshift = 1;
    else xendshift = 0;
    ierr = VecGetOwnershipRange(f,&fstart,NULL);CHKERRQ(ierr);
    ierr = VecGetLocalSize(x,&lsize);CHKERRQ(ierr);
    try {
      thrust::for_each(
        thrust::make_zip_iterator(
          thrust::make_tuple(
            farray->begin(),
            xarray->begin()+xstartshift,
            xarray->begin()+xstartshift + 1,
            xarray->begin()+xstartshift - 1,
            thrust::counting_iterator<int>(fstart),
            thrust::constant_iterator<int>(Mx),
            thrust::constant_iterator<PetscScalar>(hx))),
        thrust::make_zip_iterator(
          thrust::make_tuple(
            farray->end(),
            xarray->end()-xendshift,
            xarray->end()-xendshift + 1,
            xarray->end()-xendshift - 1,
            thrust::counting_iterator<int>(fstart) + lsize,
            thrust::constant_iterator<int>(Mx),
            thrust::constant_iterator<PetscScalar>(hx))),
        ApplyStencil());
    }
    catch (char *all) {
      ierr = PetscPrintf(PETSC_COMM_WORLD, "Thrust is not working\n");CHKERRQ(ierr);
    }
    ierr = VecCUSPRestoreArrayRead(xlocal,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(f,&farray);CHKERRQ(ierr);
  } else {
    ierr = DMDAVecGetArray(da,xlocal,&xx);CHKERRQ(ierr);
    ierr = DMDAVecGetArray(da,f,&ff);CHKERRQ(ierr);
    ierr = DMDAGetCorners(da,&xs,NULL,NULL,&xm,NULL,NULL);CHKERRQ(ierr);

    for (i=xs; i<xs+xm; i++) {
      if (i == 0 || i == Mx-1) ff[i] = xx[i]/hx;
      else ff[i] =  (2.0*xx[i] - xx[i-1] - xx[i+1])/hx - hx*PetscExpScalar(xx[i]);
    }
    ierr = DMDAVecRestoreArray(da,xlocal,&xx);CHKERRQ(ierr);
    ierr = DMDAVecRestoreArray(da,f,&ff);CHKERRQ(ierr);
  }
  ierr = DMRestoreLocalVector(da,&xlocal);CHKERRQ(ierr);
  //  VecView(x,0);printf("f\n");
  //  VecView(f,0);
  return 0;

}
PetscErrorCode ComputeJacobian(SNES snes,Vec x,Mat J,Mat B,void *ctx)
{
  DM             da = (DM) ctx;
  PetscInt       i,Mx,xm,xs;
  PetscScalar    hx,*xx;
  Vec            xlocal;
  PetscErrorCode ierr;

  ierr = DMDAGetInfo(da,PETSC_IGNORE,&Mx,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE,PETSC_IGNORE);CHKERRQ(ierr);
  hx   = 1.0/(PetscReal)(Mx-1);
  ierr = DMGetLocalVector(da,&xlocal);DMGlobalToLocalBegin(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);
  ierr = DMGlobalToLocalEnd(da,x,INSERT_VALUES,xlocal);CHKERRQ(ierr);
  ierr = DMDAVecGetArray(da,xlocal,&xx);CHKERRQ(ierr);
  ierr = DMDAGetCorners(da,&xs,NULL,NULL,&xm,NULL,NULL);CHKERRQ(ierr);

  for (i=xs; i<xs+xm; i++) {
    if (i == 0 || i == Mx-1) {
      ierr = MatSetValue(J,i,i,1.0/hx,INSERT_VALUES);CHKERRQ(ierr);
    } else {
      ierr = MatSetValue(J,i,i-1,-1.0/hx,INSERT_VALUES);CHKERRQ(ierr);
      ierr = MatSetValue(J,i,i,2.0/hx - hx*PetscExpScalar(xx[i]),INSERT_VALUES);CHKERRQ(ierr);
      ierr = MatSetValue(J,i,i+1,-1.0/hx,INSERT_VALUES);CHKERRQ(ierr);
    }
  }
  ierr  = MatAssemblyBegin(J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr  = MatAssemblyEnd(J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr  = DMDAVecRestoreArray(da,xlocal,&xx);CHKERRQ(ierr);
  ierr  = DMRestoreLocalVector(da,&xlocal);CHKERRQ(ierr);
  return 0;
}

