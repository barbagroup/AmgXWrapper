
/*
    Provides an interface to the CUFFT package.
    Testing examples can be found in ~src/mat/examples/tests
*/

#include <petsc/private/matimpl.h>          /*I "petscmat.h" I*/
EXTERN_C_BEGIN
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
EXTERN_C_END

typedef struct {
  PetscInt     ndim;
  PetscInt     *dim;
  hipfftHandle  p_forward, p_backward;
  hipfftComplex *devArray;
} Mat_CUFFT;

#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqCUFFT"
PetscErrorCode MatMult_SeqCUFFT(Mat A, Vec x, Vec y)
{
  Mat_CUFFT      *cufft    = (Mat_CUFFT*) A->data;
  hipfftComplex   *devArray = cufft->devArray;
  PetscInt       ndim      = cufft->ndim, *dim = cufft->dim;
  PetscScalar    *x_array, *y_array;
  hipfftResult    result;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecGetArray(x, &x_array);CHKERRQ(ierr);
  ierr = VecGetArray(y, &y_array);CHKERRQ(ierr);
  if (!cufft->p_forward) {
    hipfftResult result;
    /* create a plan, then execute it */
    switch (ndim) {
    case 1:
      result = hipfftPlan1d(&cufft->p_forward, dim[0], HIPFFT_C2C, 1);CHKERRQ(result != HIPFFT_SUCCESS);
      break;
    case 2:
      result = hipfftPlan2d(&cufft->p_forward, dim[0], dim[1], HIPFFT_C2C);CHKERRQ(result != HIPFFT_SUCCESS);
      break;
    case 3:
      result = hipfftPlan3d(&cufft->p_forward, dim[0], dim[1], dim[2], HIPFFT_C2C);CHKERRQ(result != HIPFFT_SUCCESS);
      break;
    default:
      SETERRQ1(PETSC_COMM_SELF, PETSC_ERR_USER, "Cannot create plan for %d-dimensional transform", ndim);
    }
  }
  /* transfer to GPU memory */
  hipMemcpy(devArray, x_array, sizeof(hipfftComplex)*dim[ndim], hipMemcpyHostToDevice);
  /* execute transform */
  result = hipfftExecC2C(cufft->p_forward, devArray, devArray, HIPFFT_FORWARD);CHKERRQ(result != HIPFFT_SUCCESS);
  /* transfer from GPU memory */
  hipMemcpy(y_array, devArray, sizeof(hipfftComplex)*dim[ndim], hipMemcpyDeviceToHost);
  ierr = VecRestoreArray(y, &y_array);CHKERRQ(ierr);
  ierr = VecRestoreArray(x, &x_array);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMultTranspose_SeqCUFFT"
PetscErrorCode MatMultTranspose_SeqCUFFT(Mat A, Vec x, Vec y)
{
  Mat_CUFFT      *cufft    = (Mat_CUFFT*) A->data;
  hipfftComplex   *devArray = cufft->devArray;
  PetscInt       ndim      = cufft->ndim, *dim = cufft->dim;
  PetscScalar    *x_array, *y_array;
  hipfftResult    result;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecGetArray(x, &x_array);CHKERRQ(ierr);
  ierr = VecGetArray(y, &y_array);CHKERRQ(ierr);
  if (!cufft->p_backward) {
    /* create a plan, then execute it */
    switch (ndim) {
    case 1:
      result = hipfftPlan1d(&cufft->p_backward, dim[0], HIPFFT_C2C, 1);CHKERRQ(result != HIPFFT_SUCCESS);
      break;
    case 2:
      result = hipfftPlan2d(&cufft->p_backward, dim[0], dim[1], HIPFFT_C2C);CHKERRQ(result != HIPFFT_SUCCESS);
      break;
    case 3:
      result = hipfftPlan3d(&cufft->p_backward, dim[0], dim[1], dim[2], HIPFFT_C2C);CHKERRQ(result != HIPFFT_SUCCESS);
      break;
    default:
      SETERRQ1(PETSC_COMM_SELF, PETSC_ERR_USER, "Cannot create plan for %d-dimensional transform", ndim);
    }
  }
  /* transfer to GPU memory */
  hipMemcpy(devArray, x_array, sizeof(hipfftComplex)*dim[ndim], hipMemcpyHostToDevice);
  /* execute transform */
  result = hipfftExecC2C(cufft->p_forward, devArray, devArray, HIPFFT_BACKWARD);CHKERRQ(result != HIPFFT_SUCCESS);
  /* transfer from GPU memory */
  hipMemcpy(y_array, devArray, sizeof(hipfftComplex)*dim[ndim], hipMemcpyDeviceToHost);
  ierr = VecRestoreArray(y, &y_array);CHKERRQ(ierr);
  ierr = VecRestoreArray(x, &x_array);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqCUFFT"
PetscErrorCode MatDestroy_SeqCUFFT(Mat A)
{
  Mat_CUFFT      *cufft = (Mat_CUFFT*) A->data;
  hipfftResult    result;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscFree(cufft->dim);CHKERRQ(ierr);
  if (cufft->p_forward)  {result = hipfftDestroy(cufft->p_forward);CHKERRQ(result != HIPFFT_SUCCESS);}
  if (cufft->p_backward) {result = hipfftDestroy(cufft->p_backward);CHKERRQ(result != HIPFFT_SUCCESS);}
  hipFree(cufft->devArray);
  ierr = PetscFree(A->data);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)A,0);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCreateSeqCUFFT"
/*@
  MatCreateSeqCUFFT - Creates a matrix object that provides sequential FFT via the external package CUFFT

  Collective on MPI_Comm

  Input Parameters:
+ comm - MPI communicator, set to PETSC_COMM_SELF
. ndim - the ndim-dimensional transform
- dim  - array of size ndim, dim[i] contains the vector length in the i-dimension

  Output Parameter:
. A - the matrix

  Options Database Keys:
. -mat_cufft_plannerflags - set CUFFT planner flags

  Level: intermediate
@*/
PetscErrorCode  MatCreateSeqCUFFT(MPI_Comm comm, PetscInt ndim, const PetscInt dim[], Mat *A)
{
  Mat_CUFFT      *cufft;
  PetscInt       m, d;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (ndim < 0) SETERRQ1(PETSC_COMM_SELF, PETSC_ERR_USER, "ndim %d must be > 0", ndim);
  ierr = MatCreate(comm, A);CHKERRQ(ierr);
  m    = 1;
  for (d = 0; d < ndim; ++d) {
    if (dim[d] < 0) SETERRQ2(PETSC_COMM_SELF, PETSC_ERR_USER, "dim[%d]=%d must be > 0", d, dim[d]);
    m *= dim[d];
  }
  ierr = MatSetSizes(*A, m, m, m, m);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)*A, MATSEQCUFFT);CHKERRQ(ierr);

  ierr       = PetscNewLog(*A,&cufft);CHKERRQ(ierr);
  (*A)->data = (void*) cufft;
  ierr       = PetscMalloc1(ndim+1, &cufft->dim);CHKERRQ(ierr);
  ierr       = PetscMemcpy(cufft->dim, dim, ndim*sizeof(PetscInt));CHKERRQ(ierr);

  cufft->ndim       = ndim;
  cufft->p_forward  = 0;
  cufft->p_backward = 0;
  cufft->dim[ndim]  = m;

  /* GPU memory allocation */
  hipMalloc((void**) &cufft->devArray, sizeof(hipfftComplex)*m);

  (*A)->ops->mult          = MatMult_SeqCUFFT;
  (*A)->ops->multtranspose = MatMultTranspose_SeqCUFFT;
  (*A)->assembled          = PETSC_TRUE;
  (*A)->ops->destroy       = MatDestroy_SeqCUFFT;

  /* get runtime options */
  ierr = PetscOptionsBegin(comm, ((PetscObject)(*A))->prefix, "CUFFT Options", "Mat");CHKERRQ(ierr);
  ierr = PetscOptionsEnd();CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
