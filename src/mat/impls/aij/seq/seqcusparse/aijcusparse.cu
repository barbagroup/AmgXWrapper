/*
  Defines the basic matrix operations for the AIJ (compressed row)
  matrix storage format using the CUSPARSE library,
*/
#define PETSC_SKIP_SPINLOCK

#include <petscconf.h>
#include <../src/mat/impls/aij/seq/aij.h>          /*I "petscmat.h" I*/
#include <../src/mat/impls/sbaij/seq/sbaij.h>
#include <../src/vec/vec/impls/dvecimpl.h>
#include <petsc/private/vecimpl.h>
#undef VecType
#include <../src/mat/impls/aij/seq/seqcusparse/cusparsematimpl.h>

const char *const MatCUSPARSEStorageFormats[] = {"CSR","ELL","HYB","MatCUSPARSEStorageFormat","MAT_CUSPARSE_",0};

static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat);
static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);

static PetscErrorCode CsrMatrix_Destroy(CsrMatrix**);
static PetscErrorCode Mat_SeqAIJCUSPARSETriFactorStruct_Destroy(Mat_SeqAIJCUSPARSETriFactorStruct**);
static PetscErrorCode Mat_SeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSEMultStruct**,MatCUSPARSEStorageFormat);
static PetscErrorCode Mat_SeqAIJCUSPARSETriFactors_Destroy(Mat_SeqAIJCUSPARSETriFactors**);
static PetscErrorCode Mat_SeqAIJCUSPARSE_Destroy(Mat_SeqAIJCUSPARSE**);

#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSESetStream"
PetscErrorCode MatCUSPARSESetStream(Mat A,const hipStream_t stream)
{
  hipsparseStatus_t   stat;
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  cusparsestruct->stream = stream;
  stat = hipsparseSetStream(cusparsestruct->handle,cusparsestruct->stream);CHKERRCUSP(stat);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSESetHandle"
PetscErrorCode MatCUSPARSESetHandle(Mat A,const hipsparseHandle_t handle)
{
  hipsparseStatus_t   stat;
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  if (cusparsestruct->handle)
    stat = hipsparseDestroy(cusparsestruct->handle);CHKERRCUSP(stat);
  cusparsestruct->handle = handle;
  stat = hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUSP(stat);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSEClearHandle"
PetscErrorCode MatCUSPARSEClearHandle(Mat A)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  PetscFunctionBegin;
  if (cusparsestruct->handle)
    cusparsestruct->handle = 0;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatFactorGetSolverPackage_seqaij_cusparse"
PetscErrorCode MatFactorGetSolverPackage_seqaij_cusparse(Mat A,const MatSolverPackage *type)
{
  PetscFunctionBegin;
  *type = MATSOLVERCUSPARSE;
  PetscFunctionReturn(0);
}

/*MC
  MATSOLVERCUSPARSE = "cusparse" - A matrix type providing triangular solvers for seq matrices
  on a single GPU of type, seqaijcusparse, aijcusparse, or seqaijcusp, aijcusp. Currently supported
  algorithms are ILU(k) and ICC(k). Typically, deeper factorizations (larger k) results in poorer
  performance in the triangular solves. Full LU, and Cholesky decompositions can be solved through the
  CUSPARSE triangular solve algorithm. However, the performance can be quite poor and thus these
  algorithms are not recommended. This class does NOT support direct solver operations.

  Level: beginner

.seealso: PCFactorSetMatSolverPackage(), MatSolverPackage, MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

#undef __FUNCT__
#define __FUNCT__ "MatGetFactor_seqaijcusparse_cusparse"
PETSC_EXTERN PetscErrorCode MatGetFactor_seqaijcusparse_cusparse(Mat A,MatFactorType ftype,Mat *B)
{
  PetscErrorCode ierr;
  PetscInt       n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),B);CHKERRQ(ierr);
  (*B)->factortype = ftype;
  ierr = MatSetSizes(*B,n,n,n,n);CHKERRQ(ierr);
  ierr = MatSetType(*B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);

  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU || ftype == MAT_FACTOR_ILUDT) {
    ierr = MatSetBlockSizesFromMats(*B,A,A);CHKERRQ(ierr);
    (*B)->ops->ilufactorsymbolic = MatILUFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->lufactorsymbolic  = MatLUFactorSymbolic_SeqAIJCUSPARSE;
  } else if (ftype == MAT_FACTOR_CHOLESKY || ftype == MAT_FACTOR_ICC) {
    (*B)->ops->iccfactorsymbolic      = MatICCFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqAIJCUSPARSE;
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Factor type not supported for CUSPARSE Matrix Types");

  ierr = MatSeqAIJSetPreallocation(*B,MAT_SKIP_ALLOCATION,NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)(*B),"MatFactorGetSolverPackage_C",MatFactorGetSolverPackage_seqaij_cusparse);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSESetFormat_SeqAIJCUSPARSE"
PETSC_INTERN PetscErrorCode MatCUSPARSESetFormat_SeqAIJCUSPARSE(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
#if CUDA_VERSION>=4020
  switch (op) {
  case MAT_CUSPARSE_MULT:
    cusparsestruct->format = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparsestruct->format = format;
    break;
  default:
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPARSEFormatOperation. MAT_CUSPARSE_MULT and MAT_CUSPARSE_ALL are currently supported.",op);
  }
#else
  if (format==MAT_CUSPARSE_ELL || format==MAT_CUSPARSE_HYB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"ELL (Ellpack) and HYB (Hybrid) storage format require CUDA 4.2 or later.");
#endif
  PetscFunctionReturn(0);
}

/*@
   MatCUSPARSESetFormat - Sets the storage format of CUSPARSE matrices for a particular
   operation. Only the MatMult operation can use different GPU storage formats
   for MPIAIJCUSPARSE matrices.
   Not Collective

   Input Parameters:
+  A - Matrix of type SEQAIJCUSPARSE
.  op - MatCUSPARSEFormatOperation. SEQAIJCUSPARSE matrices support MAT_CUSPARSE_MULT and MAT_CUSPARSE_ALL. MPIAIJCUSPARSE matrices support MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_OFFDIAG, and MAT_CUSPARSE_ALL.
-  format - MatCUSPARSEStorageFormat (one of MAT_CUSPARSE_CSR, MAT_CUSPARSE_ELL, MAT_CUSPARSE_HYB. The latter two require CUDA 4.2)

   Output Parameter:

   Level: intermediate

.seealso: MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
@*/
#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSESetFormat"
PetscErrorCode MatCUSPARSESetFormat(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A, MAT_CLASSID,1);
  ierr = PetscTryMethod(A, "MatCUSPARSESetFormat_C",(Mat,MatCUSPARSEFormatOperation,MatCUSPARSEStorageFormat),(A,op,format));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSetFromOptions_SeqAIJCUSPARSE"
static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat A)
{
  PetscErrorCode           ierr;
  MatCUSPARSEStorageFormat format;
  PetscBool                flg;
  Mat_SeqAIJCUSPARSE       *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  ierr = PetscOptionsHead(PetscOptionsObject,"SeqAIJCUSPARSE options");CHKERRQ(ierr);
  ierr = PetscObjectOptionsBegin((PetscObject)A);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = PetscOptionsEnum("-mat_cusparse_mult_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparsestruct->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT,format);CHKERRQ(ierr);
    }
  } 
  ierr = PetscOptionsEnum("-mat_cusparse_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV and TriSolve",
                          "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparsestruct->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_ALL,format);CHKERRQ(ierr);
  }
  ierr = PetscOptionsEnd();CHKERRQ(ierr);
  PetscFunctionReturn(0);

}

#undef __FUNCT__
#define __FUNCT__ "MatILUFactorSymbolic_SeqAIJCUSPARSE"
static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatILUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatLUFactorSymbolic_SeqAIJCUSPARSE"
static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatLUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatICCFactorSymbolic_SeqAIJCUSPARSE"
static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatICCFactorSymbolic_SeqAIJ(B,A,perm,info);CHKERRQ(ierr);
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCholeskyFactorSymbolic_SeqAIJCUSPARSE"
static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCholeskyFactorSymbolic_SeqAIJ(B,A,perm,info);CHKERRQ(ierr);
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEBuildILULowerTriMatrix"
static PetscErrorCode MatSeqAIJCUSPARSEBuildILULowerTriMatrix(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  PetscInt                          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  hipsparseStatus_t                  stat;
  const PetscInt                    *ai = a->i,*aj = a->j,*vi;
  const MatScalar                   *aa = a->a,*v;
  PetscInt                          *AiLo, *AjLo;
  PetscScalar                       *AALo;
  PetscInt                          i,nz, nzLower, offset, rowOffset;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    try {
      /* first figure out the number of nonzeros in the lower triangular matrix including 1's on the diagonal. */
      nzLower=n+ai[n]-ai[1];

      /* Allocate Space for the lower triangular matrix */
      ierr = hipHostMalloc((void**) &AiLo, (n+1)*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AjLo, nzLower*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AALo, nzLower*sizeof(PetscScalar));CHKERRCUSP(ierr);

      /* Fill the lower triangular matrix */
      AiLo[0]  = (PetscInt) 0;
      AiLo[n]  = nzLower;
      AjLo[0]  = (PetscInt) 0;
      AALo[0]  = (MatScalar) 1.0;
      v        = aa;
      vi       = aj;
      offset   = 1;
      rowOffset= 1;
      for (i=1; i<n; i++) {
        nz = ai[i+1] - ai[i];
        /* additional 1 for the term on the diagonal */
        AiLo[i]    = rowOffset;
        rowOffset += nz+1;

        ierr = PetscMemcpy(&(AjLo[offset]), vi, nz*sizeof(PetscInt));CHKERRQ(ierr);
        ierr = PetscMemcpy(&(AALo[offset]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);

        offset      += nz;
        AjLo[offset] = (PetscInt) i;
        AALo[offset] = (MatScalar) 1.0;
        offset      += 1;

        v  += nz;
        vi += nz;
      }

      /* allocate space for the triangular factor information */
      loTriFactor = new Mat_SeqAIJCUSPARSETriFactorStruct;

      /* Create the matrix description */
      stat = hipsparseCreateMatDescr(&loTriFactor->descr);CHKERRCUSP(stat);
      stat = hipsparseSetMatIndexBase(loTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSP(stat);
      stat = hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUSP(stat);
      stat = hipsparseSetMatFillMode(loTriFactor->descr, HIPSPARSE_FILL_MODE_LOWER);CHKERRCUSP(stat);
      stat = hipsparseSetMatDiagType(loTriFactor->descr, HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUSP(stat);

      /* Create the solve analysis information */
      stat = cusparseCreateSolveAnalysisInfo(&loTriFactor->solveInfo);CHKERRCUSP(stat);

      /* set the operation */
      loTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

      /* set the matrix */
      loTriFactor->csrMat = new CsrMatrix;
      loTriFactor->csrMat->num_rows = n;
      loTriFactor->csrMat->num_cols = n;
      loTriFactor->csrMat->num_entries = nzLower;

      loTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(n+1);
      loTriFactor->csrMat->row_offsets->assign(AiLo, AiLo+n+1);

      loTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(nzLower);
      loTriFactor->csrMat->column_indices->assign(AjLo, AjLo+nzLower);

      loTriFactor->csrMat->values = new THRUSTARRAY(nzLower);
      loTriFactor->csrMat->values->assign(AALo, AALo+nzLower);

      /* perform the solve analysis */
      stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactor->solveOp,
                               loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                               loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                               loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo);CHKERRCUSP(stat);

      /* assign the pointer. Is this really necessary? */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = loTriFactor;

      ierr = hipHostFree(AiLo);CHKERRCUSP(ierr);
      ierr = hipHostFree(AjLo);CHKERRCUSP(ierr);
      ierr = hipHostFree(AALo);CHKERRCUSP(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEBuildILUUpperTriMatrix"
static PetscErrorCode MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  PetscInt                          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t                  stat;
  const PetscInt                    *aj = a->j,*adiag = a->diag,*vi;
  const MatScalar                   *aa = a->a,*v;
  PetscInt                          *AiUp, *AjUp;
  PetscScalar                       *AAUp;
  PetscInt                          i,nz, nzUpper, offset;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    try {
      /* next, figure out the number of nonzeros in the upper triangular matrix. */
      nzUpper = adiag[0]-adiag[n];

      /* Allocate Space for the upper triangular matrix */
      ierr = hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUSP(ierr);

      /* Fill the upper triangular matrix */
      AiUp[0]=(PetscInt) 0;
      AiUp[n]=nzUpper;
      offset = nzUpper;
      for (i=n-1; i>=0; i--) {
        v  = aa + adiag[i+1] + 1;
        vi = aj + adiag[i+1] + 1;

        /* number of elements NOT on the diagonal */
        nz = adiag[i] - adiag[i+1]-1;

        /* decrement the offset */
        offset -= (nz+1);

        /* first, set the diagonal elements */
        AjUp[offset] = (PetscInt) i;
        AAUp[offset] = 1./v[nz];
        AiUp[i]      = AiUp[i+1] - (nz+1);

        ierr = PetscMemcpy(&(AjUp[offset+1]), vi, nz*sizeof(PetscInt));CHKERRQ(ierr);
        ierr = PetscMemcpy(&(AAUp[offset+1]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);
      }

      /* allocate space for the triangular factor information */
      upTriFactor = new Mat_SeqAIJCUSPARSETriFactorStruct;

      /* Create the matrix description */
      stat = hipsparseCreateMatDescr(&upTriFactor->descr);CHKERRCUSP(stat);
      stat = hipsparseSetMatIndexBase(upTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSP(stat);
      stat = hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUSP(stat);
      stat = hipsparseSetMatFillMode(upTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUSP(stat);
      stat = hipsparseSetMatDiagType(upTriFactor->descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);

      /* Create the solve analysis information */
      stat = cusparseCreateSolveAnalysisInfo(&upTriFactor->solveInfo);CHKERRCUSP(stat);

      /* set the operation */
      upTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

      /* set the matrix */
      upTriFactor->csrMat = new CsrMatrix;
      upTriFactor->csrMat->num_rows = n;
      upTriFactor->csrMat->num_cols = n;
      upTriFactor->csrMat->num_entries = nzUpper;

      upTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(n+1);
      upTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+n+1);

      upTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(nzUpper);
      upTriFactor->csrMat->column_indices->assign(AjUp, AjUp+nzUpper);

      upTriFactor->csrMat->values = new THRUSTARRAY(nzUpper);
      upTriFactor->csrMat->values->assign(AAUp, AAUp+nzUpper);

      /* perform the solve analysis */
      stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactor->solveOp,
                               upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                               upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                               upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo);CHKERRCUSP(stat);

      /* assign the pointer. Is this really necessary? */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = upTriFactor;

      ierr = hipHostFree(AiUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AjUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AAUp);CHKERRCUSP(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU"
static PetscErrorCode MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           isrow = a->row,iscol = a->icol;
  PetscBool                    row_identity,col_identity;
  const PetscInt               *r,*c;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSEBuildILULowerTriMatrix(A);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(A);CHKERRQ(ierr);

  cusparseTriFactors->workVector = new THRUSTARRAY;
  cusparseTriFactors->workVector->resize(n);
  cusparseTriFactors->nnz=a->nz;

  A->valid_GPU_matrix = PETSC_CUSP_BOTH;
  /*lower triangular indices */
  ierr = ISGetIndices(isrow,&r);CHKERRQ(ierr);
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  if (!row_identity) {
    cusparseTriFactors->rpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->rpermIndices->assign(r, r+n);
  }
  ierr = ISRestoreIndices(isrow,&r);CHKERRQ(ierr);

  /*upper triangular indices */
  ierr = ISGetIndices(iscol,&c);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (!col_identity) {
    cusparseTriFactors->cpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->cpermIndices->assign(c, c+n);
  }
  ierr = ISRestoreIndices(iscol,&c);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEBuildICCTriMatrices"
static PetscErrorCode MatSeqAIJCUSPARSEBuildICCTriMatrices(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t                  stat;
  PetscErrorCode                    ierr;
  PetscInt                          *AiUp, *AjUp;
  PetscScalar                       *AAUp;
  PetscScalar                       *AALo;
  PetscInt                          nzUpper = a->nz,n = A->rmap->n,i,offset,nz,j;
  Mat_SeqSBAIJ                      *b = (Mat_SeqSBAIJ*)A->data;
  const PetscInt                    *ai = b->i,*aj = b->j,*vj;
  const MatScalar                   *aa = b->a,*v;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    try {
      /* Allocate Space for the upper triangular matrix */
      ierr = hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AALo, nzUpper*sizeof(PetscScalar));CHKERRCUSP(ierr);

      /* Fill the upper triangular matrix */
      AiUp[0]=(PetscInt) 0;
      AiUp[n]=nzUpper;
      offset = 0;
      for (i=0; i<n; i++) {
        /* set the pointers */
        v  = aa + ai[i];
        vj = aj + ai[i];
        nz = ai[i+1] - ai[i] - 1; /* exclude diag[i] */

        /* first, set the diagonal elements */
        AjUp[offset] = (PetscInt) i;
        AAUp[offset] = 1.0/v[nz];
        AiUp[i]      = offset;
        AALo[offset] = 1.0/v[nz];

        offset+=1;
        if (nz>0) {
          ierr = PetscMemcpy(&(AjUp[offset]), vj, nz*sizeof(PetscInt));CHKERRQ(ierr);
          ierr = PetscMemcpy(&(AAUp[offset]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);
          for (j=offset; j<offset+nz; j++) {
            AAUp[j] = -AAUp[j];
            AALo[j] = AAUp[j]/v[nz];
          }
          offset+=nz;
        }
      }

      /* allocate space for the triangular factor information */
      upTriFactor = new Mat_SeqAIJCUSPARSETriFactorStruct;

      /* Create the matrix description */
      stat = hipsparseCreateMatDescr(&upTriFactor->descr);CHKERRCUSP(stat);
      stat = hipsparseSetMatIndexBase(upTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSP(stat);
      stat = hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUSP(stat);
      stat = hipsparseSetMatFillMode(upTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUSP(stat);
      stat = hipsparseSetMatDiagType(upTriFactor->descr, HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUSP(stat);

      /* Create the solve analysis information */
      stat = cusparseCreateSolveAnalysisInfo(&upTriFactor->solveInfo);CHKERRCUSP(stat);

      /* set the operation */
      upTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

      /* set the matrix */
      upTriFactor->csrMat = new CsrMatrix;
      upTriFactor->csrMat->num_rows = A->rmap->n;
      upTriFactor->csrMat->num_cols = A->cmap->n;
      upTriFactor->csrMat->num_entries = a->nz;

      upTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
      upTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+A->rmap->n+1);

      upTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(a->nz);
      upTriFactor->csrMat->column_indices->assign(AjUp, AjUp+a->nz);

      upTriFactor->csrMat->values = new THRUSTARRAY(a->nz);
      upTriFactor->csrMat->values->assign(AAUp, AAUp+a->nz);

      /* perform the solve analysis */
      stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactor->solveOp,
                               upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                               upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                               upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo);CHKERRCUSP(stat);

      /* assign the pointer. Is this really necessary? */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = upTriFactor;

      /* allocate space for the triangular factor information */
      loTriFactor = new Mat_SeqAIJCUSPARSETriFactorStruct;

      /* Create the matrix description */
      stat = hipsparseCreateMatDescr(&loTriFactor->descr);CHKERRCUSP(stat);
      stat = hipsparseSetMatIndexBase(loTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSP(stat);
      stat = hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUSP(stat);
      stat = hipsparseSetMatFillMode(loTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUSP(stat);
      stat = hipsparseSetMatDiagType(loTriFactor->descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);

      /* Create the solve analysis information */
      stat = cusparseCreateSolveAnalysisInfo(&loTriFactor->solveInfo);CHKERRCUSP(stat);

      /* set the operation */
      loTriFactor->solveOp = HIPSPARSE_OPERATION_TRANSPOSE;

      /* set the matrix */
      loTriFactor->csrMat = new CsrMatrix;
      loTriFactor->csrMat->num_rows = A->rmap->n;
      loTriFactor->csrMat->num_cols = A->cmap->n;
      loTriFactor->csrMat->num_entries = a->nz;

      loTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
      loTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+A->rmap->n+1);

      loTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(a->nz);
      loTriFactor->csrMat->column_indices->assign(AjUp, AjUp+a->nz);

      loTriFactor->csrMat->values = new THRUSTARRAY(a->nz);
      loTriFactor->csrMat->values->assign(AALo, AALo+a->nz);

      /* perform the solve analysis */
      stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactor->solveOp,
                               loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                               loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                               loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo);CHKERRCUSP(stat);

      /* assign the pointer. Is this really necessary? */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = loTriFactor;

      A->valid_GPU_matrix = PETSC_CUSP_BOTH;
      ierr = hipHostFree(AiUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AjUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AAUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AALo);CHKERRCUSP(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU"
static PetscErrorCode MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           ip = a->row;
  const PetscInt               *rip;
  PetscBool                    perm_identity;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSEBuildICCTriMatrices(A);CHKERRQ(ierr);
  cusparseTriFactors->workVector = new THRUSTARRAY;
  cusparseTriFactors->workVector->resize(n);
  cusparseTriFactors->nnz=(a->nz-n)*2 + n;

  /*lower triangular indices */
  ierr = ISGetIndices(ip,&rip);CHKERRQ(ierr);
  ierr = ISIdentity(ip,&perm_identity);CHKERRQ(ierr);
  if (!perm_identity) {
    cusparseTriFactors->rpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->rpermIndices->assign(rip, rip+n);
    cusparseTriFactors->cpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->cpermIndices->assign(rip, rip+n);
  }
  ierr = ISRestoreIndices(ip,&rip);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatLUFactorNumeric_SeqAIJCUSPARSE"
static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             isrow = b->row,iscol = b->col;
  PetscBool      row_identity,col_identity;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatLUFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);
  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (row_identity && col_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
  }

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCholeskyFactorNumeric_SeqAIJCUSPARSE"
static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             ip = b->row;
  PetscBool      perm_identity;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCholeskyFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);

  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(ip,&perm_identity);CHKERRQ(ierr);
  if (perm_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
  }

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEAnalyzeTransposeForSolve"
static PetscErrorCode MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(Mat A)
{
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  hipsparseStatus_t                  stat;
  hipsparseIndexBase_t               indexBase;
  hipsparseMatrixType_t              matrixType;
  hipsparseFillMode_t                fillMode;
  hipsparseDiagType_t                diagType;

  PetscFunctionBegin;

  /*********************************************/
  /* Now the Transpose of the Lower Tri Factor */
  /*********************************************/

  /* allocate space for the transpose of the lower triangular factor */
  loTriFactorT = new Mat_SeqAIJCUSPARSETriFactorStruct;

  /* set the matrix descriptors of the lower triangular factor */
  matrixType = hipsparseGetMatType(loTriFactor->descr);
  indexBase = hipsparseGetMatIndexBase(loTriFactor->descr);
  fillMode = hipsparseGetMatFillMode(loTriFactor->descr)==HIPSPARSE_FILL_MODE_UPPER ?
    HIPSPARSE_FILL_MODE_LOWER : HIPSPARSE_FILL_MODE_UPPER;
  diagType = hipsparseGetMatDiagType(loTriFactor->descr);

  /* Create the matrix description */
  stat = hipsparseCreateMatDescr(&loTriFactorT->descr);CHKERRCUSP(stat);
  stat = hipsparseSetMatIndexBase(loTriFactorT->descr, indexBase);CHKERRCUSP(stat);
  stat = hipsparseSetMatType(loTriFactorT->descr, matrixType);CHKERRCUSP(stat);
  stat = hipsparseSetMatFillMode(loTriFactorT->descr, fillMode);CHKERRCUSP(stat);
  stat = hipsparseSetMatDiagType(loTriFactorT->descr, diagType);CHKERRCUSP(stat);

  /* Create the solve analysis information */
  stat = cusparseCreateSolveAnalysisInfo(&loTriFactorT->solveInfo);CHKERRCUSP(stat);

  /* set the operation */
  loTriFactorT->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  /* allocate GPU space for the CSC of the lower triangular factor*/
  loTriFactorT->csrMat = new CsrMatrix;
  loTriFactorT->csrMat->num_rows = loTriFactor->csrMat->num_rows;
  loTriFactorT->csrMat->num_cols = loTriFactor->csrMat->num_cols;
  loTriFactorT->csrMat->num_entries = loTriFactor->csrMat->num_entries;
  loTriFactorT->csrMat->row_offsets = new THRUSTINTARRAY32(loTriFactor->csrMat->num_rows+1);
  loTriFactorT->csrMat->column_indices = new THRUSTINTARRAY32(loTriFactor->csrMat->num_entries);
  loTriFactorT->csrMat->values = new THRUSTARRAY(loTriFactor->csrMat->num_entries);

  /* compute the transpose of the lower triangular factor, i.e. the CSC */
  stat = cusparse_csr2csc(cusparseTriFactors->handle, loTriFactor->csrMat->num_rows,
                          loTriFactor->csrMat->num_cols, loTriFactor->csrMat->num_entries,
                          loTriFactor->csrMat->values->data().get(),
                          loTriFactor->csrMat->row_offsets->data().get(),
                          loTriFactor->csrMat->column_indices->data().get(),
                          loTriFactorT->csrMat->values->data().get(),
                          loTriFactorT->csrMat->column_indices->data().get(),
                          loTriFactorT->csrMat->row_offsets->data().get(),
                          HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUSP(stat);

  /* perform the solve analysis on the transposed matrix */
  stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactorT->solveOp,
                           loTriFactorT->csrMat->num_rows, loTriFactorT->csrMat->num_entries,
                           loTriFactorT->descr, loTriFactorT->csrMat->values->data().get(),
                           loTriFactorT->csrMat->row_offsets->data().get(), loTriFactorT->csrMat->column_indices->data().get(),
                           loTriFactorT->solveInfo);CHKERRCUSP(stat);

  /* assign the pointer. Is this really necessary? */
  ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtrTranspose = loTriFactorT;

  /*********************************************/
  /* Now the Transpose of the Upper Tri Factor */
  /*********************************************/

  /* allocate space for the transpose of the upper triangular factor */
  upTriFactorT = new Mat_SeqAIJCUSPARSETriFactorStruct;

  /* set the matrix descriptors of the upper triangular factor */
  matrixType = hipsparseGetMatType(upTriFactor->descr);
  indexBase = hipsparseGetMatIndexBase(upTriFactor->descr);
  fillMode = hipsparseGetMatFillMode(upTriFactor->descr)==HIPSPARSE_FILL_MODE_UPPER ?
    HIPSPARSE_FILL_MODE_LOWER : HIPSPARSE_FILL_MODE_UPPER;
  diagType = hipsparseGetMatDiagType(upTriFactor->descr);

  /* Create the matrix description */
  stat = hipsparseCreateMatDescr(&upTriFactorT->descr);CHKERRCUSP(stat);
  stat = hipsparseSetMatIndexBase(upTriFactorT->descr, indexBase);CHKERRCUSP(stat);
  stat = hipsparseSetMatType(upTriFactorT->descr, matrixType);CHKERRCUSP(stat);
  stat = hipsparseSetMatFillMode(upTriFactorT->descr, fillMode);CHKERRCUSP(stat);
  stat = hipsparseSetMatDiagType(upTriFactorT->descr, diagType);CHKERRCUSP(stat);

  /* Create the solve analysis information */
  stat = cusparseCreateSolveAnalysisInfo(&upTriFactorT->solveInfo);CHKERRCUSP(stat);

  /* set the operation */
  upTriFactorT->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  /* allocate GPU space for the CSC of the upper triangular factor*/
  upTriFactorT->csrMat = new CsrMatrix;
  upTriFactorT->csrMat->num_rows = upTriFactor->csrMat->num_rows;
  upTriFactorT->csrMat->num_cols = upTriFactor->csrMat->num_cols;
  upTriFactorT->csrMat->num_entries = upTriFactor->csrMat->num_entries;
  upTriFactorT->csrMat->row_offsets = new THRUSTINTARRAY32(upTriFactor->csrMat->num_rows+1);
  upTriFactorT->csrMat->column_indices = new THRUSTINTARRAY32(upTriFactor->csrMat->num_entries);
  upTriFactorT->csrMat->values = new THRUSTARRAY(upTriFactor->csrMat->num_entries);

  /* compute the transpose of the upper triangular factor, i.e. the CSC */
  stat = cusparse_csr2csc(cusparseTriFactors->handle, upTriFactor->csrMat->num_rows,
                          upTriFactor->csrMat->num_cols, upTriFactor->csrMat->num_entries,
                          upTriFactor->csrMat->values->data().get(),
                          upTriFactor->csrMat->row_offsets->data().get(),
                          upTriFactor->csrMat->column_indices->data().get(),
                          upTriFactorT->csrMat->values->data().get(),
                          upTriFactorT->csrMat->column_indices->data().get(),
                          upTriFactorT->csrMat->row_offsets->data().get(),
                          HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUSP(stat);

  /* perform the solve analysis on the transposed matrix */
  stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactorT->solveOp,
                           upTriFactorT->csrMat->num_rows, upTriFactorT->csrMat->num_entries,
                           upTriFactorT->descr, upTriFactorT->csrMat->values->data().get(),
                           upTriFactorT->csrMat->row_offsets->data().get(), upTriFactorT->csrMat->column_indices->data().get(),
                           upTriFactorT->solveInfo);CHKERRCUSP(stat);

  /* assign the pointer. Is this really necessary? */
  ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtrTranspose = upTriFactorT;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEGenerateTransposeForMult"
static PetscErrorCode MatSeqAIJCUSPARSEGenerateTransposeForMult(Mat A)
{
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  Mat_SeqAIJCUSPARSEMultStruct *matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  hipsparseStatus_t             stat;
  hipsparseIndexBase_t          indexBase;
  hipError_t                  err;

  PetscFunctionBegin;

  /* allocate space for the triangular factor information */
  matstructT = new Mat_SeqAIJCUSPARSEMultStruct;
  stat = hipsparseCreateMatDescr(&matstructT->descr);CHKERRCUSP(stat);
  indexBase = hipsparseGetMatIndexBase(matstruct->descr);
  stat = hipsparseSetMatIndexBase(matstructT->descr, indexBase);CHKERRCUSP(stat);
  stat = hipsparseSetMatType(matstructT->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSP(stat);

  /* set alpha and beta */
  err = hipMalloc((void **)&(matstructT->alpha),sizeof(PetscScalar));CHKERRCUSP(err);
  err = hipMemcpy(matstructT->alpha,&ALPHA,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUSP(err);
  err = hipMalloc((void **)&(matstructT->beta),sizeof(PetscScalar));CHKERRCUSP(err);
  err = hipMemcpy(matstructT->beta,&BETA,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUSP(err);
  stat = hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUSP(stat);

  if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
    CsrMatrix *matrix = (CsrMatrix*)matstruct->mat;
    CsrMatrix *matrixT= new CsrMatrix;
    matrixT->num_rows = A->rmap->n;
    matrixT->num_cols = A->cmap->n;
    matrixT->num_entries = a->nz;
    matrixT->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
    matrixT->column_indices = new THRUSTINTARRAY32(a->nz);
    matrixT->values = new THRUSTARRAY(a->nz);

    /* compute the transpose of the upper triangular factor, i.e. the CSC */
    indexBase = hipsparseGetMatIndexBase(matstruct->descr);
    stat = cusparse_csr2csc(cusparsestruct->handle, matrix->num_rows,
                            matrix->num_cols, matrix->num_entries,
                            matrix->values->data().get(),
                            matrix->row_offsets->data().get(),
                            matrix->column_indices->data().get(),
                            matrixT->values->data().get(),
                            matrixT->column_indices->data().get(),
                            matrixT->row_offsets->data().get(),
                            HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUSP(stat);

    /* assign the pointer */
    matstructT->mat = matrixT;

  } else if (cusparsestruct->format==MAT_CUSPARSE_ELL || cusparsestruct->format==MAT_CUSPARSE_HYB) {
#if CUDA_VERSION>=5000
    /* First convert HYB to CSR */
    CsrMatrix *temp= new CsrMatrix;
    temp->num_rows = A->rmap->n;
    temp->num_cols = A->cmap->n;
    temp->num_entries = a->nz;
    temp->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
    temp->column_indices = new THRUSTINTARRAY32(a->nz);
    temp->values = new THRUSTARRAY(a->nz);


    stat = cusparse_hyb2csr(cusparsestruct->handle,
                            matstruct->descr, (hipsparseHybMat_t)matstruct->mat,
                            temp->values->data().get(),
                            temp->row_offsets->data().get(),
                            temp->column_indices->data().get());CHKERRCUSP(stat);

    /* Next, convert CSR to CSC (i.e. the matrix transpose) */
    CsrMatrix *tempT= new CsrMatrix;
    tempT->num_rows = A->rmap->n;
    tempT->num_cols = A->cmap->n;
    tempT->num_entries = a->nz;
    tempT->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
    tempT->column_indices = new THRUSTINTARRAY32(a->nz);
    tempT->values = new THRUSTARRAY(a->nz);

    stat = cusparse_csr2csc(cusparsestruct->handle, temp->num_rows,
                            temp->num_cols, temp->num_entries,
                            temp->values->data().get(),
                            temp->row_offsets->data().get(),
                            temp->column_indices->data().get(),
                            tempT->values->data().get(),
                            tempT->column_indices->data().get(),
                            tempT->row_offsets->data().get(),
                            HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUSP(stat);

    /* Last, convert CSC to HYB */
    hipsparseHybMat_t hybMat;
    stat = hipsparseCreateHybMat(&hybMat);CHKERRCUSP(stat);
    hipsparseHybPartition_t partition = cusparsestruct->format==MAT_CUSPARSE_ELL ?
      HIPSPARSE_HYB_PARTITION_MAX : HIPSPARSE_HYB_PARTITION_AUTO;
    stat = cusparse_csr2hyb(cusparsestruct->handle, A->rmap->n, A->cmap->n,
                            matstructT->descr, tempT->values->data().get(),
                            tempT->row_offsets->data().get(),
                            tempT->column_indices->data().get(),
                            hybMat, 0, partition);CHKERRCUSP(stat);

    /* assign the pointer */
    matstructT->mat = hybMat;

    /* delete temporaries */
    if (tempT) {
      if (tempT->values) delete (THRUSTARRAY*) tempT->values;
      if (tempT->column_indices) delete (THRUSTINTARRAY32*) tempT->column_indices;
      if (tempT->row_offsets) delete (THRUSTINTARRAY32*) tempT->row_offsets;
      delete (CsrMatrix*) tempT;
    }
    if (temp) {
      if (temp->values) delete (THRUSTARRAY*) temp->values;
      if (temp->column_indices) delete (THRUSTINTARRAY32*) temp->column_indices;
      if (temp->row_offsets) delete (THRUSTINTARRAY32*) temp->row_offsets;
      delete (CsrMatrix*) temp;
    }
#else
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"ELL (Ellpack) and HYB (Hybrid) storage format for the Matrix Transpose (in MatMultTranspose) require CUDA 5.0 or later.");
#endif
  }
  /* assign the compressed row indices */
  matstructT->cprowIndices = new THRUSTINTARRAY;

  /* assign the pointer */
  ((Mat_SeqAIJCUSPARSE*)A->spptr)->matTranspose = matstructT;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSolveTranspose_SeqAIJCUSPARSE"
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  CUSPARRAY                         *xGPU, *bGPU;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  /* Analyze the matrix and create the transpose ... on the fly */
  if (!loTriFactorT && !upTriFactorT) {
    ierr = MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A);CHKERRQ(ierr);
    loTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
    upTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  }

  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* First, reorder with the row permutation */
  thrust::copy(thrust::make_permutation_iterator(bGPU->begin(), cusparseTriFactors->rpermIndices->begin()),
               thrust::make_permutation_iterator(bGPU->end(), cusparseTriFactors->rpermIndices->end()),
               xGPU->begin());

  /* First, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactorT->solveOp,
                        upTriFactorT->csrMat->num_rows, &ALPHA, upTriFactorT->descr,
                        upTriFactorT->csrMat->values->data().get(),
                        upTriFactorT->csrMat->row_offsets->data().get(),
                        upTriFactorT->csrMat->column_indices->data().get(),
                        upTriFactorT->solveInfo,
                        xGPU->data().get(), tempGPU->data().get());CHKERRCUSP(stat);

  /* Then, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactorT->solveOp,
                        loTriFactorT->csrMat->num_rows, &ALPHA, loTriFactorT->descr,
                        loTriFactorT->csrMat->values->data().get(),
                        loTriFactorT->csrMat->row_offsets->data().get(),
                        loTriFactorT->csrMat->column_indices->data().get(),
                        loTriFactorT->solveInfo,
                        tempGPU->data().get(), xGPU->data().get());CHKERRCUSP(stat);

  /* Last, copy the solution, xGPU, into a temporary with the column permutation ... can't be done in place. */
  thrust::copy(thrust::make_permutation_iterator(xGPU->begin(), cusparseTriFactors->cpermIndices->begin()),
               thrust::make_permutation_iterator(xGPU->end(), cusparseTriFactors->cpermIndices->end()),
               tempGPU->begin());

  /* Copy the temporary to the full solution. */
  thrust::copy(tempGPU->begin(), tempGPU->end(), xGPU->begin());

  /* restore */
  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);

  ierr = PetscLogFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering"
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  CUSPARRAY                         *xGPU,*bGPU;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  /* Analyze the matrix and create the transpose ... on the fly */
  if (!loTriFactorT && !upTriFactorT) {
    ierr = MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A);CHKERRQ(ierr);
    loTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
    upTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  }

  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* First, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactorT->solveOp,
                        upTriFactorT->csrMat->num_rows, &ALPHA, upTriFactorT->descr,
                        upTriFactorT->csrMat->values->data().get(),
                        upTriFactorT->csrMat->row_offsets->data().get(),
                        upTriFactorT->csrMat->column_indices->data().get(),
                        upTriFactorT->solveInfo,
                        bGPU->data().get(), tempGPU->data().get());CHKERRCUSP(stat);

  /* Then, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactorT->solveOp,
                        loTriFactorT->csrMat->num_rows, &ALPHA, loTriFactorT->descr,
                        loTriFactorT->csrMat->values->data().get(),
                        loTriFactorT->csrMat->row_offsets->data().get(),
                        loTriFactorT->csrMat->column_indices->data().get(),
                        loTriFactorT->solveInfo,
                        tempGPU->data().get(), xGPU->data().get());CHKERRCUSP(stat);

  /* restore */
  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSolve_SeqAIJCUSPARSE"
static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  CUSPARRAY                         *xGPU,*bGPU;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                    ierr;
  VecType                           t;
  PetscBool                         flg;

  PetscFunctionBegin;
  ierr = VecGetType(bb,&t);CHKERRQ(ierr);
  ierr = PetscStrcmp(t,VECSEQCUSP,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Vector of type %s passed into MatSolve_SeqAIJCUSPARSE (Arg #2). Can only deal with %s\n.",t,VECSEQCUSP);
  ierr = VecGetType(xx,&t);CHKERRQ(ierr);
  ierr = PetscStrcmp(t,VECSEQCUSP,&flg);CHKERRQ(ierr);
  if (!flg) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Vector of type %s passed into MatSolve_SeqAIJCUSPARSE (Arg #3). Can only deal with %s\n.",t,VECSEQCUSP);

  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* First, reorder with the row permutation */
  thrust::copy(thrust::make_permutation_iterator(bGPU->begin(), cusparseTriFactors->rpermIndices->begin()),
               thrust::make_permutation_iterator(bGPU->end(), cusparseTriFactors->rpermIndices->end()),
               xGPU->begin());

  /* Next, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactor->solveOp,
                        loTriFactor->csrMat->num_rows, &ALPHA, loTriFactor->descr,
                        loTriFactor->csrMat->values->data().get(),
                        loTriFactor->csrMat->row_offsets->data().get(),
                        loTriFactor->csrMat->column_indices->data().get(),
                        loTriFactor->solveInfo,
                        xGPU->data().get(), tempGPU->data().get());CHKERRCUSP(stat);

  /* Then, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactor->solveOp,
                        upTriFactor->csrMat->num_rows, &ALPHA, upTriFactor->descr,
                        upTriFactor->csrMat->values->data().get(),
                        upTriFactor->csrMat->row_offsets->data().get(),
                        upTriFactor->csrMat->column_indices->data().get(),
                        upTriFactor->solveInfo,
                        tempGPU->data().get(), xGPU->data().get());CHKERRCUSP(stat);

  /* Last, copy the solution, xGPU, into a temporary with the column permutation ... can't be done in place. */
  thrust::copy(thrust::make_permutation_iterator(xGPU->begin(), cusparseTriFactors->cpermIndices->begin()),
               thrust::make_permutation_iterator(xGPU->end(), cusparseTriFactors->cpermIndices->end()),
               tempGPU->begin());

  /* Copy the temporary to the full solution. */
  thrust::copy(tempGPU->begin(), tempGPU->end(), xGPU->begin());

  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSolve_SeqAIJCUSPARSE_NaturalOrdering"
static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  CUSPARRAY                         *xGPU,*bGPU;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* First, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactor->solveOp,
                        loTriFactor->csrMat->num_rows, &ALPHA, loTriFactor->descr,
                        loTriFactor->csrMat->values->data().get(),
                        loTriFactor->csrMat->row_offsets->data().get(),
                        loTriFactor->csrMat->column_indices->data().get(),
                        loTriFactor->solveInfo,
                        bGPU->data().get(), tempGPU->data().get());CHKERRCUSP(stat);

  /* Next, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactor->solveOp,
                        upTriFactor->csrMat->num_rows, &ALPHA, upTriFactor->descr,
                        upTriFactor->csrMat->values->data().get(),
                        upTriFactor->csrMat->row_offsets->data().get(),
                        upTriFactor->csrMat->column_indices->data().get(),
                        upTriFactor->solveInfo,
                        tempGPU->data().get(), xGPU->data().get());CHKERRCUSP(stat);

  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSECopyToGPU"
static PetscErrorCode MatSeqAIJCUSPARSECopyToGPU(Mat A)
{

  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  PetscInt                     m = A->rmap->n,*ii,*ridx;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;
  hipError_t                  err;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    ierr = PetscLogEventBegin(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
    Mat_SeqAIJCUSPARSEMultStruct_Destroy(&matstruct,cusparsestruct->format);
    try {
      cusparsestruct->nonzerorow=0;
      for (int j = 0; j<m; j++) cusparsestruct->nonzerorow += ((a->i[j+1]-a->i[j])>0);

      if (a->compressedrow.use) {
        m    = a->compressedrow.nrows;
        ii   = a->compressedrow.i;
        ridx = a->compressedrow.rindex;
      } else {
        /* Forcing compressed row on the GPU */
        int k=0;
        ierr = PetscMalloc1(cusparsestruct->nonzerorow+1, &ii);CHKERRQ(ierr);
        ierr = PetscMalloc1(cusparsestruct->nonzerorow, &ridx);CHKERRQ(ierr);
        ii[0]=0;
        for (int j = 0; j<m; j++) {
          if ((a->i[j+1]-a->i[j])>0) {
            ii[k]  = a->i[j];
            ridx[k]= j;
            k++;
          }
        }
        ii[cusparsestruct->nonzerorow] = a->nz;
        m = cusparsestruct->nonzerorow;
      }

      /* allocate space for the triangular factor information */
      matstruct = new Mat_SeqAIJCUSPARSEMultStruct;
      stat = hipsparseCreateMatDescr(&matstruct->descr);CHKERRCUSP(stat);
      stat = hipsparseSetMatIndexBase(matstruct->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSP(stat);
      stat = hipsparseSetMatType(matstruct->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUSP(stat);

      err = hipMalloc((void **)&(matstruct->alpha),sizeof(PetscScalar));CHKERRCUSP(err);
      err = hipMemcpy(matstruct->alpha,&ALPHA,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUSP(err);
      err = hipMalloc((void **)&(matstruct->beta),sizeof(PetscScalar));CHKERRCUSP(err);
      err = hipMemcpy(matstruct->beta,&BETA,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUSP(err);
      stat = hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUSP(stat);

      /* Build a hybrid/ellpack matrix if this option is chosen for the storage */
      if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
/* set the matrix */
        CsrMatrix *matrix= new CsrMatrix;
        matrix->num_rows = m;
        matrix->num_cols = A->cmap->n;
        matrix->num_entries = a->nz;
        matrix->row_offsets = new THRUSTINTARRAY32(m+1);
        matrix->row_offsets->assign(ii, ii + m+1);

        matrix->column_indices = new THRUSTINTARRAY32(a->nz);
        matrix->column_indices->assign(a->j, a->j+a->nz);

        matrix->values = new THRUSTARRAY(a->nz);
        matrix->values->assign(a->a, a->a+a->nz);

/* assign the pointer */
        matstruct->mat = matrix;

      } else if (cusparsestruct->format==MAT_CUSPARSE_ELL || cusparsestruct->format==MAT_CUSPARSE_HYB) {
#if CUDA_VERSION>=4020
        CsrMatrix *matrix= new CsrMatrix;
        matrix->num_rows = m;
        matrix->num_cols = A->cmap->n;
        matrix->num_entries = a->nz;
        matrix->row_offsets = new THRUSTINTARRAY32(m+1);
        matrix->row_offsets->assign(ii, ii + m+1);

        matrix->column_indices = new THRUSTINTARRAY32(a->nz);
        matrix->column_indices->assign(a->j, a->j+a->nz);

        matrix->values = new THRUSTARRAY(a->nz);
        matrix->values->assign(a->a, a->a+a->nz);

        hipsparseHybMat_t hybMat;
        stat = hipsparseCreateHybMat(&hybMat);CHKERRCUSP(stat);
        hipsparseHybPartition_t partition = cusparsestruct->format==MAT_CUSPARSE_ELL ?
          HIPSPARSE_HYB_PARTITION_MAX : HIPSPARSE_HYB_PARTITION_AUTO;
        stat = cusparse_csr2hyb(cusparsestruct->handle, matrix->num_rows, matrix->num_cols,
                                matstruct->descr, matrix->values->data().get(),
                                matrix->row_offsets->data().get(),
                                matrix->column_indices->data().get(),
                                hybMat, 0, partition);CHKERRCUSP(stat);
        /* assign the pointer */
        matstruct->mat = hybMat;

        if (matrix) {
          if (matrix->values) delete (THRUSTARRAY*)matrix->values;
          if (matrix->column_indices) delete (THRUSTINTARRAY32*)matrix->column_indices;
          if (matrix->row_offsets) delete (THRUSTINTARRAY32*)matrix->row_offsets;
          delete (CsrMatrix*)matrix;
        }
#endif
      }

      /* assign the compressed row indices */
      matstruct->cprowIndices = new THRUSTINTARRAY(m);
      matstruct->cprowIndices->assign(ridx,ridx+m);

      /* assign the pointer */
      cusparsestruct->mat = matstruct;

      if (!a->compressedrow.use) {
        ierr = PetscFree(ii);CHKERRQ(ierr);
        ierr = PetscFree(ridx);CHKERRQ(ierr);
      }
      cusparsestruct->workVector = new THRUSTARRAY;
      cusparsestruct->workVector->resize(m);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
    ierr = WaitForGPU();CHKERRCUSP(ierr);

    A->valid_GPU_matrix = PETSC_CUSP_BOTH;

    ierr = PetscLogEventEnd(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCreateVecs_SeqAIJCUSPARSE"
static PetscErrorCode MatCreateVecs_SeqAIJCUSPARSE(Mat mat, Vec *right, Vec *left)
{
  PetscErrorCode ierr;
  PetscInt rbs,cbs;

  PetscFunctionBegin;
  ierr = MatGetBlockSizes(mat,&rbs,&cbs);CHKERRQ(ierr);
  if (right) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),right);CHKERRQ(ierr);
    ierr = VecSetSizes(*right,mat->cmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*right,cbs);CHKERRQ(ierr);
    ierr = VecSetType(*right,VECSEQCUSP);CHKERRQ(ierr);
    ierr = PetscLayoutReference(mat->cmap,&(*right)->map);CHKERRQ(ierr);
  }
  if (left) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),left);CHKERRQ(ierr);
    ierr = VecSetSizes(*left,mat->rmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*left,rbs);CHKERRQ(ierr);
    ierr = VecSetType(*left,VECSEQCUSP);CHKERRQ(ierr);
    ierr = PetscLayoutReference(mat->rmap,&(*left)->map);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

struct VecCUSPPlusEquals
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<1>(t) + thrust::get<0>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqAIJCUSPARSE"
static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  CUSPARRAY                    *xarray,*yarray;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
    CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
    stat = cusparse_csr_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             mat->num_rows, mat->num_cols, mat->num_entries,
                             matstruct->alpha, matstruct->descr, mat->values->data().get(), mat->row_offsets->data().get(),
                             mat->column_indices->data().get(), xarray->data().get(), matstruct->beta,
                             yarray->data().get());CHKERRCUSP(stat);
  } else {
#if CUDA_VERSION>=4020
    hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstruct->mat;
    stat = cusparse_hyb_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             matstruct->alpha, matstruct->descr, hybMat,
                             xarray->data().get(), matstruct->beta,
                             yarray->data().get());CHKERRCUSP(stat);
#endif
  }
  ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (!cusparsestruct->stream) {
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  }
  ierr = PetscLogFlops(2.0*a->nz - cusparsestruct->nonzerorow);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMultTranspose_SeqAIJCUSPARSE"
static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  CUSPARRAY                    *xarray,*yarray;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  if (!matstructT) {
    ierr = MatSeqAIJCUSPARSEGenerateTransposeForMult(A);CHKERRQ(ierr);
    matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  }
  ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(yy,&yarray);CHKERRQ(ierr);

  if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
    CsrMatrix *mat = (CsrMatrix*)matstructT->mat;
    stat = cusparse_csr_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             mat->num_rows, mat->num_cols,
                             mat->num_entries, matstructT->alpha, matstructT->descr,
                             mat->values->data().get(), mat->row_offsets->data().get(),
                             mat->column_indices->data().get(), xarray->data().get(), matstructT->beta,
                             yarray->data().get());CHKERRCUSP(stat);
  } else {
#if CUDA_VERSION>=4020
    hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstructT->mat;
    stat = cusparse_hyb_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             matstructT->alpha, matstructT->descr, hybMat,
                             xarray->data().get(), matstructT->beta,
                             yarray->data().get());CHKERRCUSP(stat);
#endif
  }
  ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (!cusparsestruct->stream) {
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  }
  ierr = PetscLogFlops(2.0*a->nz - cusparsestruct->nonzerorow);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMultAdd_SeqAIJCUSPARSE"
static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  CUSPARRAY                    *xarray,*yarray,*zarray;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  try {
    ierr = VecCopy_SeqCUSP(yy,zz);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(zz,&zarray);CHKERRQ(ierr);

    /* multiply add */
    if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
      CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
    /* here we need to be careful to set the number of rows in the multiply to the
       number of compressed rows in the matrix ... which is equivalent to the
       size of the workVector */
      stat = cusparse_csr_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               mat->num_rows, mat->num_cols,
                               mat->num_entries, matstruct->alpha, matstruct->descr,
                               mat->values->data().get(), mat->row_offsets->data().get(),
                               mat->column_indices->data().get(), xarray->data().get(), matstruct->beta,
                               cusparsestruct->workVector->data().get());CHKERRCUSP(stat);
    } else {
#if CUDA_VERSION>=4020
      hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstruct->mat;
      if (cusparsestruct->workVector->size()) {
        stat = cusparse_hyb_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            matstruct->alpha, matstruct->descr, hybMat,
            xarray->data().get(), matstruct->beta,
            cusparsestruct->workVector->data().get());CHKERRCUSP(stat);
      }
#endif
    }

    /* scatter the data from the temporary into the full vector with a += operation */
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zarray->begin(), matstruct->cprowIndices->begin()))),
        thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zarray->begin(), matstruct->cprowIndices->begin()))) + cusparsestruct->workVector->size(),
        VecCUSPPlusEquals());
    ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(zz,&zarray);CHKERRQ(ierr);

  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMultTransposeAdd_SeqAIJCUSPARSE"
static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  CUSPARRAY                    *xarray,*yarray,*zarray;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  if (!matstructT) {
    ierr = MatSeqAIJCUSPARSEGenerateTransposeForMult(A);CHKERRQ(ierr);
    matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  }

  try {
    ierr = VecCopy_SeqCUSP(yy,zz);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(zz,&zarray);CHKERRQ(ierr);

    /* multiply add with matrix transpose */
    if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
      CsrMatrix *mat = (CsrMatrix*)matstructT->mat;
      /* here we need to be careful to set the number of rows in the multiply to the
         number of compressed rows in the matrix ... which is equivalent to the
         size of the workVector */
      stat = cusparse_csr_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               mat->num_rows, mat->num_cols,
                               mat->num_entries, matstructT->alpha, matstructT->descr,
                               mat->values->data().get(), mat->row_offsets->data().get(),
                               mat->column_indices->data().get(), xarray->data().get(), matstructT->beta,
                               cusparsestruct->workVector->data().get());CHKERRCUSP(stat);
    } else {
#if CUDA_VERSION>=4020
      hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstructT->mat;
      if (cusparsestruct->workVector->size()) {
        stat = cusparse_hyb_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            matstructT->alpha, matstructT->descr, hybMat,
            xarray->data().get(), matstructT->beta,
            cusparsestruct->workVector->data().get());CHKERRCUSP(stat);
      }
#endif
    }

    /* scatter the data from the temporary into the full vector with a += operation */
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zarray->begin(), matstructT->cprowIndices->begin()))),
        thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zarray->begin(), matstructT->cprowIndices->begin()))) + cusparsestruct->workVector->size(),
        VecCUSPPlusEquals());

    ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(zz,&zarray);CHKERRQ(ierr);

  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_SeqAIJCUSPARSE"
static PetscErrorCode MatAssemblyEnd_SeqAIJCUSPARSE(Mat A,MatAssemblyType mode)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatAssemblyEnd_SeqAIJ(A,mode);CHKERRQ(ierr);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  }
  if (mode == MAT_FLUSH_ASSEMBLY) PetscFunctionReturn(0);
  A->ops->mult             = MatMult_SeqAIJCUSPARSE;
  A->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  A->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  A->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

/* --------------------------------------------------------------------------------*/
#undef __FUNCT__
#define __FUNCT__ "MatCreateSeqAIJCUSPARSE"
/*@
   MatCreateSeqAIJCUSPARSE - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format). This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSPARSE library for calculations. For good matrix
   assembly performance the user should preallocate the matrix storage by setting
   the parameter nz (or the array nnz).  By setting these parameters accurately,
   performance during matrix assembly can be increased by more than a factor of 50.

   Collective on MPI_Comm

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows
         (possibly different for each row) or NULL

   Output Parameter:
.  A - the matrix

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradgm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=NULL for PETSc to control dynamic memory
   allocation.  For large problems you MUST preallocate memory or you
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to
   improve numerical efficiency of matrix-vector products and solves. We
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ(), MATSEQAIJCUSPARSE, MATAIJCUSPARSE
@*/
PetscErrorCode  MatCreateSeqAIJCUSPARSE(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt nz,const PetscInt nnz[],Mat *A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
  ierr = MatSetType(*A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation_SeqAIJ(*A,nz,(PetscInt*)nnz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqAIJCUSPARSE"
static PetscErrorCode MatDestroy_SeqAIJCUSPARSE(Mat A)
{
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (A->factortype==MAT_FACTOR_NONE) {
    if (A->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED) {
      ierr = Mat_SeqAIJCUSPARSE_Destroy((Mat_SeqAIJCUSPARSE**)&A->spptr);CHKERRQ(ierr);
    }
  } else {
    ierr = Mat_SeqAIJCUSPARSETriFactors_Destroy((Mat_SeqAIJCUSPARSETriFactors**)&A->spptr);CHKERRQ(ierr);
  }
  ierr = MatDestroy_SeqAIJ(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqAIJCUSPARSE"
PETSC_EXTERN PetscErrorCode MatCreate_SeqAIJCUSPARSE(Mat B)
{
  PetscErrorCode ierr;
  hipsparseStatus_t stat;
  hipsparseHandle_t handle=0;

  PetscFunctionBegin;
  ierr = MatCreate_SeqAIJ(B);CHKERRQ(ierr);
  if (B->factortype==MAT_FACTOR_NONE) {
    /* you cannot check the inode.use flag here since the matrix was just created.
       now build a GPU matrix data structure */
    B->spptr = new Mat_SeqAIJCUSPARSE;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->mat          = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->matTranspose = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->workVector   = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->format       = MAT_CUSPARSE_CSR;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->stream       = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->handle       = 0;
    stat = hipsparseCreate(&handle);CHKERRCUSP(stat);
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->handle       = handle;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->stream       = 0;
  } else {
    /* NEXT, set the pointers to the triangular factors */
    B->spptr = new Mat_SeqAIJCUSPARSETriFactors;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->loTriFactorPtr          = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->upTriFactorPtr          = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->loTriFactorPtrTranspose = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->upTriFactorPtrTranspose = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->rpermIndices            = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->cpermIndices            = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->workVector              = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->handle                  = 0;
    stat = hipsparseCreate(&handle);CHKERRCUSP(stat);
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->handle                  = handle;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->nnz                     = 0;
  }

  B->ops->assemblyend      = MatAssemblyEnd_SeqAIJCUSPARSE;
  B->ops->destroy          = MatDestroy_SeqAIJCUSPARSE;
  B->ops->getvecs          = MatCreateVecs_SeqAIJCUSPARSE;
  B->ops->setfromoptions   = MatSetFromOptions_SeqAIJCUSPARSE;
  B->ops->mult             = MatMult_SeqAIJCUSPARSE;
  B->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  B->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  B->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);

  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;

  ierr = PetscObjectComposeFunction((PetscObject)B, "MatCUSPARSESetFormat_C", MatCUSPARSESetFormat_SeqAIJCUSPARSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*M
   MATSEQAIJCUSPARSE - MATAIJCUSPARSE = "(seq)aijcusparse" - A matrix type to be used for sparse matrices.

   A matrix type type whose data resides on Nvidia GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. The ELL and HYB formats require CUDA 4.2 or later.
   All matrix calculations are performed on Nvidia GPUs using the CUSPARSE library.

   Options Database Keys:
+  -mat_type aijcusparse - sets the matrix type to "seqaijcusparse" during a call to MatSetFromOptions()
.  -mat_cusparse_storage_format csr - sets the storage format of matrices (for MatMult and factors in MatSolve) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
.  -mat_cusparse_mult_storage_format csr - sets the storage format of matrices (for MatMult) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).

  Level: beginner

.seealso: MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

PETSC_EXTERN PetscErrorCode MatGetFactor_seqaijcusparse_cusparse(Mat,MatFactorType,Mat*);


#undef __FUNCT__
#define __FUNCT__ "MatSolverPackageRegister_CUSPARSE"
PETSC_EXTERN PetscErrorCode MatSolverPackageRegister_CUSPARSE(void)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolverPackageRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_LU,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverPackageRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_CHOLESKY,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverPackageRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_ILU,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverPackageRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_ICC,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "Mat_SeqAIJCUSPARSE_Destroy"
static PetscErrorCode Mat_SeqAIJCUSPARSE_Destroy(Mat_SeqAIJCUSPARSE **cusparsestruct)
{
  hipsparseStatus_t stat;
  hipsparseHandle_t handle;

  PetscFunctionBegin;
  if (*cusparsestruct) {
    Mat_SeqAIJCUSPARSEMultStruct_Destroy(&(*cusparsestruct)->mat,(*cusparsestruct)->format);
    Mat_SeqAIJCUSPARSEMultStruct_Destroy(&(*cusparsestruct)->matTranspose,(*cusparsestruct)->format);
    delete (*cusparsestruct)->workVector;
    if (handle = (*cusparsestruct)->handle) {
      stat = hipsparseDestroy(handle);CHKERRCUSP(stat);
    }
    delete *cusparsestruct;
    *cusparsestruct = 0;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "CsrMatrix_Destroy"
static PetscErrorCode CsrMatrix_Destroy(CsrMatrix **mat)
{
  PetscFunctionBegin;
  if (*mat) {
    delete (*mat)->values;
    delete (*mat)->column_indices;
    delete (*mat)->row_offsets;
    delete *mat;
    *mat = 0;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "Mat_SeqAIJCUSPARSETriFactorStruct_Destroy"
static PetscErrorCode Mat_SeqAIJCUSPARSETriFactorStruct_Destroy(Mat_SeqAIJCUSPARSETriFactorStruct **trifactor)
{
  hipsparseStatus_t stat;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (*trifactor) {
    if ((*trifactor)->descr) { stat = hipsparseDestroyMatDescr((*trifactor)->descr);CHKERRCUSP(stat); }
    if ((*trifactor)->solveInfo) { stat = cusparseDestroySolveAnalysisInfo((*trifactor)->solveInfo);CHKERRCUSP(stat); }
    ierr = CsrMatrix_Destroy(&(*trifactor)->csrMat);CHKERRQ(ierr);
    delete *trifactor;
    *trifactor = 0;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "Mat_SeqAIJCUSPARSEMultStruct_Destroy"
static PetscErrorCode Mat_SeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSEMultStruct **matstruct,MatCUSPARSEStorageFormat format)
{
  CsrMatrix        *mat;
  hipsparseStatus_t stat;
  hipError_t      err;

  PetscFunctionBegin;
  if (*matstruct) {
    if ((*matstruct)->mat) {
      if (format==MAT_CUSPARSE_ELL || format==MAT_CUSPARSE_HYB) {
        hipsparseHybMat_t hybMat = (hipsparseHybMat_t)(*matstruct)->mat;
        stat = hipsparseDestroyHybMat(hybMat);CHKERRCUSP(stat);
      } else {
        mat = (CsrMatrix*)(*matstruct)->mat;
        CsrMatrix_Destroy(&mat);
      }
    }
    if ((*matstruct)->descr) { stat = hipsparseDestroyMatDescr((*matstruct)->descr);CHKERRCUSP(stat); }
    delete (*matstruct)->cprowIndices;
    if ((*matstruct)->alpha) { err=hipFree((*matstruct)->alpha);CHKERRCUSP(err); }
    if ((*matstruct)->beta) { err=hipFree((*matstruct)->beta);CHKERRCUSP(err); }
    delete *matstruct;
    *matstruct = 0;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "Mat_SeqAIJCUSPARSETriFactors_Destroy"
static PetscErrorCode Mat_SeqAIJCUSPARSETriFactors_Destroy(Mat_SeqAIJCUSPARSETriFactors** trifactors)
{
  hipsparseHandle_t handle;
  hipsparseStatus_t stat;

  PetscFunctionBegin;
  if (*trifactors) {
    Mat_SeqAIJCUSPARSETriFactorStruct_Destroy(&(*trifactors)->loTriFactorPtr);
    Mat_SeqAIJCUSPARSETriFactorStruct_Destroy(&(*trifactors)->upTriFactorPtr);
    Mat_SeqAIJCUSPARSETriFactorStruct_Destroy(&(*trifactors)->loTriFactorPtrTranspose);
    Mat_SeqAIJCUSPARSETriFactorStruct_Destroy(&(*trifactors)->upTriFactorPtrTranspose);
    delete (*trifactors)->rpermIndices;
    delete (*trifactors)->cpermIndices;
    delete (*trifactors)->workVector;
    if (handle = (*trifactors)->handle) {
      stat = hipsparseDestroy(handle);CHKERRCUSP(stat);
    }
    delete *trifactors;
    *trifactors = 0;
  }
  PetscFunctionReturn(0);
}

