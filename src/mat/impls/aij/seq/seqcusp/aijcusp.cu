/*
  Defines the basic matrix operations for the AIJ (compressed row)
  matrix storage format.
*/
#define PETSC_SKIP_COMPLEX
#define PETSC_SKIP_SPINLOCK

#include <petscconf.h>
#include <../src/mat/impls/aij/seq/aij.h>          /*I "petscmat.h" I*/
#include <petscbt.h>
#include <../src/vec/vec/impls/dvecimpl.h>
#include <petsc/private/vecimpl.h>
#undef VecType
#include <../src/mat/impls/aij/seq/seqcusp/cuspmatimpl.h>

const char *const MatCUSPStorageFormats[] = {"CSR","DIA","ELL","MatCUSPStorageFormat","MAT_CUSP_",0};

#undef __FUNCT__
#define __FUNCT__ "MatCUSPSetStream"
PetscErrorCode MatCUSPSetStream(Mat A,const hipStream_t stream)
{
  Mat_SeqAIJCUSP *cuspstruct = (Mat_SeqAIJCUSP*)A->spptr;

  PetscFunctionBegin;
  cuspstruct->stream = stream;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCUSPSetFormat_SeqAIJCUSP"
PetscErrorCode MatCUSPSetFormat_SeqAIJCUSP(Mat A,MatCUSPFormatOperation op,MatCUSPStorageFormat format)
{
  Mat_SeqAIJCUSP *cuspMat = (Mat_SeqAIJCUSP*)A->spptr;

  PetscFunctionBegin;
  switch (op) {
  case MAT_CUSP_MULT:
    cuspMat->format = format;
    break;
  case MAT_CUSP_ALL:
    cuspMat->format = format;
    break;
  default:
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPFormatOperation. Only MAT_CUSP_MULT and MAT_CUSP_ALL are currently supported.",op);
  }
  PetscFunctionReturn(0);
}

/*@
   MatCUSPSetFormat - Sets the storage format of CUSP matrices for a particular
   operation. Only the MatMult operation can use different GPU storage formats
   for AIJCUSP matrices.

   Not Collective

   Input Parameters:
+  A - Matrix of type SEQAIJCUSP
.  op - MatCUSPFormatOperation. SEQAIJCUSP matrices support MAT_CUSP_MULT and MAT_CUSP_ALL. MPIAIJCUSP matrices support MAT_CUSP_MULT_DIAG, MAT_CUSP_MULT_OFFDIAG, and MAT_CUSP_ALL.
-  format - MatCUSPStorageFormat (one of MAT_CUSP_CSR, MAT_CUSP_DIA, MAT_CUSP_ELL)

   Output Parameter:

   Level: intermediate

.seealso: MatCUSPStorageFormat, MatCUSPFormatOperation
@*/
#undef __FUNCT__
#define __FUNCT__ "MatCUSPSetFormat"
PetscErrorCode MatCUSPSetFormat(Mat A,MatCUSPFormatOperation op,MatCUSPStorageFormat format)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A, MAT_CLASSID,1);
  ierr = PetscTryMethod(A, "MatCUSPSetFormat_C",(Mat,MatCUSPFormatOperation,MatCUSPStorageFormat),(A,op,format));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSetFromOptions_SeqAIJCUSP"
PetscErrorCode MatSetFromOptions_SeqAIJCUSP(PetscOptionItems *PetscOptionsObject,Mat A)
{
  Mat_SeqAIJCUSP       *cuspMat = (Mat_SeqAIJCUSP*)A->spptr;
  PetscErrorCode       ierr;
  MatCUSPStorageFormat format;
  PetscBool            flg;

  PetscFunctionBegin;
  ierr = PetscOptionsHead(PetscOptionsObject,"SeqAIJCUSP options");CHKERRQ(ierr);
  ierr = PetscObjectOptionsBegin((PetscObject)A);
  ierr = PetscOptionsEnum("-mat_cusp_mult_storage_format","sets storage format of (seq)aijcusp gpu matrices for SpMV",
                          "MatCUSPSetFormat",MatCUSPStorageFormats,(PetscEnum)cuspMat->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = MatCUSPSetFormat(A,MAT_CUSP_MULT,format);CHKERRQ(ierr);
  }
  ierr = PetscOptionsEnum("-mat_cusp_storage_format","sets storage format of (seq)aijcusp gpu matrices for SpMV",
                          "MatCUSPSetFormat",MatCUSPStorageFormats,(PetscEnum)cuspMat->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = MatCUSPSetFormat(A,MAT_CUSP_ALL,format);CHKERRQ(ierr);
  }
  ierr = PetscOptionsEnd();CHKERRQ(ierr);
  PetscFunctionReturn(0);

}


#undef __FUNCT__
#define __FUNCT__ "MatCUSPCopyToGPU"
PetscErrorCode MatCUSPCopyToGPU(Mat A)
{

  Mat_SeqAIJCUSP *cuspstruct = (Mat_SeqAIJCUSP*)A->spptr;
  Mat_SeqAIJ     *a          = (Mat_SeqAIJ*)A->data;
  PetscInt       m           = A->rmap->n,*ii,*ridx;
  CUSPMATRIX     *mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    ierr = PetscLogEventBegin(MAT_CUSPCopyToGPU,A,0,0,0);CHKERRQ(ierr);
    /*
      It may be possible to reuse nonzero structure with new matrix values but
      for simplicity and insured correctness we delete and build a new matrix on
      the GPU. Likely a very small performance hit.
    */
    if (cuspstruct->mat) {
      try {
        if (cuspstruct->format==MAT_CUSP_ELL)
          delete (CUSPMATRIXELL *) cuspstruct->mat;
        else if (cuspstruct->format==MAT_CUSP_DIA)
          delete (CUSPMATRIXDIA *) cuspstruct->mat;
        else
          delete (CUSPMATRIX *) cuspstruct->mat;
      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
      }
    }
    try {
      cuspstruct->nonzerorow=0;
      for (int j = 0; j<m; j++) cuspstruct->nonzerorow += ((a->i[j+1]-a->i[j])>0);
      if (a->compressedrow.use) {
        m    = a->compressedrow.nrows;
        ii   = a->compressedrow.i;
        ridx = a->compressedrow.rindex;
      } else {
        /* Forcing compressed row on the GPU */
        int k=0;
        ierr = PetscMalloc1(cuspstruct->nonzerorow+1, &ii);CHKERRQ(ierr);
        ierr = PetscMalloc1(cuspstruct->nonzerorow, &ridx);CHKERRQ(ierr);
        ii[0]=0;
        for (int j = 0; j<m; j++) {
          if ((a->i[j+1]-a->i[j])>0) {
            ii[k]  = a->i[j];
            ridx[k]= j;
            k++;
          }
        }
        ii[cuspstruct->nonzerorow] = a->nz;
        m = cuspstruct->nonzerorow;
      }

      /* now build matrix */
      mat = new CUSPMATRIX;
      mat->resize(m,A->cmap->n,a->nz);
      mat->row_offsets.assign(ii,ii+m+1);
      mat->column_indices.assign(a->j,a->j+a->nz);
      mat->values.assign(a->a,a->a+a->nz);

      /* convert to other formats if selected */
      if (a->compressedrow.use || cuspstruct->format==MAT_CUSP_CSR) {
        cuspstruct->mat = mat;
        cuspstruct->format = MAT_CUSP_CSR;
      } else {
        if (cuspstruct->format==MAT_CUSP_ELL) {
          CUSPMATRIXELL *ellMat = new CUSPMATRIXELL(*mat);
          cuspstruct->mat = ellMat;
        } else {
          CUSPMATRIXDIA *diaMat = new CUSPMATRIXDIA(*mat);
          cuspstruct->mat = diaMat;
        }
        delete (CUSPMATRIX*) mat;
      }

      /* assign the compressed row indices */
      if (cuspstruct->indices) delete (CUSPINTARRAYGPU*)cuspstruct->indices;
      cuspstruct->indices = new CUSPINTARRAYGPU;
      cuspstruct->indices->assign(ridx,ridx+m);

      /* free the temporaries */
      if (!a->compressedrow.use) {
        ierr = PetscFree(ii);CHKERRQ(ierr);
        ierr = PetscFree(ridx);CHKERRQ(ierr);
      }
      if (cuspstruct->tempvec) delete (CUSPARRAY*)cuspstruct->tempvec;
      cuspstruct->tempvec = new CUSPARRAY;
      cuspstruct->tempvec->resize(m);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
    }
    ierr = WaitForGPU();CHKERRCUSP(ierr);

    A->valid_GPU_matrix = PETSC_CUSP_BOTH;

    ierr = PetscLogEventEnd(MAT_CUSPCopyToGPU,A,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCUSPCopyFromGPU"
PetscErrorCode MatCUSPCopyFromGPU(Mat A, CUSPMATRIX *Agpu)
{
  Mat_SeqAIJCUSP *cuspstruct = (Mat_SeqAIJCUSP*) A->spptr;
  Mat_SeqAIJ     *a          = (Mat_SeqAIJ*) A->data;
  PetscInt       m           = A->rmap->n;
  CUSPMATRIX *mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* if the data is stored in non-CSR format, create a temporary */
  if (cuspstruct->format==MAT_CUSP_ELL) {
    mat = new CUSPMATRIX(*((CUSPMATRIXELL*)cuspstruct->mat));
  } else if (cuspstruct->format==MAT_CUSP_DIA) {
    mat = new CUSPMATRIX(*((CUSPMATRIXDIA*)cuspstruct->mat));
  } else {
    mat = (CUSPMATRIX*) cuspstruct->mat;
  }

  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED) {
    if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED) {
      try {
        mat = Agpu;
        if (a->compressedrow.use) SETERRQ(PETSC_COMM_WORLD, PETSC_ERR_ARG_WRONG, "Cannot handle row compression for GPU matrices");
        else {
          PetscInt i;

          if (m+1 != (PetscInt) mat->row_offsets.size()) SETERRQ2(PETSC_COMM_WORLD, PETSC_ERR_ARG_SIZ, "GPU matrix has %d rows, should be %d", mat->row_offsets.size()-1, m);
          a->nz           = mat->values.size();
          a->maxnz        = a->nz; /* Since we allocate exactly the right amount */
          A->preallocated = PETSC_TRUE;
          if (a->singlemalloc) {
            if (a->a) {ierr = PetscFree3(a->a,a->j,a->i);CHKERRQ(ierr);}
          } else {
            if (a->i) {ierr = PetscFree(a->i);CHKERRQ(ierr);}
            if (a->j) {ierr = PetscFree(a->j);CHKERRQ(ierr);}
            if (a->a) {ierr = PetscFree(a->a);CHKERRQ(ierr);}
          }
          ierr = PetscMalloc3(a->nz,&a->a,a->nz,&a->j,m+1,&a->i);CHKERRQ(ierr);
          ierr = PetscLogObjectMemory((PetscObject)A, a->nz*(sizeof(PetscScalar)+sizeof(PetscInt))+(m+1)*sizeof(PetscInt));CHKERRQ(ierr);

          a->singlemalloc = PETSC_TRUE;
          thrust::copy(mat->row_offsets.begin(), mat->row_offsets.end(), a->i);
          thrust::copy(mat->column_indices.begin(), mat->column_indices.end(), a->j);
          thrust::copy(mat->values.begin(), mat->values.end(), a->a);
          /* Setup row lengths */
          if (a->imax) {ierr = PetscFree2(a->imax,a->ilen);CHKERRQ(ierr);}
          ierr = PetscMalloc2(m,&a->imax,m,&a->ilen);CHKERRQ(ierr);
          ierr = PetscLogObjectMemory((PetscObject)A, 2*m*sizeof(PetscInt));CHKERRQ(ierr);
          for (i = 0; i < m; ++i) a->imax[i] = a->ilen[i] = a->i[i+1] - a->i[i];
          /* a->diag?*/
        }
        cuspstruct->tempvec = new CUSPARRAY;
        cuspstruct->tempvec->resize(m);
      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF, PETSC_ERR_LIB, "CUSP error: %s", ex);
      }
    }
    /* This assembly prevents resetting the flag to PETSC_CUSP_CPU and recopying */
    ierr = MatAssemblyBegin(A, MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
    ierr = MatAssemblyEnd(A, MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);

    A->valid_GPU_matrix = PETSC_CUSP_BOTH;

    /* delete the temporary */
    if (cuspstruct->format==MAT_CUSP_ELL || cuspstruct->format==MAT_CUSP_DIA)
      delete (CUSPMATRIX*) mat;
  } else SETERRQ(PETSC_COMM_WORLD, PETSC_ERR_ARG_WRONG, "Only valid for unallocated GPU matrices");
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCreateVecs_SeqAIJCUSP"
PetscErrorCode MatCreateVecs_SeqAIJCUSP(Mat mat, Vec *right, Vec *left)
{
  PetscErrorCode ierr;
  PetscInt rbs,cbs;

  PetscFunctionBegin;
  ierr = MatGetBlockSizes(mat,&rbs,&cbs);CHKERRQ(ierr);
  if (right) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),right);CHKERRQ(ierr);
    ierr = VecSetSizes(*right,mat->cmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*right,cbs);CHKERRQ(ierr);
    ierr = VecSetType(*right,VECSEQCUSP);CHKERRQ(ierr);
    ierr = PetscLayoutReference(mat->cmap,&(*right)->map);CHKERRQ(ierr);
  }
  if (left) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),left);CHKERRQ(ierr);
    ierr = VecSetSizes(*left,mat->rmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*left,rbs);CHKERRQ(ierr);
    ierr = VecSetType(*left,VECSEQCUSP);CHKERRQ(ierr);
    ierr = PetscLayoutReference(mat->rmap,&(*left)->map);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqAIJCUSP"
PetscErrorCode MatMult_SeqAIJCUSP(Mat A,Vec xx,Vec yy)
{
  Mat_SeqAIJ       *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode   ierr;
  Mat_SeqAIJCUSP   *cuspstruct = (Mat_SeqAIJCUSP*)A->spptr;
  PetscBool        usecprow = a->compressedrow.use;
  CUSPARRAY        *xarray=NULL,*yarray=NULL;
  static PetscBool cite = PETSC_FALSE;

  PetscFunctionBegin;
  /*
    DM 1/28/2014: As of version 0.4.0 cusp does not handle the case of
    zero matrices well.  It produces segfaults on some platforms.
    Therefore we manually check for the case of a zero matrix here.
  */
  if (a->nz == 0) {
    PetscFunctionReturn(0);
  }
  ierr = PetscCitationsRegister("@incollection{msk2013,\n  author = {Victor Minden and Barry F. Smith and Matthew G. Knepley},\n  title = {Preliminary Implementation of {PETSc} Using {GPUs}},\n  booktitle = {GPU Solutions to Multi-scale Problems in Science and Engineering},\n  series = {Lecture Notes in Earth System Sciences},\n  editor = {David A. Yuen and Long Wang and Xuebin Chi and Lennart Johnsson and Wei Ge and Yaolin Shi},\n  publisher = {Springer Berlin Heidelberg},\n  pages = {131--140},\n  year = {2013},\n}\n",&cite);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(yy,&yarray);CHKERRQ(ierr);
  try {
    if (usecprow) {
      /* use compressed row format */
      CUSPMATRIX *mat = (CUSPMATRIX*)cuspstruct->mat;
      cusp::multiply(*mat,*xarray,*cuspstruct->tempvec);
      ierr = VecSet_SeqCUSP(yy,0.0);CHKERRQ(ierr);
      thrust::copy(cuspstruct->tempvec->begin(),cuspstruct->tempvec->end(),thrust::make_permutation_iterator(yarray->begin(),cuspstruct->indices->begin()));
    } else {
      /* do not use compressed row format */
      if (cuspstruct->format==MAT_CUSP_ELL) {
        CUSPMATRIXELL *mat = (CUSPMATRIXELL*)cuspstruct->mat;
        cusp::multiply(*mat,*xarray,*yarray);
      } else if (cuspstruct->format==MAT_CUSP_DIA) {
        CUSPMATRIXDIA *mat = (CUSPMATRIXDIA*)cuspstruct->mat;
        cusp::multiply(*mat,*xarray,*yarray);
      } else {
        CUSPMATRIX *mat = (CUSPMATRIX*)cuspstruct->mat;
        cusp::multiply(*mat,*xarray,*yarray);
      }
    }

  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (!cuspstruct->stream) {
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  }
  ierr = PetscLogFlops(2.0*a->nz - cuspstruct->nonzerorow);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


struct VecCUSPPlusEquals
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<1>(t) + thrust::get<0>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "MatMultAdd_SeqAIJCUSP"
PetscErrorCode MatMultAdd_SeqAIJCUSP(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode ierr;
  PetscBool      usecprow = a->compressedrow.use;
  Mat_SeqAIJCUSP *cuspstruct = (Mat_SeqAIJCUSP*)A->spptr;
  CUSPARRAY      *xarray     = NULL,*yarray=NULL,*zarray=NULL;

  PetscFunctionBegin;
  /*
    DM 1/28/2014: As of version 0.4.0 cusp does not handle the case of
    zero matrices well.  It produces segfaults on some platforms.
    Therefore we manually check for the case of a zero matrix here.
  */
  if (a->nz == 0) {
    PetscFunctionReturn(0);
  }
  try {
    ierr = VecCopy_SeqCUSP(yy,zz);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(zz,&zarray);CHKERRQ(ierr);

    if (usecprow) {
      /* use compressed row format */
      CUSPMATRIX *mat = (CUSPMATRIX*)cuspstruct->mat;
      cusp::multiply(*mat,*xarray,*cuspstruct->tempvec);
      thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cuspstruct->tempvec->begin(),
                                                                    thrust::make_permutation_iterator(zarray->begin(), cuspstruct->indices->begin()))),
                       thrust::make_zip_iterator(thrust::make_tuple(cuspstruct->tempvec->end(),
                                                                    thrust::make_permutation_iterator(zarray->end(),cuspstruct->indices->end()))),
                       VecCUSPPlusEquals());
    } else {

      if (cuspstruct->format==MAT_CUSP_ELL) {
        CUSPMATRIXELL *mat = (CUSPMATRIXELL*)cuspstruct->mat;
        cusp::multiply(*mat,*xarray,*cuspstruct->tempvec);
      } else if (cuspstruct->format==MAT_CUSP_DIA) {
        CUSPMATRIXDIA *mat = (CUSPMATRIXDIA*)cuspstruct->mat;
        cusp::multiply(*mat,*xarray,*cuspstruct->tempvec);
      } else {
        CUSPMATRIX *mat = (CUSPMATRIX*)cuspstruct->mat;
        cusp::multiply(*mat,*xarray,*cuspstruct->tempvec);
      }

      if (zarray->size() == cuspstruct->indices->size()) {
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cuspstruct->tempvec->begin(),zarray->begin())),
            thrust::make_zip_iterator(thrust::make_tuple(cuspstruct->tempvec->end(),zarray->end())),
            VecCUSPPlusEquals());
      } else {
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cuspstruct->tempvec->begin(),
                thrust::make_permutation_iterator(zarray->begin(), cuspstruct->indices->begin()))),
            thrust::make_zip_iterator(thrust::make_tuple(cuspstruct->tempvec->end(),
                thrust::make_permutation_iterator(zarray->end(),cuspstruct->indices->end()))),
            VecCUSPPlusEquals());
      }
    }
    ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(zz,&zarray);CHKERRQ(ierr);

  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_SeqAIJCUSP"
PetscErrorCode MatAssemblyEnd_SeqAIJCUSP(Mat A,MatAssemblyType mode)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatAssemblyEnd_SeqAIJ(A,mode);CHKERRQ(ierr);
  ierr = MatCUSPCopyToGPU(A);CHKERRQ(ierr);
  if (mode == MAT_FLUSH_ASSEMBLY) PetscFunctionReturn(0);
  A->ops->mult    = MatMult_SeqAIJCUSP;
  A->ops->multadd = MatMultAdd_SeqAIJCUSP;
  PetscFunctionReturn(0);
}

/* --------------------------------------------------------------------------------*/
#undef __FUNCT__
#define __FUNCT__ "MatCreateSeqAIJCUSP"
/*@
   MatCreateSeqAIJCUSP - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format).  This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSP library for calculations. For good matrix
   assembly performance the user should preallocate the matrix storage by setting
   the parameter nz (or the array nnz).  By setting these parameters accurately,
   performance during matrix assembly can be increased by more than a factor of 50.


   Collective on MPI_Comm

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows
         (possibly different for each row) or NULL

   Output Parameter:
.  A - the matrix

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradigm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=NULL for PETSc to control dynamic memory
   allocation.  For large problems you MUST preallocate memory or you
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to
   improve numerical efficiency of matrix-vector products and solves. We
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ()

@*/
PetscErrorCode  MatCreateSeqAIJCUSP(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt nz,const PetscInt nnz[],Mat *A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
  ierr = MatSetType(*A,MATSEQAIJCUSP);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation_SeqAIJ(*A,nz,(PetscInt*)nnz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqAIJCUSP"
PetscErrorCode MatDestroy_SeqAIJCUSP(Mat A)
{
  PetscErrorCode ierr;
  Mat_SeqAIJCUSP *cuspcontainer = (Mat_SeqAIJCUSP*)A->spptr;

  PetscFunctionBegin;
  try {
    if (A->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED) {
      if (cuspcontainer->format==MAT_CUSP_ELL) {
        delete (CUSPMATRIXELL*)(cuspcontainer->mat);
      } else if (cuspcontainer->format==MAT_CUSP_DIA) {
        delete (CUSPMATRIXDIA*)(cuspcontainer->mat);
      } else {
        delete (CUSPMATRIX*)(cuspcontainer->mat);
      }
      if (cuspcontainer->indices) delete (CUSPINTARRAYGPU*)cuspcontainer->indices;
      if (cuspcontainer->tempvec) delete (CUSPARRAY*)cuspcontainer->tempvec;
    }
    delete cuspcontainer;
    A->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSP error: %s", ex);
  }
  /*this next line is because MatDestroy tries to PetscFree spptr if it is not zero, and PetscFree only works if the memory was allocated with PetscNew or PetscMalloc, which don't call the constructor */
  A->spptr = 0;
  ierr     = MatDestroy_SeqAIJ(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

extern PetscErrorCode MatSetValuesBatch_SeqAIJCUSP(Mat, PetscInt, PetscInt, PetscInt*,const PetscScalar*);

#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqAIJCUSP"
PETSC_EXTERN PetscErrorCode MatCreate_SeqAIJCUSP(Mat B)
{
  PetscErrorCode ierr;
  Mat_SeqAIJ     *aij;

  PetscFunctionBegin;
  ierr            = MatCreate_SeqAIJ(B);CHKERRQ(ierr);
  aij             = (Mat_SeqAIJ*)B->data;
  aij->inode.use  = PETSC_FALSE;
  B->ops->mult    = MatMult_SeqAIJCUSP;
  B->ops->multadd = MatMultAdd_SeqAIJCUSP;
  B->spptr        = new Mat_SeqAIJCUSP;

  if (B->factortype==MAT_FACTOR_NONE) {
    ((Mat_SeqAIJCUSP*)B->spptr)->mat     = 0;
    ((Mat_SeqAIJCUSP*)B->spptr)->tempvec = 0;
    ((Mat_SeqAIJCUSP*)B->spptr)->indices = 0;
    ((Mat_SeqAIJCUSP*)B->spptr)->nonzerorow = 0;
    ((Mat_SeqAIJCUSP*)B->spptr)->stream = 0;
    ((Mat_SeqAIJCUSP*)B->spptr)->format = MAT_CUSP_CSR;
  }

  B->ops->assemblyend    = MatAssemblyEnd_SeqAIJCUSP;
  B->ops->destroy        = MatDestroy_SeqAIJCUSP;
  B->ops->getvecs        = MatCreateVecs_SeqAIJCUSP;
  B->ops->setvaluesbatch = MatSetValuesBatch_SeqAIJCUSP;
  B->ops->setfromoptions = MatSetFromOptions_SeqAIJCUSP;

  ierr = PetscObjectComposeFunction((PetscObject)B,"MatCUSPSetFormat_C", MatCUSPSetFormat_SeqAIJCUSP);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQAIJCUSP);CHKERRQ(ierr);

  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  PetscFunctionReturn(0);
}

/*M
   MATSEQAIJCUSP - MATAIJCUSP = "aijcusp" = "seqaijcusp" - A matrix type to be used for sparse matrices.

   A matrix type type whose data resides on Nvidia GPUs. These matrices are in CSR format by
   default. All matrix calculations are performed using the CUSP library. DIA and ELL formats are 
   also available.

   Options Database Keys:
+  -mat_type aijcusp - sets the matrix type to "seqaijcusp" during a call to MatSetFromOptions()
.  -mat_cusp_storage_format csr - sets the storage format of matrices for MatMult during a call to MatSetFromOptions(). Other options include dia (diagonal) or ell (ellpack).
-  -mat_cusp_mult_storage_format csr - sets the storage format of matrices for MatMult during a call to MatSetFromOptions(). Other options include dia (diagonal) or ell (ellpack).

  Level: beginner

.seealso: MatCreateSeqAIJCUSP(), MATAIJCUSP, MatCreateAIJCUSP(), MatCUSPSetFormat(), MatCUSPStorageFormat, MatCUSPFormatOperation
M*/

